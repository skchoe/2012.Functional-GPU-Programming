
#include <hip/hip_runtime.h>

	#define TILE_DIM 24
	//#define BLOCK_ROWS 16 

__global__ void copy(float *odata, float* idata, int width,  
                                     int height, int nreps, const int BLOCK_ROWS) 
{ 
  int xIndex = blockIdx.x*TILE_DIM + threadIdx.x; 
  int yIndex = blockIdx.y*TILE_DIM + threadIdx.y; 
 
  int index  = xIndex + width*yIndex; 
  for (int r=0; r < nreps; r++) { 
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      odata[index+i*width] = idata[index+i*width]; 
    } 
  } 
} 

__global__ void copySharedMem(float *odata, float *idata,  
                          int width, int height, int nreps,const int BLOCK_ROWS) 
{ 
	__shared__ float tile[TILE_DIM][TILE_DIM]; 
 
  int xIndex = blockIdx.x*TILE_DIM + threadIdx.x; 
  int yIndex = blockIdx.y*TILE_DIM + threadIdx.y; 
   
  int index  = xIndex + width*yIndex; 
  for (int r=0; r < nreps; r++) { 
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      tile[threadIdx.y+i][threadIdx.x] =  
        idata[index+i*width]; 
    } 
   
    __syncthreads(); 
   
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      odata[index+i*width] =  
        tile[threadIdx.y+i][threadIdx.x]; 
    } 
  } 
}


__global__ void transposeNaive(float *odata, float* idata,  
                         int width, int height, int nreps,const int BLOCK_ROWS) 
{ 
  int xIndex = blockIdx.x*TILE_DIM + threadIdx.x; 
  int yIndex = blockIdx.y*TILE_DIM + threadIdx.y; 
 
  int index_in  = xIndex + width * yIndex; 
  int index_out = yIndex + height * xIndex; 
  for (int r=0; r < nreps; r++) { 
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      odata[index_out+i] = idata[index_in+i*width]; 
    } 
  } 
}

__global__ void transposeCoalesced(float *odata,  
            float *idata, int width, int height, int nreps,const int BLOCK_ROWS) 
{ 
  __shared__ float tile[TILE_DIM][TILE_DIM]; 
 
  int xIndex = blockIdx.x*TILE_DIM + threadIdx.x; 
  int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;   
  int index_in = xIndex + (yIndex)*width; 
 
  xIndex = blockIdx.y * TILE_DIM + threadIdx.x; 
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y; 
  int index_out = xIndex + (yIndex)*height; 
 
  for (int r=0; r < nreps; r++) { 
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      tile[threadIdx.y+i][threadIdx.x] =  
        idata[index_in+i*width]; 
    } 
   
    __syncthreads(); 
   
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      odata[index_out+i*height] =  
        tile[threadIdx.x][threadIdx.y+i]; 
    } 
  } 
}

__global__ void transposeNoBankConflicts(float *odata,  
            float *idata, int width, int height, int nreps,const int BLOCK_ROWS) 
{ 
  __shared__ float tile[TILE_DIM][TILE_DIM+1]; 
 
  int xIndex = blockIdx.x*TILE_DIM + threadIdx.x; 
  int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;   
  int index_in = xIndex + (yIndex)*width; 
 
  xIndex = blockIdx.y * TILE_DIM + threadIdx.x; 
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y; 
  int index_out = xIndex + (yIndex)*height; 
 
  for (int r=0; r < nreps; r++) { 
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      tile[threadIdx.y+i][threadIdx.x] =  
        idata[index_in+i*width]; 
    } 
   
    __syncthreads(); 
   
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      odata[index_out+i*height] =  
        tile[threadIdx.x][threadIdx.y+i]; 
    } 
  } 
}

__global__ void transposeFineGrained(float *odata,  
           float *idata, int width, int height,  int nreps,const int BLOCK_ROWS) 
{ 
  __shared__ float block[TILE_DIM][TILE_DIM+1]; 
 
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x; 
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y; 
  int index = xIndex + (yIndex)*width; 
 
  for (int r=0; r<nreps; r++) { 
    for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) { 
      block[threadIdx.y+i][threadIdx.x] =  
        idata[index+i*width]; 
    }   
      
    __syncthreads(); 
 
    for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) { 
      odata[index+i*height] =  
        block[threadIdx.x][threadIdx.y+i]; 
    } 
  } 
}

__global__ void transposeCoarseGrained(float *odata,  
      float *idata, int width, int height, int nreps,const int BLOCK_ROWS) 
{ 
  __shared__ float block[TILE_DIM][TILE_DIM+1]; 
 
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x; 
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y; 
  int index_in = xIndex + (yIndex)*width; 
 
  xIndex = blockIdx.y * TILE_DIM + threadIdx.x; 
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y; 
  int index_out = xIndex + (yIndex)*height; 
 
  for (int r=0; r<nreps; r++) { 
    for (int i=0; i<TILE_DIM; i += BLOCK_ROWS) { 
      block[threadIdx.y+i][threadIdx.x] =  
        idata[index_in+i*width]; 
    } 
   
    __syncthreads(); 
 
    for (int i=0; i<TILE_DIM; i += BLOCK_ROWS) { 
      odata[index_out+i*height] =  
        block[threadIdx.y+i][threadIdx.x]; 
    } 
  } 
}

__global__ void transposeDiagonal(float *odata,  
            float *idata, int width, int height, int nreps,const int BLOCK_ROWS) 
{ 
  __shared__ float tile[TILE_DIM][TILE_DIM+1]; 
 
  int blockIdx_x, blockIdx_y; 
 
  // diagonal reordering 
  if (width == height) { 
    blockIdx_y = blockIdx.x; 
    blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x; 
  } else { 
    int bid = blockIdx.x + gridDim.x*blockIdx.y; 
    blockIdx_y = bid%gridDim.y; 
    blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x; 
  }     
 
  int xIndex = blockIdx_x*TILE_DIM + threadIdx.x; 
  int yIndex = blockIdx_y*TILE_DIM + threadIdx.y;   
  int index_in = xIndex + (yIndex)*width; 
 
  xIndex = blockIdx_y*TILE_DIM + threadIdx.x; 
  yIndex = blockIdx_x*TILE_DIM + threadIdx.y; 
  int index_out = xIndex + (yIndex)*height; 
 
  for (int r=0; r < nreps; r++) { 
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      tile[threadIdx.y+i][threadIdx.x] =  
        idata[index_in+i*width]; 
    } 
   
    __syncthreads(); 
   
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) { 
      odata[index_out+i*height] =  
        tile[threadIdx.x][threadIdx.y+i]; 
    } 
  } 
}

