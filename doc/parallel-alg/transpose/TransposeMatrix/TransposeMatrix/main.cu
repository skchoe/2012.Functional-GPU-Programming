#include <stdio.h> 
#include "kernel.cu" 
#include <cutil_inline.h>


// kernels transpose/copy a tile of TILE_DIM x TILE_DIM elements 
// using a TILE_DIM x BLOCK_ROWS thread block, so that each thread 
// transposes TILE_DIM/BLOCK_ROWS elements.  TILE_DIM must be an  
// integral multiple of BLOCK_ROWS 
 
 
// Number of repetitions used for timing.   
 
#define NUM_REPS  100 


//extern "C" void (*computeTransposeGold)(float* transposeGold, float *h_idata, int size_x,int size_y);
 
void computeTransposeGold( float* transposeGold, float* h_idata, 
            const int size_x, const int size_y ) 
{
    // transpose matrix
    for( int y = 0; y < size_y; ++y) 
    {
        for(  int x = 0; x < size_x; ++x) 
        {
			transposeGold[(x * size_y) + y] = h_idata[(y * size_x) + x];
        }
    }  
}

int 
main( int argc, char** argv)  
{ 
  
	 /* printf("Enter the size_x and size_y\n");
	scanf("%d,%d", &size_x,size_y);*/
	// set matrix size 
	//int size_x = 1024 , size_y = 1024;  
	int size_x = 0, size_y = 0;
	int  temp_BLOCK_ROWS = 0;

	if(argc > 1)
	{
		size_x = atoi(argv[1]);
		size_y = atoi(argv[2]);
		//temp_TILE_DIM = atoi(argv[3]);
		temp_BLOCK_ROWS = atoi(argv[3]);
	}
    //const int TILE_DIM = temp_TILE_DIM;
	const int BLOCK_ROWS = temp_BLOCK_ROWS;
	// kernel pointer and descriptor 
  void (*kernel)(float *, float *,int,int,int,const int); 
  char *kernelName; 

const int mem_size = sizeof(float) * size_x*size_y; 
 
  // allocate host memory 
  float *h_idata = (float*) malloc(mem_size); 
  float *h_odata = (float*) malloc(mem_size); 
  float *transposeGold = (float *) malloc(mem_size);   
  float *gold; 
  
 // float *compareF(float *,float *,const int);
 
  // execution configuration parameters 
  dim3 grid(size_x/TILE_DIM, size_y/TILE_DIM),   
       threads(TILE_DIM,BLOCK_ROWS); 
 
  // CUDA events 
  hipEvent_t start, stop; 
 
  // size of memory required to store the matrix 
  


 
  // allocate device memory 
  float *d_idata, *d_odata; 
  hipMalloc( (void**) &d_idata, mem_size); 
  hipMalloc( (void**) &d_odata, mem_size); 
 
  // initalize host data computeTransposeGold
  for(int i = 0; i < (size_x*size_y); ++i) 
    h_idata[i] = (float) i; 
   
  // copy host data to device 
  hipMemcpy(d_idata, h_idata, mem_size,    
             hipMemcpyHostToDevice ); 
  // Compute reference transpose solution 
  computeTransposeGold(transposeGold, h_idata, size_x, size_y);
 
  // print out common data for all kernels 
  printf("\nMatrix size: %dx%d, tile: %dx%d, block: %dx%d\n\n",  
  size_x, size_y, TILE_DIM, TILE_DIM, TILE_DIM, BLOCK_ROWS); 
   
  printf("Kernel\t\t\tLoop over kernel\tLoop within kernel\tLoop over kernel Time\t\tLoop within kernel Time\n"); 
  printf("------\t\t\t----------------\t------------------\t---------------------\t\t-----------------------\n"); 
 
  // 
  // loop over different kernels 
  // 
 
  for (int k = 0; k<8; k++) { 
    // set kernel pointer 
    switch (k) { 
    case 0: 
      kernel = &copy;  
      kernelName = "simple copy           "; break; 
    case 1: 
      kernel = &copySharedMem;                   
      kernelName = "shared memory copy    "; break; 
    case 2: 
		   kernel = &transposeCoalesced;              
      kernelName = "coalesced transpose   "; break;
      
    case 3: 
	  kernel = &transposeNaive;                  
      kernelName = "naive transpose       "; break; 
    case 4: 
    kernel = &transposeNoBankConflicts;        
      kernelName = "no bank conflict trans"; break; 
    case 5: 
      kernel = &transposeCoarseGrained;          
      kernelName = "coarse-grained        "; break; 
    case 6: 
      kernel = &transposeFineGrained;            
      kernelName = "fine-grained          "; break; 
    case 7: 
      kernel = &transposeDiagonal;               
      kernelName = "diagonal transpose    "; break; 
    }       
 
    // set reference solution 
    // NB: fine- and coarse-grained kernels are not full 
    //     transposes, so bypass check 
    if (kernel == &copy || kernel == &copySharedMem) { 
      gold = h_idata; 
    } else if (kernel == &transposeCoarseGrained ||  
               kernel == &transposeFineGrained) { 
      gold = h_odata; 
    } else { 
      gold = transposeGold; 
    } 
 
     
    // initialize events, EC parameters 
    hipEventCreate(&start); 
    hipEventCreate(&stop); 
 
    // warmup to avoid timing startup 

	 kernel<<<grid, threads>>>(d_odata, d_idata, size_x,size_y, 1, BLOCK_ROWS); 
 
    // take measurements for loop over kernel launches 
    hipEventRecord(start, 0); 
    for (int i=0; i < NUM_REPS; i++) { 
      kernel<<<grid, threads>>>(d_odata, d_idata,size_x,size_y,1, BLOCK_ROWS); 
    } 
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop); 
    float outerTime; 
    hipEventElapsedTime(&outerTime, start, stop);     
 
    hipMemcpy(h_odata,d_odata, mem_size, hipMemcpyDeviceToHost); 
    CUTBoolean res = cutComparef(gold, h_odata, size_x*size_y); 
    if (res != 1) 
      printf("*** %s kernel FAILED ***\n", kernelName); 
	else
		printf("***Loop over kernel test PASSED***\n");
 
    // take measurements for loop inside kernel 
    hipEventRecord(start, 0); 
    kernel<<<grid,threads>>> 
        (d_odata, d_idata, size_x, size_y, NUM_REPS, BLOCK_ROWS); 
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop); 
    float innerTime; 
    hipEventElapsedTime(&innerTime, start, stop);     
 
    hipMemcpy(h_odata,d_odata, mem_size, hipMemcpyDeviceToHost); 
    res = cutComparef(gold, h_odata, size_x*size_y); 
    if (res != 1) 
      printf("*** %s kernel FAILED ***\n", kernelName); 
	else
		printf("***Loop over kernel test PASSED***\n");
     
    // report effective bandwidths 
    float outerBandwidth =  
       2.0f*1000.0f*mem_size/(1024*1024*1024)/(outerTime/NUM_REPS); 
    float innerBandwidth =  
       2.0f*1000.0f*mem_size/(1024*1024*1024)/(innerTime/NUM_REPS); 
	float loopOuterTime = outerTime/(NUM_REPS);
	float loopInnerTime = innerTime/(NUM_REPS);
    printf("%s\t%5.2f GB/s\t\t%5.2f GB/s\t\t%.5f ms\t\t\t%.5f ms\n",  
       kernelName, outerBandwidth, innerBandwidth,loopOuterTime, loopInnerTime); 
  } 
   
  // cleanup 
 
  free(h_idata); free(h_odata); free(transposeGold); 
  hipFree(d_idata); hipFree(d_odata); 
  hipEventDestroy(start); hipEventDestroy(stop); 
   
  return 0; 
} 