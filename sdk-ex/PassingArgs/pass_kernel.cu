
#include <hip/hip_runtime.h>
//#ifndef _MATRIXMUL_KERNEL_H_
//#define _MATRIXMUL_KERNEL_H_

extern "C"

/* Signature:
*/
__global__ void
pass_kernel(
int arg1, int arg2, int arg3, int arg4, int arg5, int arg6, int arg7, int arg8, int arg9, 
int* bargs,
int* barg1)
{
    int i = threadIdx.x;
    switch (i) {
    case 0:
	bargs[0] = arg1 * 10;
	break;
    case 1:
	bargs[1] = arg2 * 10;
	break;
    case 2:
	bargs[2] = arg3 * 10;
	break;
    case 3:
	bargs[3] = arg4 * 10;
	break;
    case 4:
	bargs[4] = arg5 * 10;
	break;
    case 5:
	bargs[5] = arg6 * 10;
	break;
    case 6:
	bargs[6] = arg7 * 10;
	break;
    case 7:
	bargs[7] = arg8 * 10;
	break;
    case 8:
	bargs[8] = arg9 * 10;
	break;
    }

    *barg1 = arg9;
}
