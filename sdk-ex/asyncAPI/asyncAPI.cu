#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>

extern "C"

__global__ void increment_kernel(int *g_data, int inc_value)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   g_data[idx] = g_data[idx] + inc_value;
}

int correct_output(int *data, const int n, const int x)
{
    for(int i = 0; i < n; i++)
        if(data[i] != x)
            return 0;
    return 1;
}

int main(int argc, char *argv[])
{
    CUT_DEVICE_INIT(argc, argv);

    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

#ifdef __DEVICE_EMULATION__
    n = 1024;   // reduce workload for emulation
#endif
    
    // allocate host memory
    int *a = 0;
    CUDA_SAFE_CALL( hipHostMalloc((void**)&a, nbytes) );
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a=0;
    CUDA_SAFE_CALL( hipMalloc((void**)&d_a, nbytes) );
    CUDA_SAFE_CALL( hipMemset(d_a, 255, nbytes) );

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks  = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    CUDA_SAFE_CALL( hipEventCreate(&start) );
    CUDA_SAFE_CALL( hipEventCreate(&stop)  );
    
    unsigned int timer;
    CUT_SAFE_CALL(  cutCreateTimer(&timer)  );
    CUT_SAFE_CALL(  cutResetTimer(timer)    );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    CUT_SAFE_CALL( cutStartTimer(timer) );
        hipEventRecord(start, 0);
        hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
        increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
        hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
        hipEventRecord(stop, 0);
    CUT_SAFE_CALL(cutStopTimer(timer) );

    printf("start status = %d\n", hipEventQuery(start));
//    hipDeviceSynchronize();
    printf("stop status = %d\n", hipEventQuery(stop));

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;
    while( hipEventQuery(stop) == hipErrorNotReady )
    {
        counter++;
    }
   
    printf("counting finished: stop status = %d\n", hipEventQuery(stop));
    CUDA_SAFE_CALL(hipEventElapsedTime(&gpu_time, start, stop) );

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", cutGetTimerValue(timer) );
    printf("CPU executed %d iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    printf("--------------------------------------------------------------\n");
    if( correct_output(a, n, value) )
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");

    // release resources
    CUDA_SAFE_CALL( hipEventDestroy(start) );
    CUDA_SAFE_CALL( hipEventDestroy(stop) );
    CUDA_SAFE_CALL( hipHostFree(a) );
    CUDA_SAFE_CALL( hipFree(d_a) );

    CUT_EXIT(argc, argv);

    return 0;
}
