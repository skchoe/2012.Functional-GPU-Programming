#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

/*
(: increment_kernel ((Listof Integer) Integer -> (Listof Integer)))
(define (increment_kernel lin inc_value)
  (map (lambda (x) (+ inc_value x)) lin))

*/

#define ALIGN_UP(offset, alignment) (offset) = ((offset) + (alignment) - 1) & ~((alignment) - 1)


extern "C"
__global__ void increment_kernel(int *g_data, int N, int *go_data, int* No, int inc_value)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

   *No = N;
   if(idx < N) go_data[idx] = g_data[idx] + inc_value;
}

int correct_output(int *data, const int n, const int x)
{
    for(int i = 0; i < n; i++) {
	if(i==n-1) printf ("correct_output-%d:[%d]%d, w/val:%d\n", n, i, data[i], x);
        if(data[i] != x)
            return 0;
    }
    return 1;
}

////////////////////////////////////////////////////////////////////////////////
//! Init for CUDA
////////////////////////////////////////////////////////////////////////////////
static hipError_t
initCuda(hipFunction_t *pKernel, 
	int argc, char** argv)
{
    hipDevice_t cuDevice = 0;
    hipCtx_t cuContext;
    hipModule_t cuModule;    
    hipFunction_t cuFunction = 0;

    CUT_DEVICE_INIT_DRV(cuDevice, argc, argv);

    CUdevprop dp;
    cuDeviceGetProperties(&dp, cuDevice);
    int a[3];
    a[0] = dp.maxThreadsDim[0];
    a[1] = dp.maxThreadsDim[1];
    a[2] = dp.maxThreadsDim[2];
    printf("cuDevice prop:Max threads = %d, %d, %d\n", a[0], a[1], a[2]);

    hipError_t status = hipCtxCreate( &cuContext, 0, cuDevice );
    printf ("Ctx Creation: %d\n", status);
    if ( hipSuccess != status )
    {    
      cuCtxDetach(cuContext);
      return status;
    }

    status = hipModuleLoad(&cuModule, "data/asyncAPIDrv.sm_10.cubin");
    printf ("ModuleLoad: %d\n", status);

//    cutFree(module_path);
    if ( hipSuccess != status )
    {    
      cuCtxDetach(cuContext);
      return status;
    } 

    // Global function
    status = hipModuleGetFunction( &cuFunction, cuModule, "increment_kernel" );
    printf("function loading: %d\n", status);
    if ( hipSuccess != status)
    {    
      cuCtxDetach(cuContext);
      return status;
    }     

    *pKernel = cuFunction;

    return hipSuccess;
}

int main(int argc, char *argv[])
{
    hipFunction_t kernel = NULL;
    CU_SAFE_CALL(initCuda(&kernel, argc, argv));

    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

    printf("Initial length of array :%d\n", n);


    // allocate host memory
    int init_value = 1;

    int *a = NULL;
    CU_SAFE_CALL(hipMemAllocHost((void**)&a, nbytes));
    //a = (int*)malloc(nbytes);
    for(int i=0 ; i < n ; i++)
      a[i] = init_value;
    //memset(a, 0, nbytes);

    int* A = NULL;
    CU_SAFE_CALL(hipMemAllocHost((void**)&A, nbytes));
    //A = (int*)malloc(nbytes);
    memset(A, 0, nbytes);

    int* oN = NULL;
    CU_SAFE_CALL(hipMemAllocHost((void**)&oN, sizeof(int)));
    //oN = (int*)malloc(sizeof(int));
    memset(oN, 0, sizeof(int));

    hipStream_t strm;
    hipStreamCreateWithFlags(&strm, 0);
/*
printf("Free host mem = %d, %d, %c: \n", n-1, sizeof(int), a[n-1]);
printf("Free host mem = %d: \n", hipHostFree(a));
*/
    // allocate device memory
    hipDeviceptr_t d_a;
    CU_SAFE_CALL( hipMalloc(&d_a, nbytes));
    CU_SAFE_CALL( hipMemsetD32(d_a, 255, n));

    // alloc dev mem for return
    hipDeviceptr_t do_a;
    CU_SAFE_CALL( hipMalloc(&do_a, nbytes));
    CU_SAFE_CALL( hipMemsetD32(do_a, 255, n));

    // alloc dev mem for length of return
    hipDeviceptr_t o_n;
    CU_SAFE_CALL( hipMalloc(&o_n, sizeof(int)));
    CU_SAFE_CALL( hipMemsetD32(o_n, 255, 1));

    // create cuda event handles
    hipEvent_t start, stop;
    CU_SAFE_CALL( hipEventCreateWithFlags(&start, 0) );
    CU_SAFE_CALL( hipEventCreateWithFlags(&stop, 0)  );

    unsigned int timer;
    CUT_SAFE_CALL(  cutCreateTimer(&timer)  );
    CUT_SAFE_CALL(  cutResetTimer(timer)    );
    CU_SAFE_CALL( hipCtxSynchronize() );
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    CUT_SAFE_CALL( cutStartTimer(timer) );
        hipEventRecord(start, 0);

      printf("memcpy H to D 1= %d\n", hipMemcpyHtoDAsync(d_a, a, nbytes, 0));

      int offset = 0;
      void* ptr;
      ptr = (void*)(size_t)d_a;
      ALIGN_UP(offset, __alignof(ptr));
      printf("1. device array set = %d, offset = %d\n", cuParamSetv(kernel, offset, &ptr, sizeof(ptr)), offset);
      offset += sizeof(void*);

      ALIGN_UP(offset, __alignof(n));
      printf("2. array size set = %d, offset = %d\n", cuParamSeti(kernel, offset, n), offset);
      offset += sizeof(int);

      ptr = (void*)(size_t)do_a;
      ALIGN_UP(offset, __alignof(ptr));
      printf("3. return dev addr set= %d, offset = %d\n", cuParamSetv(kernel, offset, &ptr, sizeof(ptr)), offset);
      offset += sizeof(void*);

      ptr = (void*)(size_t)o_n;
      ALIGN_UP(offset, __alignof(ptr));
      printf("4. return length addr set= %d, offset = %d\n", cuParamSetv(kernel, offset, &ptr, sizeof(ptr)), offset);
      offset += sizeof(void*);

      ALIGN_UP(offset, __alignof(value));
      printf("5. value set: %d, offset = %d\n", cuParamSeti(kernel, offset, value), offset);
      offset += sizeof(int);

      printf("6. param setisze set:%d, offset = %d\n", cuParamSetSize(kernel, offset), offset);

      printf("setblock = %d\n", cuFuncSetBlockShape(kernel, 512, 1, 1));

      printf("Launching kernel = %d\n", cuLaunchGrid(kernel, n/512, 1));

      printf("DtoH copy:%d\n", hipMemcpyDtoHAsync(A, do_a, nbytes, strm));
      //printf("DtoH copy:%d\n", hipMemcpyDtoH(A, do_a, nbytes));

      printf ("Fetching array length = %d\n", hipMemcpyDtoHAsync(oN, o_n, sizeof(int), strm));
      //printf ("Fetching array length = %d\n", hipMemcpyDtoH(oN, o_n, sizeof(int)));

      hipEventRecord(stop, 0);

    CUT_SAFE_CALL( cutStopTimer(timer) );

    // have CPU do some work while waiting for stage 1 to finish
    int counter = 0;
    while( hipEventQuery(stop) == hipErrorNotReady )
    {
        counter++;
	//printf("stop? = %d\n", hipEventQuery(stop));

    }
    printf("stop finally? = %d\n", hipEventQuery(stop));
    CU_SAFE_CALL( hipEventElapsedTime(&gpu_time, start, stop));

    hipStreamSynchronize(strm);
    printf ("Output array length = %d\n", *oN);

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", cutGetTimerValue(timer) );
    printf("CPU executed %d iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    printf("--------------------------------------------------------------\n");
    if( correct_output(A, n, init_value + value) )
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");


/*
      for(int i = 0 ; i < n ; i++)
      printf ("[%d]%d\t", i, A[i]);
      printf ("\n");
      */

    // release resources

    CU_SAFE_CALL( hipStreamDestroy(strm));
    CU_SAFE_CALL( hipEventDestroy(start) );
    CU_SAFE_CALL( hipEventDestroy(stop) );
    CU_SAFE_CALL( hipHostFree(a) );
    CU_SAFE_CALL( hipHostFree(A) );
    CU_SAFE_CALL( hipHostFree(oN) );
    //free(oN);
    CU_SAFE_CALL( hipFree(d_a) );

    CUT_EXIT(argc, argv);

    return 0;
}
