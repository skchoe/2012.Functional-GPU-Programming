#include "hip/hip_runtime.h"
/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <cutil.h>

// includes, kernels
#include <cppIntegration_kernel.cu>
//
////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C" void
computeGold(char* reference, char* idata, const unsigned int len);
extern "C" void
computeGold2(int2* reference, int2* idata, const unsigned int len);


////////////////////////////////////////////////////////////////////////////////
//! Init for CUDA
////////////////////////////////////////////////////////////////////////////////
//    CU_SAFE_CALL(initCudaContext(&cuDevice, &cuContext, argc, (char**)argv));
static hipError_t
initCudaContext(hipCtx_t *pcuContext, hipDevice_t *pcuDevice,
	int argc, char** argv)
{
    CUT_DEVICE_INIT_DRV(*pcuDevice, argc, argv);

    hipError_t status = hipCtxCreate( pcuContext, 0, *pcuDevice );
    printf ("Ctx Creation: %d\n", status);
    if ( hipSuccess != status )
    {    
      cuCtxDetach(*pcuContext);
      return status;
    }
    return hipSuccess;
}

static hipError_t
loadModule(hipModule_t* pcuModule, hipCtx_t* pcuContext, char* cubinName)
{
    hipError_t status = hipModuleLoad(pcuModule, cubinName);
    printf ("ModuleLoad: %s: %d\n", cubinName, status);

    if ( hipSuccess != status )
    {    
      cuCtxDetach(*pcuContext);
      return status;
    }
    return hipSuccess;
}

static hipError_t
loadKernel(hipFunction_t* pcuFunction, hipModule_t cuModule, char* kernelName)
{
    // Global function
    hipFunction_t cuFunction;
    hipError_t status = hipModuleGetFunction( &cuFunction, cuModule, kernelName);
    printf("function loading: %d\n", status);
    if ( hipSuccess != status)
    {    
      *pcuFunction = NULL;
      return status;
    }     

    *pcuFunction = cuFunction;

    return hipSuccess;
}


////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" void
runTest(const int argc, const char** argv, char* data, int2* data_int2, unsigned int len)
{

    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;

    initCudaContext(&cuContext, &cuDevice, argc, (char**)argv);
    loadModule(&cuModule, &cuContext, "data/cppIntegrationDrv.sm_10.cubin");

    hipFunction_t kernel, kernel2;
    loadKernel(&kernel, cuModule, "kernel");
    loadKernel(&kernel2, cuModule, "kernel2");

    CUT_DEVICE_INIT_DRV(cuDevice, argc, argv);

    const unsigned int num_threads = len / 4;
    CUT_CONDITION(0 == (len % 4));
    const unsigned int mem_size = sizeof(char) * len;
    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    hipDeviceptr_t d_data;
    CU_SAFE_CALL(hipMalloc(&d_data, mem_size));
    // copy host memory to device
    CU_SAFE_CALL(hipMemcpyHtoD(d_data, data, mem_size));
printf("00\n");                             
    // allocate device memory for int2 version
    hipDeviceptr_t d_data_int2;
    CU_SAFE_CALL(hipMalloc(&d_data_int2, mem_size_int2));
    // copy host memory to device
    CU_SAFE_CALL(hipMemcpyHtoD(d_data_int2, data_int2, mem_size_int2));
printf("01\n");                             

    // kernel calling
    // setup execution parameters for kernel
    CU_SAFE_CALL(cuFuncSetBlockShape(kernel, num_threads, 1, 1));
    CU_SAFE_CALL(cuParamSeti(kernel, 0, d_data));
    CU_SAFE_CALL(cuParamSetSize(kernel, 4));
    CU_SAFE_CALL(cuLaunchGrid(kernel, 1, 1));
printf("02\n");                             

    // kernel2 calling
    // setup execution parameters for kernel
    CU_SAFE_CALL(cuFuncSetBlockShape(kernel2, len, 1, 1));
    CU_SAFE_CALL(cuParamSeti(kernel2, 0, d_data_int2));
    CU_SAFE_CALL(cuParamSetSize(kernel2, 4));
    CU_SAFE_CALL(cuLaunchGrid(kernel2, 1, 1));
printf("03\n");                             

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // compute reference solutions
    char* reference = (char*) malloc(mem_size);
    computeGold(reference, data, len);
    int2* reference2 = (int2*) malloc(mem_size_int2);
    computeGold2(reference2, data_int2, len);

    // copy results from device to host
    CU_SAFE_CALL(hipMemcpyDtoH(data, d_data, mem_size));
    CU_SAFE_CALL(hipMemcpyDtoH(data_int2, d_data_int2, mem_size_int2));

    // check result
    bool success = true;
    for(unsigned int i = 0; i < len; i++ )
    {
        if( reference[i] != data[i] || 
	    reference2[i].x != data_int2[i].x || 
	    reference2[i].y != data_int2[i].y)
            success = false;
    }
    printf("Test %s\n", success ? "PASSED" : "FAILED");

    // cleanup memory
    CU_SAFE_CALL(hipFree(d_data));
    CU_SAFE_CALL(hipFree(d_data_int2));
    free(reference);
    free(reference2);
}
