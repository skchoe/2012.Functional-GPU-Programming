#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <math_functions.h>

extern "C"
__global__ void
cpyTest(float* answer, int size, float cnst)
{
  int i=0;
  for(i=0;i<size;i++)
    answer[i] = i * cnst;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    int sf = sizeof(float);
    int size = 1000;
    float scale = 2.3f;

    float* d_0;
    CUDA_SAFE_CALL(hipMalloc( (void**) &d_0, sf * size));

    dim3 threads(1, 1);
    dim3 grids(1, 1);
    // Calling kernel
    cpyTest<<<grids, threads>>>(d_0, size, scale);


    float *h_0 = (float*)malloc(sf*size);
    CUDA_SAFE_CALL(hipMemcpy(h_0, d_0, sf*size, hipMemcpyDeviceToHost));

    int i;
    for(i=0;i<size;i++)
      printf("%dth answer = %f\n", i, h_0[i]);
    free(h_0);

    CUT_EXIT(argc, argv);
}
