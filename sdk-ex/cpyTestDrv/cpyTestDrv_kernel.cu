
#include <hip/hip_runtime.h>
//#ifndef _MATRIXMUL_KERNEL_H_
//#define _MATRIXMUL_KERNEL_H_

/*

(define (gpu-info)
  (let* ([info (cuGPUinfo)])
    (values (gridDim-x info) ......)))

(: cpyTestDrv_kernel ((Listof Float) Integer -> (Listof Float) (Listof Integer) Integer)
(define (cpyTestDrv_kernel d_array_in d_single_in)
  (let*-values  ([(d_array_out) (take d_array_in 0)]
         	[(memstruct) (gpu-info)]
         	[(d_single_out) d_single_in])
    (values d_array_out memstruct d_single_out)))


*/

extern "C"

/* Signature:
  float* d_array_in, int count, uint single_in 
  ->
  float* d_array_out, int* memstruct, uint* single_out
*/
__global__ void
cpyTestDrv_kernel(float* d_array_out, float* d_array_in, int count,
        unsigned int* memstruct,
        unsigned int* d_single_out, unsigned int d_single_in)
{
    // copy - single value
    *d_single_out = d_single_in;

    // copy of default variables
    unsigned int gdm_x = gridDim.x;
    unsigned int gdm_y = gridDim.y;
    unsigned int gdm_z = gridDim.z;
    unsigned int bdm_x = blockDim.x;
    unsigned int bdm_y = blockDim.y;
    unsigned int bdm_z = blockDim.z;
    unsigned int bid_x = blockIdx.x;
    unsigned int bid_y = blockIdx.y;
    unsigned int bid_z = blockIdx.z;
    unsigned int tid_x = threadIdx.x;
    unsigned int tid_y = threadIdx.y;
    unsigned int tid_z = threadIdx.z;

    *memstruct = gdm_x;
    *(memstruct+1) = gdm_y;
    *(memstruct+2) = gdm_z;
    *(memstruct+3) = bdm_x;
    *(memstruct+4) = bdm_y;
    *(memstruct+5) = bdm_z;
    *(memstruct+6) = bid_x;
    *(memstruct+7) = bid_y;
    *(memstruct+8) = bid_z;
    *(memstruct+9) = tid_x;
    *(memstruct+10) = tid_y;
    *(memstruct+11) = tid_z;

    // copy of array variables
    for(int j = 0 ; j < count ; j++)
    {
      *(d_array_out+j) = d_array_in[j];
      *(d_array_out+j) = j;
    }
}
