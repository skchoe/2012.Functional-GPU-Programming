
#include <hip/hip_runtime.h>
//#ifndef _MATRIXMUL_KERNEL_H_
//#define _MATRIXMUL_KERNEL_H_

/*

(define (gpu-info)
  (let* ([info (cuGPUinfo)])
    (values (gridDim-x info) ......)))

(: test_kernel ((Listof Float) Integer -> (Listof Float) (Listof Integer) Integer)
(define (test_kernel d_array_in d_single_in)
  (let*-values  ([(d_array_out) (take d_array_in 0)]
         	[(memstruct) (gpu-info)]
         	[(d_single_out) d_single_in])
    (values d_array_out memstruct d_single_out)))


*/

extern "C"

/* Signature:
  float* d_array_in, int count, uint single_in 
  ->
  float* d_array_out, uint* d_array_len, int* memstruct, uint* single_out
*/
/*
In typed/Racket, (test_kernel) has type:
(Vectorof Float) Integer Integer -> (Vectorof Float) Integer Integer
*/
__global__ void
test_kernel(
float* d_array_in, 
uint count,
int d_single_in, 
float* d_array_out,
uint* d_array_out_len,
int* d_single_out
) 
{
    // copy - single value
    *d_single_out = d_single_in;

*(d_array_out+0) = d_array_in[0];
*(d_array_out+1) = d_array_in[1];
*(d_array_out+2) = d_array_in[2];
*(d_array_out+3) = d_array_in[3];
*(d_array_out+4) = d_array_in[4];

*d_array_out_len = count;
*d_single_out = d_single_in;

// what's problem in my for loop?

//    // copy of array variables
//    for(int j = 0 ; j < count ; j++)
//    {
//      *(d_array_out+j) = d_array_in[j];
//    //  *(d_array_out+j) = j;
//    //  *(d_array_in+j) = j;
//    }
   *d_array_out_len = count;
/*
    // copy of default variables
    unsigned int gdm_x = gridDim.x;
    unsigned int gdm_y = gridDim.y;
    unsigned int gdm_z = gridDim.z;
    unsigned int bdm_x = blockDim.x;
    unsigned int bdm_y = blockDim.y;
    unsigned int bdm_z = blockDim.z;
    unsigned int bid_x = blockIdx.x;
    unsigned int bid_y = blockIdx.y;
    unsigned int bid_z = blockIdx.z;
    unsigned int tid_x = threadIdx.x;
    unsigned int tid_y = threadIdx.y;
    unsigned int tid_z = threadIdx.z;

    *memstruct = gdm_x;
    *(memstruct+1) = gdm_y;
    *(memstruct+2) = gdm_z;
    *(memstruct+3) = bdm_x;
    *(memstruct+4) = bdm_y;
    *(memstruct+5) = bdm_z;
    *(memstruct+6) = bid_x;
    *(memstruct+7) = bid_y;
    *(memstruct+8) = bid_z;
    *(memstruct+9) = tid_x;
    *(memstruct+10) = tid_y;
    *(memstruct+11) = tid_z;
*/
}
