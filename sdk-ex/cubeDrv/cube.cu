#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, GL
#include <GL/glew.h>
#include <GL/glut.h>

// includes
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_gl_error.h>
#include <cudaGL.h>


////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 8;
const unsigned int mesh_height = 9;

// vbo variables
GLuint *vboId;

float anim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

int flagG = 0;

const float vertices [] = 
{1.0, 1.0, 1.0,  -1.0, 1.0, 1.0,  -1.0, -1.0, 1.0,  1.0, -1.0, 1.0, //0-1-2-3
1.0, -1.0, -1.0,  -1.0, -1.0, -1.0,  -1.0, 1.0, -1.0,  1.0, 1.0, -1.0,//4-7-6-5
-1.0, -1.0, -1.0,  1.0, -1.0, -1.0,  1.0, -1.0, 1.0,  -1.0, -1.0, 1.0,//7-4-3-2
-1.0, 1.0, 1.0,  -1.0, 1.0, -1.0,  -1.0, -1.0, -1.0,  -1.0, -1.0, 1.0,//1-6-7-2
1.0, 1.0, 1.0,  1.0, 1.0, -1.0,  -1.0, 1.0, -1.0,  -1.0, 1.0, 1.0, //0-5-6-1
1.0, 1.0, 1.0,  1.0, -1.0, 1.0,  1.0, -1.0, -1.0,  1.0, 1.0, -1.0, //0-3-4-5
};

const float normals  [] = 
{0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 0.0, 0.0, 1.0,// v0-v1-v2-v3
0.0, 0.0, -1.0, 0.0, 0.0, -1.0, 0.0,  0.0, -1.0, 0.0, 0.0, -1.0,// v4-v7-v6-v5
0.0, -1.0, 0.0, 0.0, -1.0,  0.0, 0.0, -1.0,  0.0, 0.0, -1.0, 0.0,// v7-v4-v3-v2
-1.0, 0.0, 0.0, -1.0, 0.0, 0.0, -1.0, 0.0, 0.0, -1.0, 0.0, 0.0,// v1-v6-v7-v2
0.0, 1.0, 0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 0.0,  0.0, 1.0, 0.0,// v0-v5-v6-v1
1.0, 0.0, 0.0, 1.0, 0.0, 0.0, 1.0, 0.0, 0.0,  1.0, 0.0, 0.0,// v0-v3-v4-v5
};

const float colors [] = 
{1.0, 1.0, 1.0,  1.0, 1.0, 0.0,  1.0, 0.0, 0.0,  1.0, 0.0, 1.0,// v0-v1-v2-v3
0.0, 0.0, 1.0,  0.0, 0.0, 0.0,  0.0, 1.0, 0.0,  0.0, 1.0, 1.0, // v4-v7-v6-v5
0.0, 0.0, 0.0,  0.0, 0.0, 1.0,  1.0, 0.0, 1.0,  1.0, 0.0, 0.0, // v7-v4-v3-v2
1.0, 1.0, 0.0,  0.0, 1.0, 0.0,  0.0, 0.0, 0.0,  1.0, 0.0, 0.0, // v1-v6-v7-v2
1.0, 1.0, 1.0,  0.0, 1.0, 1.0,  0.0, 1.0, 0.0,  1.0, 1.0, 0.0, // v0-v5-v6-v1
1.0, 1.0, 1.0,  1.0, 0.0, 1.0,  0.0, 0.0, 1.0,  0.0, 1.0, 1.0, // v0-v3-v4-v5
}; 
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// GL functionality
CUTBoolean initGL();
void createVBO( GLuint* vbo);
void deleteVBO( GLuint* vbo);

// rendering callbacks
void display();
void keyboard( unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

// Cuda functionality
void runCuda( GLuint* vbo, hipFunction_t knl);

hipFunction_t kernel;
hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Init for CUDA
////////////////////////////////////////////////////////////////////////////////
static hipError_t
initCuda(hipFunction_t *pKernel, 
	int argc, char** argv)
{
    hipFunction_t cuFunction = 0;

    CUT_DEVICE_INIT_DRV(cuDevice);

    hipError_t status = hipCtxCreate( &cuContext, 0, cuDevice );
    printf ("Ctx Creation: %d\n", status);
    if ( hipSuccess != status )
    {    
      cuCtxDetach(cuContext);
      return status;
    }

    status = hipModuleLoad(&cuModule, "data/cube_kernel.cubin");
    printf ("ModuleLoad: %d\n", status);

//    cutFree(module_path);
    if ( hipSuccess != status )
    {    
      cuCtxDetach(cuContext);
      return status;
    }     

    // Global function
    status = hipModuleGetFunction( &cuFunction, cuModule, "kernel" );
    printf("function loading: %d\n", status);
    if ( hipSuccess != status)
    {    
      cuCtxDetach(cuContext);
      return status;
    }     

    *pKernel = cuFunction;

    return hipSuccess;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, char** argv)
{
    CUT_DEVICE_INIT(argc, argv);

    // Create GL context
    glutInit( &argc, argv);
    glutInitDisplayMode( GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize( window_width, window_height);
    glutCreateWindow( "Cuda GL interop");

    // initialize GL
    if( CUTFalse == initGL()) {
        return;
    }

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);

    // Cuda Driver Init
    kernel = NULL;
    CU_SAFE_CALL(initCuda(&kernel, argc, argv));


    // create VBO
    createVBO(vboId);
    printf ("INITIALIZE OF SIMPLEL\n");
    // run the cuda part
    runCuda(vboId, kernel);

    // start rendering mainloop
    glutMainLoop();

}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda( GLuint* vbo, hipFunction_t func)
{
    if(flagG==0) flagG = 1; 
    else flagG = 0;

    // map OpenGL buffer object for writing from CUDA
    hipDeviceptr_t dptr;
    unsigned int size;
    CU_SAFE_CALL(cuGLMapBufferObject( &dptr, &size, *vbo));

    //printf("runCuda (vbo, anim) = %d, %f\n", *vbo, anim);

    // execute the func
    int BLOCK_SIZE_X = 8;
    int BLOCK_SIZE_Y = 9;

    hipDeviceptr_t dim_dx;
    hipDeviceptr_t dim_dy;
    hipMalloc(&dim_dx, sizeof(float));
    hipMalloc(&dim_dy, sizeof(float));

    // setup execution parameters
    CU_SAFE_CALL(cuFuncSetBlockShape( func, BLOCK_SIZE_X, BLOCK_SIZE_Y, 1 ));
    CU_SAFE_CALL(cuFuncSetSharedSize( func,
	3*BLOCK_SIZE_X*BLOCK_SIZE_Y*sizeof(float) ) );

    CU_SAFE_CALL(cuParamSeti( func, 0,  dptr));
    CU_SAFE_CALL(cuParamSeti( func, 4,  BLOCK_SIZE_X ));
    CU_SAFE_CALL(cuParamSetf( func, 8,  0.5f));
    CU_SAFE_CALL(cuParamSeti( func, 12, flagG));
    CU_SAFE_CALL(cuParamSeti( func, 16, dim_dx));
    CU_SAFE_CALL(cuParamSeti( func, 20, dim_dy));
    CU_SAFE_CALL(cuParamSetSize( func, 24 ));
    CU_SAFE_CALL(cuLaunchGrid( func, 1, 1)); 

//;    dim3 block(8, 8, 1);
//;    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
//;    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, anim);

    // unmap buffer object
    CU_SAFE_CALL(cuGLUnmapBufferObject(*vbo));
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
CUTBoolean initGL()
{
    // initialize necessary OpenGL extensions

    glewInit();
    if (! glewIsSupported( "GL_VERSION_2_0 " 
        "GL_ARB_pixel_buffer_object"
		)) {
        fprintf( stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush( stderr);
        return CUTFalse;
    }

    // default initialization
    glClearColor( 0.8, 0.8, 0.8, 1.0);
    glDisable( GL_DEPTH_TEST);

    // viewport
    glViewport( 0, 0, window_width, window_height);

    // projection
    glMatrixMode( GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    CUT_CHECK_ERROR_GL();

    return CUTTrue;
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo)
{
    unsigned int size = mesh_width * mesh_height  * sizeof( float);

    // create buffer object
    glGenBuffers( 3, vbo);
printf("1: %d\n", 1);

    glBindBuffer( GL_ARRAY_BUFFER, *vbo);
    glBufferData( GL_ARRAY_BUFFER, size, vertices, GL_DYNAMIC_DRAW);
printf("2\n");

    glBindBuffer( GL_ARRAY_BUFFER, *(vbo+1));
    glBufferData( GL_ARRAY_BUFFER, size, normals, GL_DYNAMIC_DRAW);
printf("3\n");

    glBindBuffer( GL_ARRAY_BUFFER, *(vbo+2));
    glBufferData( GL_ARRAY_BUFFER, size, colors, GL_DYNAMIC_DRAW);
printf("4\n");

    glBindBuffer( GL_ARRAY_BUFFER, 0);
printf("5\n");
    // register buffer object with CUDA
    CU_SAFE_CALL(cuGLInit());
    CU_SAFE_CALL(cuGLRegisterBufferObject(*vbo));
printf("6\n");

    CUT_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO( GLuint* vbo)
{
    glBindBuffer( 1, *vbo);
    glDeleteBuffers( 1, vbo);

    CU_SAFE_CALL(cuGLUnregisterBufferObject(*vbo));

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    // run CUDA kernel to generate vertex positions
    runCuda(vboId, kernel);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, *(vboId+1));
    glNormalPointer(GL_FLOAT, 0, 0);

    glBindBuffer(GL_ARRAY_BUFFER, *(vboId+2));
    glColorPointer(3, GL_FLOAT, 0, 0);

    glBindBuffer(GL_ARRAY_BUFFER, *vboId);
    glVertexPointer(3, GL_FLOAT, 0, 0);

    glDrawArrays(GL_QUADS, 0, 24);

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);

    glutSwapBuffers();
    glutPostRedisplay();

    anim += 1.0;
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard( unsigned char key, int /*x*/, int /*y*/)
{
    switch( key) {
    case( 27) :
        deleteVBO( vboId);
        exit( 0);
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
        mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx, dy;
    dx = x - mouse_old_x;
    dy = y - mouse_old_y;

    if (mouse_buttons & 1) {
        rotate_x += dy * 0.2;
        rotate_y += dx * 0.2;
    } else if (mouse_buttons & 4) {
        translate_z += dy * 0.01;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}
