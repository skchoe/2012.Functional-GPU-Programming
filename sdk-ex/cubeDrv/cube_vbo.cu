
/* Copyright (c) Mark J. Kilgard, 1997. */

/* This program is freely distributable without licensing fees 
   and is provided without guarantee or warrantee expressed or 
   implied. This program is -not- in the public domain. */

/* This program was requested by Patrick Earl; hopefully someone else
   will write the equivalent Direct3D immediate mode program. */

/* Seungkeol Choe modify the code to support vbo, CUDA */

#define GL_GLEXT_PROTOTYPES

#include <assert.h>
#include <stdlib.h>
#include <stdio.h>
#include <GL/gl.h>
#include <GL/glut.h>
#include <GL/glext.h>

// includes
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_gl_error.h>
#include <cudaGL.h>

// cube ///////////////////////////////////////////////////////////////////////
//    v6----- v5
//   /|      /|
//  v1------v0|
//  | |     | |
//  | |v7---|-|v4
//  |/      |/
//  v2------v3

// vertex coords array
GLfloat vertices[] = {1,1,1,  -1,1,1,  -1,-1,1,  1,-1,1,        // v0-v1-v2-v3
                      1,1,1,  1,-1,1,  1,-1,-1,  1,1,-1,        // v0-v3-v4-v5
                      1,1,1,  1,1,-1,  -1,1,-1,  -1,1,1,        // v0-v5-v6-v1
                      -1,1,1,  -1,1,-1,  -1,-1,-1,  -1,-1,1,    // v1-v6-v7-v2
                      -1,-1,-1,  1,-1,-1,  1,-1,1,  -1,-1,1,    // v7-v4-v3-v2
                      1,-1,-1,  -1,-1,-1,  -1,1,-1,  1,1,-1
		      
		      };   // v4-v7-v6-v5

// normal array
GLfloat normals[] = {0,0,1,  0,0,1,  0,0,1,  0,0,1,             // v0-v1-v2-v3
		     1,0,0,  1,0,0,  1,0,0, 1,0,0,              // v0-v3-v4-v5
                     0,1,0,  0,1,0,  0,1,0, 0,1,0,              // v0-v5-v6-v1
                     -1,0,0,  -1,0,0, -1,0,0,  -1,0,0,          // v1-v6-v7-v2
                     0,-1,0,  0,-1,0,  0,-1,0,  0,-1,0,         // v7-v4-v3-v2
                     0,0,-1,  0,0,-1,  0,0,-1,  0,0,-1
		     
};        // v4-v7-v6-v5

// color array
GLfloat colors[] = {1,1,1,  1,1,0,  1,0,0,  1,0,1,              // v0-v1-v2-v3
                    1,1,1,  1,0,1,  0,0,1,  0,1,1,              // v0-v3-v4-v5
                    1,1,1,  0,1,1,  0,1,0,  1,1,0,              // v0-v5-v6-v1
                    1,1,0,  0,1,0,  0,0,0,  1,0,0,              // v1-v6-v7-v2
                    0,0,0,  0,0,1,  1,0,1,  1,0,0,              // v7-v4-v3-v2
                    0,0,1,  0,0,0,  0,1,0,  0,1,1               // v4-v7-v6-v5
		    };             


GLuint vboId = 0;

// prototypes
void display(void);
void reshape(int w, int h);
void init(void);
void initVBO(void);

hipFunction_t kernel;
hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;

int flagG = 1;

void runCuda( GLuint vbo, hipFunction_t func)
{

    // flag control
    float var;
    if(flagG==0){ var = 0.5f; flagG=1;}
    else{ var = 2.0f;  flagG=0;}

    // map OpenGL buffer object for writing from CUDA
    hipDeviceptr_t dptr;
    unsigned int size;
    CU_SAFE_CALL(cuGLMapBufferObject( &dptr, &size, vbo));

    printf("runCuda (vbo, anim) = %d\n", vbo);

    // execute the func
    int BLOCK_SIZE_X = 8;
    int BLOCK_SIZE_Y = 9;

    // idx vars
    hipDeviceptr_t dim_dx;
    hipDeviceptr_t dim_dy;

    // random gen.
    float rd = (float)((double)rand()/((double)(RAND_MAX)+(double)(1)));
    printf ("random no = %f\n", rd);


    CU_SAFE_CALL(hipMalloc(&dim_dx, (BLOCK_SIZE_X * sizeof(int))));
    CU_SAFE_CALL(hipMalloc(&dim_dy, (BLOCK_SIZE_Y * sizeof(int))));

    // setup execution parameters
    CU_SAFE_CALL(cuFuncSetBlockShape( func, BLOCK_SIZE_X, BLOCK_SIZE_Y, 1 ));
    CU_SAFE_CALL(cuParamSeti( func, 0,  dptr));
    CU_SAFE_CALL(cuParamSeti( func, 4,  BLOCK_SIZE_X));
    CU_SAFE_CALL(cuParamSetf( func, 8,  rd));
    CU_SAFE_CALL(cuParamSetf( func, 12, var));
    CU_SAFE_CALL(cuParamSeti( func, 16, dim_dx));
    CU_SAFE_CALL(cuParamSeti( func, 20, dim_dy));
    CU_SAFE_CALL(cuParamSetSize( func, 24 ));

//    CU_SAFE_CALL(cuFuncSetSharedSize( func,
//        3*BLOCK_SIZE_X*BLOCK_SIZE_Y*sizeof(float) ) );

    CU_SAFE_CALL(cuLaunchGrid( func, 1, 1));

    int* dim_hx = (int*)malloc(BLOCK_SIZE_X * sizeof(int));
    int* dim_hy = (int*)malloc(BLOCK_SIZE_Y * sizeof(int));
    CU_SAFE_CALL(hipMemcpyDtoH((void *)dim_hx, dim_dx, (BLOCK_SIZE_X) * sizeof(int)));
    CU_SAFE_CALL(hipMemcpyDtoH(dim_hy, dim_dy, (BLOCK_SIZE_Y) * sizeof(int)));

    int j;
    for(j=0;j < BLOCK_SIZE_X; j++)
      printf ("Index X = %d\t", *(dim_hx + j));
    printf ("\n");
    for(j=0;j < BLOCK_SIZE_Y; j++)
      printf ("Index Y = %d\t", *(dim_hy + j));
    printf ("\n");


    // unmap buffer object
    CU_SAFE_CALL(cuGLUnmapBufferObject(vbo));
}

void
display(void)
{
  runCuda(vboId, kernel);

  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT);

  glPushMatrix();
  
  glTranslatef(-.5, -.5, -.5);

  // bind VBOs with IDs and set the buffer offsets of the bound VBOs
  // When buffer object is bound with its ID, all pointers in gl*Pointer()
  // are treated as offset instead of real pointer.
  //  glBindBufferARB(GL_ARRAY_BUFFER_ARB, vboId);
  glBindBuffer(GL_ARRAY_BUFFER_ARB, vboId);

  printf ("glBindBuffer -> vboId = %d\n", vboId);

  // enable vertex arrays
  glEnableClientState(GL_NORMAL_ARRAY);
  glEnableClientState(GL_COLOR_ARRAY);
  glEnableClientState(GL_VERTEX_ARRAY);
  
  // before draw, specify vertex and index arrays with their offsets
  glNormalPointer(GL_FLOAT, 0, (void*)sizeof(vertices));
  glColorPointer(3, GL_FLOAT, 0, (void*)(sizeof(vertices)+sizeof(normals)));
  glVertexPointer(3, GL_FLOAT, 0, 0);

  glDrawArrays(GL_QUADS, 0, 24);

  glDisableClientState(GL_VERTEX_ARRAY);  // disable vertex arrays
  glDisableClientState(GL_COLOR_ARRAY);
  glDisableClientState(GL_NORMAL_ARRAY);

  // it is good idea to release VBOs with ID 0 after use.
  // Once bound with 0, all pointers in gl*Pointer() behave as real
  // pointer, so, normal vertex array operations are re-activated
  glBindBuffer(GL_ARRAY_BUFFER_ARB, 0);

  glPopMatrix();

  glutSwapBuffers();
}
void 
reshape(int w, int h)
{
    // set viewport to be the entire window
    glViewport(0, 0, (GLsizei)w, (GLsizei)h);

    // set perspective viewing frustum
    // float aspectRatio = (float)w / h;
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0f, (float)(w)/h, 1.0f, 1000.0f); // FOV, AspectRatio, NearClip, FarClip

    // switch to modelview matrix in order to set scene
    glMatrixMode(GL_MODELVIEW);
}

void
init(void)
{
  glShadeModel(GL_SMOOTH);
  glEnable(GL_DEPTH_TEST);
  glEnable(GL_LIGHTING);

  glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
  glEnable(GL_COLOR_MATERIAL);

  glClearColor(0, 0, 0, 1);

  // set up light colors (ambient, diffuse, specular)
  GLfloat lightKa[] = {.2f, .2f, .2f, 1.0f};  // ambient light
  GLfloat lightKd[] = {.7f, .7f, .7f, 1.0f};  // diffuse light
  GLfloat lightKs[] = {1, 1, 1, 1};           // specular light
  glLightfv(GL_LIGHT0, GL_AMBIENT, lightKa);
  glLightfv(GL_LIGHT0, GL_DIFFUSE, lightKd);
  glLightfv(GL_LIGHT0, GL_SPECULAR, lightKs);

  // position the light
  float lightPos[4] = {0, 0, 20, 1}; // positional light
  glLightfv(GL_LIGHT0, GL_POSITION, lightPos);
  
  glEnable(GL_LIGHT0); // MUST enable each light source after configuration

  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();

  int posX=0, posY=0, posZ=5;
  int targetX=0, targetY=0, targetZ=0;
  int upX=0, upY=1, upZ=0;
  gluLookAt(posX, posY, posZ, targetX, targetY, targetZ, upX, upY, upZ);

}

void
initVBO(void)
{
  // create vertex buffer objects, you need to delete them when program exits
  // Try to put both vertex coords array, vertex normal array and vertex color 
  //   in the same buffer object.
  // glBufferDataARB with NULL pointer reserves only memory space.
  // Copy actual data with 2 calls of glBufferSubDataARB, one for vertex coords 
  //   and one for normals.
  // target flag is GL_ARRAY_BUFFER_ARB, and usage flag is GL_STATIC_DRAW_ARB
  //  glGenBuffersARB(1, &vboId);
  glGenBuffers(1, &vboId);

  printf ("glGenBuffers -> vboId = %d\n", vboId);

  glBindBuffer(GL_ARRAY_BUFFER_ARB, vboId);

  glBufferData(GL_ARRAY_BUFFER_ARB, sizeof(vertices)
                                   +sizeof(normals)   
                                   +sizeof(colors), 
		  0, GL_DYNAMIC_DRAW_ARB);

  // copy vertices starting from 0 offest  
  glBufferSubData(GL_ARRAY_BUFFER_ARB, 0, sizeof(vertices), vertices);

  // copy normals after vertices
  glBufferSubData(GL_ARRAY_BUFFER_ARB, sizeof(vertices), sizeof(normals), normals);

  // copy colours after normals
  glBufferSubData(GL_ARRAY_BUFFER_ARB, sizeof(vertices)+sizeof(normals), 
		     sizeof(colors), colors);

  glBindBuffer (GL_ARRAY_BUFFER_ARB, 0);


  // register buffer object with CUDA
  CU_SAFE_CALL(cuGLInit());
  CU_SAFE_CALL(cuGLRegisterBufferObject(vboId));

  CUT_CHECK_ERROR_GL();
}

static hipError_t
initCuda(hipFunction_t *pKernel,
        int argc, char** argv)
{
    hipFunction_t cuFunction = 0;

    CUT_DEVICE_INIT_DRV(cuDevice, argc, argv);

    hipError_t status = hipCtxCreate( &cuContext, 0, cuDevice );
    printf ("Ctx Creation: %d\n", status);
    if ( hipSuccess != status )
    {
      cuCtxDetach(cuContext);
      return status;
    }

    status = hipModuleLoad(&cuModule, "data/cube_kernel.cubin");
    printf ("ModuleLoad: %d\n", status);

    if ( hipSuccess != status )
    {
      cuCtxDetach(cuContext);
      return status;
    }

    // Global function
    status = hipModuleGetFunction( &cuFunction, cuModule, "kernel" );
    printf("function loading: %d\n", status);
    if ( hipSuccess != status)
    {
      cuCtxDetach(cuContext);
      return status;
    }

    *pKernel = cuFunction;

    return hipSuccess;
}



int
main(int argc, char **argv)
{
  CUT_DEVICE_INIT(argc, argv);

  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
  glutInitWindowSize(800, 600);
  glutInitWindowPosition(400, 300);
  glutCreateWindow("color cube by vertex buffer object");
  glutDisplayFunc(display);
  glutReshapeFunc(reshape);
  init();

  kernel = NULL;
  CU_SAFE_CALL(initCuda(&kernel, argc, argv));

  initVBO();

  int seed = 1000;
  srand(seed);

  runCuda(vboId, kernel);

  glutMainLoop();
  return 0;             /* ANSI C requires main to return int. */
}
