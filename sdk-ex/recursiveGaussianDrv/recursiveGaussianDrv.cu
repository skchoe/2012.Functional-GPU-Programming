#include "hip/hip_runtime.h"
/*
    Recursive Gaussian filter
    sgreen 8/1/08

    This code sample implements a Gaussian blur using Deriche's recursive method:
    http://citeseer.ist.psu.edu/deriche93recursively.html

    This is similar to the box filter sample in the SDK, but it uses the previous
    outputs of the filter as well as the previous inputs. This is also known as an
    IIR (infinite impulse response) filter, since its response to an input impulse
    can last forever.

    The main advantage of this method is that the execution time is independent of
    the filter width.
    
    The GPU processes columns of the image in parallel. To avoid uncoalesced reads
    for the row pass we transpose the image and then transpose it back again
    afterwards.

    The implementation is based on code from the CImg library:
    http://cimg.sourceforge.net/
    Thanks to David Tschumperl� and all the CImg contributors!
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <GL/glew.h>

#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <hip/hip_runtime.h>
#include <cudaGL.h>
#include <cutil.h>

#include <cutil_math.h>

#define BLOCK_DIM 16 

typedef unsigned int uint;
typedef unsigned char uchar;

char *image_filename = "lena.ppm";
float sigma = 10.0f;
int order = 0;
int nthreads = 64;

unsigned int width, height;
uint* h_img = NULL;
hipDeviceptr_t d_img;
hipDeviceptr_t d_temp;

GLuint pbo = 0;     // OpenGL pixel buffer object
GLuint texid = 0;   // texture

unsigned int timer = 0;
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling

hipFunction_t d_transpose;
hipFunction_t d_recursiveGaussian_rgba;

/*
    Transpose a 2D array (see SDK transpose example)
*/
void proc_transpose(hipDeviceptr_t d_dst, hipDeviceptr_t d_src, uint width, int height)
{
  //dim3 grid(width / BLOCK_DIM, height / BLOCK_DIM, 1);
  //dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
    CU_SAFE_CALL(cuFuncSetBlockShape(d_transpose, BLOCK_DIM, BLOCK_DIM, 1));
    CU_SAFE_CALL(cuParamSeti(d_transpose, 0, d_dst));
    CU_SAFE_CALL(cuParamSeti(d_transpose, 4, d_src));
    CU_SAFE_CALL(cuParamSeti(d_transpose, 8, width));
    CU_SAFE_CALL(cuParamSeti(d_transpose, 12,height));
    CU_SAFE_CALL(cuParamSetSize(d_transpose, 16));
    CU_SAFE_CALL(cuLaunchGrid(d_transpose, width / BLOCK_DIM, height / BLOCK_DIM));
    
    //d_transpose<<< grid, threads >>>(d_dest, d_src, width, height);
    CUT_CHECK_ERROR("Kernel execution failed");
}


/*
    Perform Gaussian filter on a 2D image using CUDA

    Parameters:
    d_src  - pointer to input image in device memory
    d_dest - pointer to destination image in device memory
    d_temp - pointer to temporary storage in device memory
    width  - image width
    height - image height
    sigma  - sigma of Gaussian
    order  - filter order (0, 1 or 2)
*/

void proc_gaussianFilter_rgba(hipDeviceptr_t d_src, hipDeviceptr_t d_dst, 
			      int length, int nthreads, 
			      float a0, float a1, float a2, float a3, 
			      float b1, float b2, float coefp, float coefn)
{
  int ngrid = length / nthreads;
  CU_SAFE_CALL(cuFuncSetBlockShape(d_recursiveGaussian_rgba, nthreads, 1, 1));
  CU_SAFE_CALL(cuParamSeti(d_recursiveGaussian_rgba, 0, d_src));
  CU_SAFE_CALL(cuParamSeti(d_recursiveGaussian_rgba, 4, d_dst));
  CU_SAFE_CALL(cuParamSeti(d_recursiveGaussian_rgba, 8, length));
  CU_SAFE_CALL(cuParamSeti(d_recursiveGaussian_rgba, 12, length));
  CU_SAFE_CALL(cuParamSetf(d_recursiveGaussian_rgba, 16, a0));
  CU_SAFE_CALL(cuParamSetf(d_recursiveGaussian_rgba, 20, a1));
  CU_SAFE_CALL(cuParamSetf(d_recursiveGaussian_rgba, 24, a2));
  CU_SAFE_CALL(cuParamSetf(d_recursiveGaussian_rgba, 28, a3));
  CU_SAFE_CALL(cuParamSetf(d_recursiveGaussian_rgba, 32, b1));
  CU_SAFE_CALL(cuParamSetf(d_recursiveGaussian_rgba, 36, b2));
  CU_SAFE_CALL(cuParamSetf(d_recursiveGaussian_rgba, 40, coefp));
  CU_SAFE_CALL(cuParamSetf(d_recursiveGaussian_rgba, 44, coefn));
  CU_SAFE_CALL(cuParamSetSize(d_recursiveGaussian_rgba, 48));
  CU_SAFE_CALL(cuLaunchGrid(d_recursiveGaussian_rgba, ngrid, 1));
}


// 8-bit RGBA version
void gaussianFilterRGBA(hipDeviceptr_t d_src, hipDeviceptr_t *pd_dest, hipDeviceptr_t d_temp, 
int width, int height, float sigma, int order)
{
    const float
        nsigma = sigma < 0.1f ? 0.1f : sigma,
        alpha = 1.695f / nsigma,
        ema = (float)std::exp(-alpha),
        ema2 = (float)std::exp(-2*alpha),
        b1 = -2*ema,
        b2 = ema2;

    float a0 = 0, a1 = 0, a2 = 0, a3 = 0, coefp = 0, coefn = 0;
    switch (order) {
    case 0: {
        const float k = (1-ema)*(1-ema)/(1+2*alpha*ema-ema2);
        a0 = k;
        a1 = k*(alpha-1)*ema;
        a2 = k*(alpha+1)*ema;
        a3 = -k*ema2;
    } break;

    case 1: {
        const float k = (1-ema)*(1-ema)/ema;
        a0 = k*ema;
        a1 = a3 = 0;
        a2 = -a0;
    } break;

    case 2: {
        const float
            ea = (float)std::exp(-alpha),
            k = -(ema2-1)/(2*alpha*ema),
            kn = (-2*(-1+3*ea-3*ea*ea+ea*ea*ea)/(3*ea+1+3*ea*ea+ea*ea*ea));
        a0 = kn;
        a1 = -kn*(1+k*alpha)*ema;
        a2 = kn*(1-k*alpha)*ema;
        a3 = -kn*ema2;
    } break;

    default:
        fprintf(stderr, "gaussianFilter: invalid order parameter!\n");
        return;
    }
    coefp = (a0+a1)/(1+b1+b2);
    coefn = (a2+a3)/(1+b1+b2);

    // process columns
//    d_simpleRecursive_rgba<<< width / nthreads, nthreads >>>(d_src, d_temp, width, height, sigma);

    proc_gaussianFilter_rgba(d_src, d_temp, width, nthreads, a0, a1, a2, a3, b1, b2, coefp, coefn);
    CUT_CHECK_ERROR("Kernel execution failed");

    proc_transpose(*pd_dest, d_temp, width, height);

    // process rows
//    d_simpleRecursive_rgba<<< width / nthreads, nthreads >>>(d_dest, d_temp, width, height, sigma);
    proc_gaussianFilter_rgba(*pd_dest, d_temp, height, nthreads, a0, a1, a2, a3, b1, b2, coefp, coefn);
    CUT_CHECK_ERROR("Kernel execution failed");

    proc_transpose(*pd_dest, d_temp, height, width);
}

// display results using OpenGL
void display()
{
    CUT_SAFE_CALL(cutStartTimer(timer));  

    // execute filter, writing results to pbo
    hipDeviceptr_t d_result;
    size_t size;
    CU_SAFE_CALL(cuGLMapBufferObject(&d_result, &size, pbo));
    //printf("**BO mapping done w/ %d, %d\n", d_result, size);
    gaussianFilterRGBA(d_img, &d_result, d_temp, width, height, sigma, order);
    CU_SAFE_CALL(cuGLUnmapBufferObject(pbo));
    //printf("**BO unmapping done\n");

    // load texture from pbo
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBindTexture(GL_TEXTURE_2D, texid);
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    // display results
    glClear(GL_COLOR_BUFFER_BIT);

    glEnable(GL_TEXTURE_2D);
    glDisable(GL_DEPTH_TEST);

    glBegin(GL_QUADS);
    glVertex2f(0, 0); glTexCoord2f(0, 0);
    glVertex2f(0, 1); glTexCoord2f(1, 0);
    glVertex2f(1, 1); glTexCoord2f(1, 1);
    glVertex2f(1, 0); glTexCoord2f(0, 1);
    glEnd();

    glDisable(GL_TEXTURE_2D);

    glutSwapBuffers();

    CUT_SAFE_CALL(cutStopTimer(timer));  

    fpsCount++;
    if (fpsCount == fpsLimit) {
        char fps[256];
        float ifps = 1.f / (cutGetAverageTimerValue(timer) / 1000.f);
        sprintf(fps, "CUDA Recursive Gaussian filter: %3.1f fps", ifps);
        glutSetWindowTitle(fps);
        fpsCount = 0; 
        fpsLimit = (int)max(ifps, 1.f);
        CUT_SAFE_CALL(cutResetTimer(timer));  
    }
}

void idle()
{
    glutPostRedisplay();
}

void keyboard(unsigned char key, int x, int y)
{
    switch(key) {
        case 27:
            exit(0);
            break;
        case '=':
        case '+':
            sigma+=0.1f;
            break;
        case '-':
            sigma-=0.1f;
            if (sigma < 0.0) sigma = 0.0f;
            break;
        case '0':
            order = 0;
            break;
        case '1':
            order = 1;
            sigma = 0.5f;
            break;
        case '2':
            order = 2;
            sigma = 0.5f;
            break;

        default:
            break;
    }
    printf("sigma = %f\n", sigma);
    glutPostRedisplay();
}

void reshape(int x, int y)
{
    glViewport(0, 0, x, y);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0); 
}

void cleanup()
{
    CUT_SAFE_CALL( cutDeleteTimer( timer));
    free(h_img);

    CU_SAFE_CALL(hipFree(d_img));
    CU_SAFE_CALL(hipFree(d_temp));

    if (pbo) {
        CU_SAFE_CALL(cuGLUnregisterBufferObject(pbo));    
        glDeleteBuffersARB(1, &pbo);
    }
    if (texid) {
        glDeleteTextures(1, &texid);
    }
}

void initOpenGL()
{
    // create pixel buffer object to store final image
    glGenBuffersARB(1, &pbo);
    printf("pixel buffer id = %d\n", pbo);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, width*height*sizeof(GLubyte)*4, h_img, GL_STREAM_DRAW_ARB);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    CU_SAFE_CALL(cuGLInit());
    CU_SAFE_CALL(cuGLRegisterBufferObject(pbo));

    // create texture for display
    glGenTextures(1, &texid);
    printf("tex-id  = %d\n", texid);
    glBindTexture(GL_TEXTURE_2D, texid);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_2D, 0);
}

void
benchmark(int iterations) 
{
    // allocate memory for result
    hipDeviceptr_t d_result;
    unsigned int size = width * height * sizeof(uint);
    printf("---begin benchmark\n");
    CU_SAFE_CALL(hipMalloc(&d_result, size));

    // warm-up
    gaussianFilterRGBA(d_img, &d_result, d_temp, width, height, sigma, order);

    CU_SAFE_CALL( hipCtxSynchronize() );
    CUT_SAFE_CALL( cutStartTimer( timer));

    // execute the kernel
    for(int i=0; i<iterations; i++) {
        gaussianFilterRGBA(d_img, &d_result, d_temp, width, height, sigma, order);
    }

    CU_SAFE_CALL( hipCtxSynchronize() );
    CUT_SAFE_CALL( cutStopTimer( timer));

    // check if kernel execution generated an error
    CUT_CHECK_ERROR("Kernel execution failed");

    printf("Processing time: %f (ms)\n", cutGetTimerValue( timer));
    printf("%.2f Mpixels/sec\n", (width*height*iterations / (cutGetTimerValue( timer) / 1000.0f)) / 1e6);

    CU_SAFE_CALL(hipFree(d_result));
    printf("--end benchmark\n");
}


////////////////////////////////////////////////////////////////////////////////
//! Init for CUDA
////////////////////////////////////////////////////////////////////////////////
//    CU_SAFE_CALL(initCudaContext(&cuDevice, &cuContext, argc, (char**)argv));
static hipError_t
initCudaContext(hipCtx_t *pcuContext, hipDevice_t *pcuDevice,
        int argc, char** argv)
{
  //hipInit() is called by CUT_DEVICE_INIT_DRV
    CUT_DEVICE_INIT_DRV(*pcuDevice, argc, argv);

    hipError_t status = hipCtxCreate( pcuContext, 0, *pcuDevice );
    printf ("Ctx Creation: %d\n", status);
    if ( hipSuccess != status )
    {
      cuCtxDetach(*pcuContext);
      return status;
    }
    return hipSuccess;
}

static hipError_t
loadModule(hipModule_t* pcuModule, hipCtx_t* pcuContext, char* cubinName)
{
    hipError_t status = hipModuleLoad(pcuModule, cubinName);
    printf ("ModuleLoad: %s: %d\n", cubinName, status);

    if ( hipSuccess != status )
    {
      cuCtxDetach(*pcuContext);
      return status;
    }
    return hipSuccess;
}

static hipError_t
loadKernel(hipFunction_t* pcuFunction, hipModule_t cuModule, char* kernelName)
{
    // Global function
    hipFunction_t cuFunction;
    hipError_t status = hipModuleGetFunction( &cuFunction, cuModule, kernelName);
    printf("function loading: %d\n", status);
    if ( hipSuccess != status)
    {
      *pcuFunction = NULL;
      return status;
    }

    *pcuFunction = cuFunction;

    return hipSuccess;
}

CUTBoolean initGL()
{
    // initialize necessary OpenGL extensions
    glewInit();
    if (! glewIsSupported( "GL_VERSION_2_0 " 
			   "GL_ARB_pixel_buffer_object")){
        fprintf( stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush( stderr);
        return CUTFalse;
    }
    return CUTTrue;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{

    CUT_DEVICE_INIT(argc, argv);

    char *filename;
    if (cutGetCmdLineArgumentstr(argc, (const char**) argv, "image", &filename)) {
        image_filename = filename;
    }
    cutGetCmdLineArgumenti(argc, (const char**) argv, "threads", &nthreads);
    cutGetCmdLineArgumentf(argc, (const char**) argv, "sigma", &sigma);
    CUTBoolean runBenchmark = cutCheckCmdLineFlag(argc, (const char**) argv, "bench");

    // load image from disk
    char* image_path = cutFindFilePath(image_filename, argv[0]);
    if (image_path == 0) {
        fprintf(stderr, "Error finding image file '%s'\n", image_filename);
        exit(EXIT_FAILURE);
    }

    CUT_SAFE_CALL( cutLoadPPM4ub(image_path, (unsigned char **) &h_img, &width, &height));
    if (!h_img) {
        printf("Error opening file '%s'\n", image_path);
        exit(-1);
    }
    printf("Loaded '%s', %d x %d pixels\n", image_path, width, height);

    // initialize cuda, module, kernel
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    initCudaContext(&cuContext, &cuDevice, argc, (char**)argv);
    loadModule(&cuModule, &cuContext, "data/recursiveGaussian_kernel.sm_10.cubin");
    loadKernel(&d_transpose, cuModule, "d_transpose");
    loadKernel(&d_recursiveGaussian_rgba, cuModule, "d_recursiveGaussian_rgba");
    printf("context Creation\n moduleLoading\ncalling kernel successful\n");

    // allocate device memory
    unsigned int size = width * height * sizeof(uint);
    /* runtime API
    CUDA_SAFE_CALL(hipMalloc( (void**) &d_img, size));
    CUDA_SAFE_CALL(hipMalloc( (void**) &d_temp, size));
    CUDA_SAFE_CALL(hipMemcpy( d_img, h_img, size, hipMemcpyHostToDevice));
    */
    // driver API
    CU_SAFE_CALL(hipMalloc(&d_img, size));
    CU_SAFE_CALL(hipMalloc(&d_temp, size));
    CU_SAFE_CALL(hipMemcpyHtoD(d_img, h_img, size));

    CUT_SAFE_CALL( cutCreateTimer( &timer));

    if (runBenchmark) {
        benchmark(100);
        cleanup();
        exit(0);
    }

    // initialize GLUT
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
    glutInitWindowSize(width, height);
    glutCreateWindow("CUDA Recursive Gaussian filter");

    // initialize GL
    if( CUTFalse == initGL() ) return 0;

    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutReshapeFunc(reshape);
    glutIdleFunc(idle);

    printf("Press '+' and '-' to change filter width\n");
    printf("0, 1, 2 - change filter order\n");

    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object")) {
        fprintf(stderr, "Required OpenGL extensions missing.");
        exit(-1);
    }
    else {printf("GLEW init detect Opengl 2.0 supported\n");}

    initOpenGL();

    atexit(cleanup);

    glutMainLoop();
    return 0;
}
