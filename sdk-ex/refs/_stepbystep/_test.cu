#include <GL/glew.h>
#include <GL/glut.h>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cudaGL.h>
#include <stdio.h>
#include <stdlib.h>

hipFunction_t kernel;
hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;

void display()
{
 glClearColor( 0.0, 0.0, 0.8, 1.0);
 glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
 glutSwapBuffers();
}


////////////////////////////////////////////////////////////////////////////////
//! Init for CUDA
////////////////////////////////////////////////////////////////////////////////
static hipError_t
initCuda(hipFunction_t *pKernel, 
	int argc, char** argv)
{
    hipFunction_t cuFunction = 0;

    CUT_DEVICE_INIT_DRV(cuDevice);

    hipError_t status = hipCtxCreate( &cuContext, 0, cuDevice );
    printf ("Ctx Creation: %d\n", status);
    if ( hipSuccess != status )
    {    
      cuCtxDetach(cuContext);
      return status;
    }

    status = hipModuleLoad(&cuModule, "data/_test_kernel.cubin");
    printf ("ModuleLoad: %d\n", status);

    if ( hipSuccess != status )
    {    
      cuCtxDetach(cuContext);
      return status;
    }     

    // Global function
    status = hipModuleGetFunction( &cuFunction, cuModule, "kernel" );
    printf("function loading: %d\n", status);
    if ( hipSuccess != status)
    {    
      cuCtxDetach(cuContext);
      return status;
    }     

    *pKernel = cuFunction;

    return hipSuccess;
}

//----------------------------------------------------------------------
int main(int argc, char** argv)
{
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
  glutInitWindowSize(512, 512);
  glutCreateWindow("BasicGL");

  glutDisplayFunc(display);
            
  kernel=NULL;
  initCuda(&kernel, argc, argv); 

  glewInit();
printf("cuGLInit()\t");
  CU_SAFE_CALL(cuGLInit());
printf("\n");
                                
  int sz = 20000;
  float* ptr = new float[sz];
  unsigned int size = sz*sizeof(float);
                                                              
  unsigned int vbo;
  glGenBuffers(1, &vbo);
  printf("VbO = %d\n", vbo);

  glBindBuffer(GL_ARRAY_BUFFER, vbo);
  glBufferData(GL_ARRAY_BUFFER,  size, (const void*) ptr, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);
                                                              
printf("CUT check error gl\t");
  CUT_CHECK_ERROR_GL();
printf("\n");
							   
  //glutPostRedisplay();
                                                              
  hipDeviceptr_t devPtr;
  unsigned int ssize;
                                                              
printf("registerbufferobject\t");
  CU_SAFE_CALL(cuGLRegisterBufferObject(vbo));
printf("\n");
printf("glmapbufferobject\t");
  CU_SAFE_CALL(cuGLMapBufferObject(&devPtr, &ssize, vbo));
printf("\n");
                                                              
  // <<<<   PAY ATTENTION ....
  CU_SAFE_CALL(cuGLMapBufferObject(&devPtr, &ssize, vbo));
  printf("- size= %d, ssize= %d, devPtr= %d\n", size, ssize, devPtr);
                                                              
  CU_SAFE_CALL(cuGLMapBufferObject(&ssize, &devPtr, vbo));
  printf("- size= %d, ssize= %d, devPtr= %d\n", size, ssize, devPtr);
                                    
  glutMainLoop();
  
}

