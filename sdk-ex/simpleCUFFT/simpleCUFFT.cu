#include "hip/hip_runtime.h"
/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hipfft/hipfft.h>
#include <cutil.h>

// Complex data type
typedef float2 Complex; 
extern "C"
extern __global__ void ComplexPointwiseMulAndScale(Complex*, const Complex*, int, float);



// Filtering functions
extern "C"
void Convolve(const Complex*, int, const Complex*, int, Complex*);

// Padding functions
int PadData(const Complex*, Complex**, int,
            const Complex*, Complex**, int);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

// The filter size is assumed to be a number smaller than the signal size
#define SIGNAL_SIZE        50
#define FILTER_KERNEL_SIZE 11

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
    runTest(argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv) 
{
    CUT_DEVICE_INIT(argc, argv);

    // Allocate host memory for the signal
    Complex* h_signal = (Complex*)malloc(sizeof(Complex) * SIGNAL_SIZE);
    // Initalize the memory for the signal
    for (unsigned int i = 0; i < SIGNAL_SIZE; ++i) {
        h_signal[i].x = rand() / (float)RAND_MAX;
        h_signal[i].y = 0;
    }

    // Allocate host memory for the filter
    Complex* h_filter_kernel = (Complex*)malloc(sizeof(Complex) * FILTER_KERNEL_SIZE);
    // Initalize the memory for the filter
    for (unsigned int i = 0; i < FILTER_KERNEL_SIZE; ++i) {
        h_filter_kernel[i].x = rand() / (float)RAND_MAX;
        h_filter_kernel[i].y = 0;
    }

    // Pad signal and filter kernel
    Complex* h_padded_signal;
    Complex* h_padded_filter_kernel;
    int new_size = PadData(h_signal, &h_padded_signal, SIGNAL_SIZE,
                           h_filter_kernel, &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
    int mem_size = sizeof(Complex) * new_size;

    // Allocate device memory for signal
    Complex* d_signal;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_signal, mem_size));
    // Copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(d_signal, h_padded_signal, mem_size,
                              hipMemcpyHostToDevice));

    // Allocate device memory for filter kernel
    Complex* d_filter_kernel;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_filter_kernel, mem_size));

    // Copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
                              hipMemcpyHostToDevice));

    // CUFFT plan
    hipfftHandle plan;
    CUFFT_SAFE_CALL(hipfftPlan1d(&plan, new_size, HIPFFT_C2C, 1));

    // Transform signal and kernel
    CUFFT_SAFE_CALL(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD));
    CUFFT_SAFE_CALL(hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernel, (hipfftComplex *)d_filter_kernel, HIPFFT_FORWARD));

    // Multiply the coefficients together and normalize the result
    ComplexPointwiseMulAndScale<<<32, 256>>>(d_signal, d_filter_kernel, new_size, 1.0f / new_size);

    // Check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

    // Transform signal back
    CUFFT_SAFE_CALL(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD));

    // Copy device memory to host
    Complex* h_convolved_signal = h_padded_signal;
    CUDA_SAFE_CALL(hipMemcpy(h_convolved_signal, d_signal, mem_size,
                              hipMemcpyDeviceToHost));

    // Allocate host memory for the convolution result
    Complex* h_convolved_signal_ref = (Complex*)malloc(sizeof(Complex) * SIGNAL_SIZE);

    // Convolve on the host
    Convolve(h_signal, SIGNAL_SIZE,
             h_filter_kernel, FILTER_KERNEL_SIZE,
             h_convolved_signal_ref);

    // check result
    CUTBoolean res = cutCompareL2fe((float*)h_convolved_signal_ref, (float*)h_convolved_signal, 2 * SIGNAL_SIZE, 1e-5f);
    printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

    //Destroy CUFFT context
    CUFFT_SAFE_CALL(hipfftDestroy(plan));

    // cleanup memory
    free(h_signal);
    free(h_filter_kernel);
    free(h_padded_signal);
    free(h_padded_filter_kernel);
    free(h_convolved_signal_ref);
    CUDA_SAFE_CALL(hipFree(d_signal));
    CUDA_SAFE_CALL(hipFree(d_filter_kernel));
}

// Pad data
int PadData(const Complex* signal, Complex** padded_signal, int signal_size,
            const Complex* filter_kernel, Complex** padded_filter_kernel, int filter_kernel_size)
{
    int minRadius = filter_kernel_size / 2;
    int maxRadius = filter_kernel_size - minRadius;
    int new_size = signal_size + maxRadius;
    
    // Pad signal
    Complex* new_data = (Complex*)malloc(sizeof(Complex) * new_size);
    memcpy(new_data +           0, signal,              signal_size * sizeof(Complex));
    memset(new_data + signal_size,      0, (new_size - signal_size) * sizeof(Complex));
    *padded_signal = new_data;
    
    // Pad filter
    new_data = (Complex*)malloc(sizeof(Complex) * new_size);  
    memcpy(new_data +                    0, filter_kernel + minRadius,                       maxRadius * sizeof(Complex));
    memset(new_data +            maxRadius,                         0, (new_size - filter_kernel_size) * sizeof(Complex));   
    memcpy(new_data + new_size - minRadius,             filter_kernel,                       minRadius * sizeof(Complex));
    *padded_filter_kernel = new_data;
    
    return new_size;
}

