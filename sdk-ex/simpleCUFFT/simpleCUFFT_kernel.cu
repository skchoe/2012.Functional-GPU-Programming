/* Example showing the use of CUFFT for fast 1D-convolution using FFT. -KERNEL part separated from original source*/


#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

// Complex data type
typedef float2 Complex; 
static __device__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);

extern "C"
// Complex pointwise multiplication
// Note the static function cannot be applicable to extern "C"
/*static*/ __global__ void ComplexPointwiseMulAndScale(Complex* a, const Complex* b, int size, float scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
        a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);     
}

// The filter size is assumed to be a number smaller than the signal size
#define SIGNAL_SIZE        50
#define FILTER_KERNEL_SIZE 11

// Complex scale
static __device__ inline Complex ComplexScale(Complex a, float s)
{
    Complex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}
