#include "hip/hip_runtime.h"
/* 
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, GL
#include <GL/glew.h>
#include <GL/glut.h>

// includes
#include <cutil.h>
#include <cutil_gl_error.h>
#include <cudaGL.h>


////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// vbo variables
GLuint vbo;

float anim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

////////////////////////////////////////////////////////////////////////////////
// kernels
#include <simpleGL_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// GL functionality
CUTBoolean initGL();
void createVBO( GLuint* vbo);
void deleteVBO( GLuint* vbo);

// rendering callbacks
void display();
void keyboard( unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

// Cuda functionality
void runCuda( GLuint vbo);
void checkResultCuda( int argc, char** argv, const GLuint& vbo);


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, char** argv)
{

    CUT_DEVICE_INIT();

    // Create GL context
    glutInit( &argc, argv);
    glutInitDisplayMode( GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize( window_width, window_height);
    glutCreateWindow( "Cuda GL interop");

    // initialize GL
    if( CUTFalse == initGL()) {
        return;
    }

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);

    // create VBO
    createVBO(&vbo);
printf ("INITIALIZE OF SIMPLEL\n");
    // run the cuda part
    //runCuda(vbo);

    // check result of Cuda step
    checkResultCuda(argc, argv, vbo);

    // start rendering mainloop
    glutMainLoop();
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda( GLuint vbo)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    int size;
    cuGLMapBufferObject( (void**)&dptr, &size, vbo));

    printf("runCuda (vbo, anim) = %d, %f\n", vbo, anim);

    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, anim);

    // unmap buffer object
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( vbo));
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
CUTBoolean initGL()
{
    // initialize necessary OpenGL extensions
    glewInit();
    if (! glewIsSupported( "GL_VERSION_2_0 " 
        "GL_ARB_pixel_buffer_object"
		)) {
        fprintf( stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush( stderr);
        return CUTFalse;
    }

    // default initialization
    glClearColor( 0.0, 1.0, 1.0, 1.0);
    glDisable( GL_DEPTH_TEST);

    // viewport
    glViewport( 0, 0, window_width, window_height);

    // projection
    glMatrixMode( GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    CUT_CHECK_ERROR_GL();

    return CUTTrue;
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo)
{
    // create buffer object
    glGenBuffers( 1, vbo);
    glBindBuffer( GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof( float);
    glBufferData( GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer( GL_ARRAY_BUFFER, 0);

/*
    // register buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(*vbo));

    CUT_CHECK_ERROR_GL();
*/
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO( GLuint* vbo)
{
    glBindBuffer( 1, *vbo);
    glDeleteBuffers( 1, vbo);

    CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(*vbo));

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    // run CUDA kernel to generate vertex positions
    //runCuda(vbo);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();
    glutPostRedisplay();

    anim += 1.0;
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard( unsigned char key, int /*x*/, int /*y*/)
{
    switch( key) {
    case( 27) :
        deleteVBO( &vbo);
        exit( 0);
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
        mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx, dy;
    dx = x - mouse_old_x;
    dy = y - mouse_old_y;

    if (mouse_buttons & 1) {
        rotate_x += dy * 0.2;
        rotate_y += dx * 0.2;
    } else if (mouse_buttons & 4) {
        translate_z += dy * 0.01;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda( int argc, char** argv, const GLuint& vbo)
{
    printf ("check Result cuda called\n");
/*
    CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(vbo));

    // map buffer object
    glBindBuffer( GL_ARRAY_BUFFER_ARB, vbo );
    float* data = (float*) glMapBuffer( GL_ARRAY_BUFFER, GL_READ_ONLY);

    // check result
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) {
        // write file for regression test
        CUT_SAFE_CALL( cutWriteFilef( "./data/regression.dat",
            data, mesh_width * mesh_height * 3, 0.0));
    }

    // unmap GL buffer object
    if( ! glUnmapBuffer( GL_ARRAY_BUFFER)) {
        fprintf( stderr, "Unmap buffer failed.\n");
        fflush( stderr);
    }

    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(vbo));

    CUT_CHECK_ERROR_GL();
*/
}
