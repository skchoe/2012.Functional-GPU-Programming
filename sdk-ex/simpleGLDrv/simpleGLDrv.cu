/* 
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, GL
#include <GL/glew.h>
#include <GL/glut.h>

// includes
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_gl_error.h>
#include <cudaGL.h>


////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// vbo variables
GLuint vbo;

float anim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

// GL functionality
CUTBoolean initGL();
void createVBO(GLuint* vbo);
void deleteVBO(GLuint* vbo);

// rendering callbacks
void display();
void keyboard( unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

// Cuda functionality
void runCuda(GLuint vbo, hipFunction_t knl);
void checkResultCuda( int argc, char** argv, const GLuint& vbo);

hipFunction_t kernel;
hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Init for CUDA
////////////////////////////////////////////////////////////////////////////////
static hipError_t
initCuda_getKernel(hipFunction_t *pKernel, 
	int argc, char** argv)
{
    hipFunction_t cuFunction = 0;

    CUT_DEVICE_INIT_DRV(cuDevice, argc, argv);

    hipError_t status = hipCtxCreate( &cuContext, 0, cuDevice );
    printf ("Ctx Creation: %d\n", status);
    if ( hipSuccess != status )
    {    
      cuCtxDetach(cuContext);
      return status;
    }

    status = hipModuleLoad(&cuModule, "data/simpleGLDrv_kernel.sm_10.cubin");
    printf ("ModuleLoad: %d\n", status);

//    cutFree(module_path);
    if ( hipSuccess != status )
    {    
      cuCtxDetach(cuContext);
      return status;
    }     

    // Global function
    status = hipModuleGetFunction( &cuFunction, cuModule, "kernel" );
    printf("function loading: %d\n", status);
    if ( hipSuccess != status)
    {    
      cuCtxDetach(cuContext);
      return status;
    }     

    *pKernel = cuFunction;

    return hipSuccess;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, char** argv)
{
    CUT_DEVICE_INIT(argc, argv);

    // Create GL context
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL interop");

    // initialize GL
    if( CUTFalse == initGL()) return;

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);

    // Cuda Driver Init
    kernel = NULL;
    CU_SAFE_CALL(initCuda_getKernel(&kernel, argc, argv));

    // create VBO
    createVBO(&vbo);
    printf ("INITIALIZE OF SIMPLEL\n");
    // run the cuda part
    runCuda(vbo, kernel);

    // check result of Cuda step
    checkResultCuda(argc, argv, vbo);

    // start rendering mainloop
    glutMainLoop();
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda( GLuint vbo, hipFunction_t func)
{
    // map OpenGL buffer object for writing from CUDA
    hipDeviceptr_t dptr;
    unsigned int size;
    CU_SAFE_CALL(cuGLMapBufferObject( &dptr, &size, vbo));

    //printf("runCuda (vbo, anim) = %d, %f\n", vbo, anim);

    // execute the func
    int BLOCK_SIZE = 8;
    // setup execution parameters
    CU_SAFE_CALL(cuFuncSetBlockShape( func, BLOCK_SIZE, BLOCK_SIZE, 1 ));
    CU_SAFE_CALL(cuFuncSetSharedSize( func,
	4*BLOCK_SIZE*BLOCK_SIZE*sizeof(float) ) );
    CU_SAFE_CALL(cuParamSeti( func, 0,  dptr));
    CU_SAFE_CALL(cuParamSeti( func, 4,  mesh_width ));
    CU_SAFE_CALL(cuParamSeti( func, 8,  mesh_height));
    CU_SAFE_CALL(cuParamSetf( func, 12, anim));
    CU_SAFE_CALL(cuParamSetSize( func, 16 ));
    CU_SAFE_CALL(cuLaunchGrid( func, 
	mesh_width / BLOCK_SIZE, mesh_height / BLOCK_SIZE ));

//;    dim3 block(8, 8, 1);
//;    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
//;    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, anim);

    // unmap buffer object
    CU_SAFE_CALL(cuGLUnmapBufferObject(vbo));
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
CUTBoolean initGL()
{
    // initialize necessary OpenGL extensions

    glewInit();
    if (! glewIsSupported( "GL_VERSION_2_0 " 
        "GL_ARB_pixel_buffer_object"
		)) {
        fprintf( stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush( stderr);
        return CUTFalse;
    }

    // default initialization
    glClearColor( 0.8, 0.8, 0.8, 1.0);
    glDisable( GL_DEPTH_TEST);

    // viewport
    glViewport( 0, 0, window_width, window_height);

    // projection
    glMatrixMode( GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    CUT_CHECK_ERROR_GL();

    return CUTTrue;
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo)
{
    // create buffer object
    glGenBuffers( 1, vbo);
    glBindBuffer( GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof( float);
    glBufferData( GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer( GL_ARRAY_BUFFER, 0);

    // register buffer object with CUDA
    CU_SAFE_CALL(cuGLInit());
    CU_SAFE_CALL(cuGLRegisterBufferObject(*vbo));

    CUT_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO( GLuint* vbo)
{
    glBindBuffer( 1, *vbo);
    glDeleteBuffers( 1, vbo);

    CU_SAFE_CALL(cuGLUnregisterBufferObject(*vbo));

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    // run CUDA kernel to generate vertex positions
    runCuda(vbo, kernel);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();
    glutPostRedisplay();

    anim += 1.0;
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard( unsigned char key, int /*x*/, int /*y*/)
{
    switch( key) {
    case( 27) :
        deleteVBO( &vbo);
        exit( 0);
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
        mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx, dy;
    dx = x - mouse_old_x;
    dy = y - mouse_old_y;

    if (mouse_buttons & 1) {
        rotate_x += dy * 0.2;
        rotate_y += dx * 0.2;
    } else if (mouse_buttons & 4) {
        translate_z += dy * 0.01;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda( int argc, char** argv, const GLuint& vbo)
{
    printf ("check Result cuda called\n");
/*
    CU_SAFE_CALL(cuGLUnregisterBufferObject(vbo));

    // map buffer object
    glBindBuffer( GL_ARRAY_BUFFER_ARB, vbo );
    float* data = (float*) glMapBuffer( GL_ARRAY_BUFFER, GL_READ_ONLY);

    // check result
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) {
        // write file for regression test
        CUT_SAFE_CALL( cutWriteFilef( "./data/regression.dat",
            data, mesh_width * mesh_height * 3, 0.0));
    }

    // unmap GL buffer object
    if( ! glUnmapBuffer( GL_ARRAY_BUFFER)) {
        fprintf( stderr, "Unmap buffer failed.\n");
        fflush( stderr);
    }

    CU_SAFE_CALL(cuGLRegisterBufferObject(vbo));

    CUT_CHECK_ERROR_GL();
*/
}
