#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<cutil.h>
#include<boolean.h>
#include "def.h"
#include "hashtab.h"
#include "cu_lib.h"
//#include "lock.h"

#define islower(c) (((c) >= 'a') && ((c) <= 'b'))
#define isupper(c) (((c) >= 'A') && ((c) <= 'Z'))
#define isalpha(c) (islower(c) && isupper(c))
#define tolower(c) ((c)-'A'+'a')

// Hash Function

extern "C" {

__device__ int get_hash_from_char(char c){
        if (isalpha(c)){
                if (isupper(c)){
                        return tolower(c) - 'a' + 26;
                } else {
                        return c - 'a';
                }
        }
        return -1;
}

__device__ int hashfunc(char *to){
        int v1 = get_hash_from_char(*to);
        if (v1 == -1) return -1;
        if ( *(to+1) == ' ' || *(to+1) == '\0')
        {
                return v1;
        }

        int v2 = get_hash_from_char(*(to+1));
        if (v2 == -1) return -1;

        return (1+v1)*52 + v2;
}

// return pointer of h[key,index]

__device__ char* get_element2(char* h, int num_var, int key, int index, int c_size){
	int row = (key + index*num_var)*c_size;
	char *elem_addr = h + row;
	return elem_addr;
}


char* get_element3(char* h, int num_var, int key, int index, int c_size){
	int row = (key + index*num_var)*c_size;
	char *elem_addr = h + row;
	return elem_addr;
}

__device__ int put_element2(char* h, int num_var, int num_const, int c_size, int key, char* constraint, /*Lock* lock,*/  int flag){
	int i;

	for (i=0 ; i < num_const; i++){
		int row = (key + i*num_var)*c_size;
		char* temp = h + row;
		if (*temp == '\0'){
			//lock[flag].lock();
			memcpy(temp, constraint, c_size);
			//lock[flag].unlock();
			return 1;
		}
	}
	return -1;
}
 

__device__ void memcopy(char* dest, char* sour, int size){
	int i;
	for(i=0; i < size; i++)
		dest[i] = sour[i];

}

__global__ void init_constraints_kernel(char *constraints, 
                                        int num_var, 
					int num_const, 
					int size_constraint, 
					char* new_constraints, 
					char* analysis, 
					int* empty_constraint)
{
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	
	char* a_constraint = get_element2(constraints, num_var, idx, idy, size_constraint);

	int pos = (idx + idy * num_var) * size_constraint;
	if( (constraints[pos] == '_') 
		&& ( constraints[pos+1] == 'v' || constraints[pos+1] == 'c' || constraints[pos+1] == 'l')){
       		memcpy(analysis+pos, a_constraint, size_constraint);
	} 
	else {
       		memcpy(new_constraints+pos, a_constraint, size_constraint);
		*empty_constraint = 0; // constraint is not empty now.
	}
	__syncthreads();
}

extern	__shared__ char sh_array[];
					 
__global__ void solve_constraints_kernel(char* reflection, 
                                         char* constraint, 
					 int num_var, 
					 int num_const, 
					 int size_constraint, 
					 char* new_constraints, 
					 char* analysis, 
					 int* empty_constraint, 
					 //Lock* lock,
					 char* out1,
					 char* out2)
{
	char* sh_constraints = (char*)sh_array;
	register char n_const[8];
	register char n_const2[8];
	register char value[8];

	int idx = threadIdx.x;
	int idy = threadIdx.y;
  	int i;

	char* sharemem_addr = sh_constraints; 
	sharemem_addr = sharemem_addr + idx * size_constraint * BLOCK_SIZE_X + idy * size_constraint;

	memcpy(sharemem_addr, get_element2(constraint, num_var, idx, idy, size_constraint), size_constraint);	
	memcpy(out1, sharemem_addr, size_constraint);


	for(i=0; i < num_const; i++)
	{
		memcopy(value, get_element2(analysis, num_var, idx ,i, size_constraint), size_constraint);
		//memcopy(out2,value,size_constraint);	
		if (*value != '\0')
		{
			memset(n_const, '\0', 8);
			memset(n_const2, '\0', 8);
			if((*sharemem_addr=='_' && *(sharemem_addr+1)=='P') && (value[0]=='_' && value[1] == 'v'))
			{
				int index = hashfunc(sharemem_addr+2);
				memcpy(n_const,value,8);
				put_element2(analysis,num_var,num_const, size_constraint, index,n_const,0); //lock,0);
				int j;
				for(j=0;j <num_const; j++){
					char *c = get_element2(reflection,num_var,index, j, size_constraint);
					if (*c != '\0'){
						put_element2(new_constraints,num_var,num_const, size_constraint, index,c, 1);//lock,1);
						*empty_constraint = 0;
					}
					else *empty_constraint = 50;
				}
			} 
			else if((*sharemem_addr=='_' && *(sharemem_addr+1)=='C') && (value[0]=='_' && value[1] == 'c'))
			{
				int index = hashfunc(value+2);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,sharemem_addr+2,6);
				put_element2(new_constraints, num_var,num_const, size_constraint, index,n_const,1);//lock,1);
				*empty_constraint = 0;
			} 
			else if(((*sharemem_addr=='_' && *(sharemem_addr+1)=='D')) && (value[0]=='_' && value[1] == 'c'))
			{
				int index = hashfunc(value+4);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,sharemem_addr+2,6);
				put_element2(new_constraints, num_var, num_const, size_constraint, index, n_const, 0); //lock,0);
				*empty_constraint = 0;
			} 
			else if((*sharemem_addr=='_' && *(sharemem_addr+1)=='A') && (value[0]=='_' && value[1] == 'l'))
			{
				int index1 = hashfunc(value+6);
				int index2 = hashfunc(sharemem_addr +4);
				n_const[0] = '_';
				n_const2[0]= '_';
				n_const[1] = 'P';
				n_const2[1]= 'P';
				memcopy(n_const+2,sharemem_addr+2,2);
				memcopy(n_const2+2,value+2,2);
				put_element2(new_constraints, num_var, num_const, size_constraint, index1, n_const, 1);//lock,1);
				put_element2(new_constraints, num_var, num_const, size_constraint, index2, n_const2, 1);//lock,1);
				memcopy(out1,n_const,8);
				memcopy(out2,n_const2,8);
				*empty_constraint = 0;
			} 
			else if((*sharemem_addr=='_' && *(sharemem_addr+1) =='A') && (value[0]=='_' && value[1] == 't'))
			{
				int index = hashfunc(sharemem_addr+4);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,value+2,6);
				put_element2(new_constraints, num_var, num_const, size_constraint, index, n_const, 1);//lock,1);
				*empty_constraint = 0;
			} 
			else 
			{
				// *empty_constraint = 100; // - should never used.
			}
		}
		else 
		{
			// *empty_constraint = 200;
		}
	}

	__syncthreads();
}
} // end of extern "C"



extern "C"
void print_memory(char* dish, int szx, int szy, int szelt)
{
  int i,j,k,pos;
  for(i = 0 ; i < szx ; i++) {
    for(j = 0 ; j < szy ; j++) {
      pos = (j*szx + i) * szelt;
      printf("[%d,%d]\t", i, j);
      for(k=0;k<szelt;k++)
        printf("%c", dish[pos+k]);
      printf("\t");
    }
    printf ("\n");
  }
  printf ("\n");
}

// those wrappers call kernel functions
extern "C"
void solver_constraint_wrapper(hash_tab* c, char* out_analysis){
	int num_var = c->num_variable;
	int num_const = c->num_constraint;
	int size_constraint = c->size_constraint;
	int size = num_var * num_const * size_constraint; // bytes(char=1byte)
	char *c_array;
	char *c_new_array;
	char *analysis;
	char *reflection;
	char *temp; // debug 
	char *temp2; // debug 
	/*Lock lock[2];
	Lock *dev_lock;

	CUDA_SAFE_CALL(hipMalloc( (void**) &dev_lock, 2*sizeof(Lock)));
	CUDA_SAFE_CALL(hipMemcpy(dev_lock, lock, 2*sizeof(Lock), hipMemcpyHostToDevice));
	*/

	unsigned int timer = 0;

	CUDA_SAFE_CALL(hipMalloc( (void**)&c_array, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&c_new_array, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&analysis, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&reflection, size));


	temp = (char*) malloc(size);
	memset(temp,'\0',size);	

	temp2 = (char*) malloc(size);
	memset(temp2,'\0',size);	

	CUDA_SAFE_CALL(hipMemcpy(c_array, c->ht, size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(c_new_array, '\0', size));
	CUDA_SAFE_CALL(hipMemset(analysis, '\0', size));
		
	dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 grid(1);

	int empty_const= 1; // initialize that yes, it's empty. ->turn to false if put_element() occured
	int *d_empty_const; // global var on gpu as a flag of constraint_emptyness
	int size_int = sizeof(int);

	CUDA_SAFE_CALL(hipMalloc((void**)&d_empty_const, size_int));
	CUDA_SAFE_CALL(hipMemcpy(d_empty_const, &empty_const, size_int, hipMemcpyHostToDevice));

	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));

	/* Call Kernel function */
	init_constraints_kernel<<< grid, threads >>>(c_array, 
	                                             num_var, 
						     num_const, 
						     size_constraint, 
						     c_new_array, 
						     analysis, 
						     d_empty_const); 

	CUDA_SAFE_CALL(hipMemcpy(&empty_const, d_empty_const, 4, hipMemcpyDeviceToHost));
	printf("INIT-const : emptyconst flag: %d , zero means not-empty\n", empty_const);

	CUDA_SAFE_CALL(hipMemcpy(reflection, c_new_array, size, hipMemcpyDeviceToDevice));
	//printf("(let (a (lambda c c)) in\n (let b 1) in\n (app a b) \n");
	
	printf("first constraint \n");
	CUDA_SAFE_CALL(hipMemcpy(temp, c_new_array, size, hipMemcpyDeviceToHost));
	//print_memory(c->ht, BLOCK_SIZE_X, BLOCK_SIZE_Y, size_constraint);
	print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, size_constraint);
		
	printf("first analysis \n");
	CUDA_SAFE_CALL(hipMemcpy(temp2, analysis, size, hipMemcpyDeviceToHost));
	print_memory(temp2, BLOCK_SIZE_X, BLOCK_SIZE_Y, size_constraint);
		
	char* out_c_new_array;
	char* out1, *out2;

	char* o1,*o2;

	o1 =(char*) malloc(9);
	o2 =(char*) malloc(9);
	bzero(o1,9);
	bzero(o2,9);	

	hipMalloc(&out1,8);
	hipMalloc(&out2,8);

        int shared_size = BLOCK_SIZE_X * BLOCK_SIZE_Y * size_constraint;

	while (!empty_const) 
	{
		// TODO: we set constraint is empty and see whether new constraint is generated inside or not.
		empty_const = 10; 
		CUDA_SAFE_CALL(hipMemcpy(d_empty_const, &empty_const, size_int, hipMemcpyHostToDevice));
		
		CUDA_SAFE_CALL(hipMalloc(&out_c_new_array, size));
		CUDA_SAFE_CALL(hipMemset(out_c_new_array, '\0', size));
	
		// call a kernel function
		solve_constraints_kernel<<< grid, threads, shared_size>>> (
			reflection, 
			c_new_array, 
			num_var, 
			num_const, 
			size_constraint,
			out_c_new_array, 
			analysis, 
			d_empty_const,
			out1,
			out2); //dev_lock		//printf("debug11\n");

		CUDA_SAFE_CALL(hipMemcpy(&empty_const, d_empty_const, size_int, hipMemcpyDeviceToHost));
		printf ("empty_constraint flag returned = %d\n", empty_const);
		
        	hipMemcpy(o1, out1, 8, hipMemcpyDeviceToHost);
        	hipMemcpy(o2, out2, 8, hipMemcpyDeviceToHost);
		printf ("output o1, o2: %s, %s \n", o1, o2);

		printf("new constraint1:\n");		
		hipMemcpy(temp, out_c_new_array, size, hipMemcpyDeviceToHost);
		print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, size_constraint);

		hipFree(c_new_array);

		c_new_array = out_c_new_array;	
		out_c_new_array = NULL;

		printf("new analysis:\n");
		hipMemcpy(temp2, analysis, size, hipMemcpyDeviceToHost);
		print_memory(temp2, BLOCK_SIZE_X, BLOCK_SIZE_Y, size_constraint);

		printf("new constraint2:\n");
		hipMemcpy(temp, c_new_array, size, hipMemcpyDeviceToHost);
		print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, size_constraint);
	}


	CUT_SAFE_CALL(cutStopTimer(timer));
	printf("Processing time:%f (ms)\n", cutGetTimerValue(timer));
	CUT_SAFE_CALL(cutDeleteTimer(timer));

 	// now analysis contains final answers at each colume(var).
	// Need to show them: var -> value mapping by having way of finding var from column index.
	


	printf("\nSOLUTIONS=========\n");
	CUDA_SAFE_CALL(hipMemcpy(out_analysis, analysis, size, hipMemcpyDeviceToHost));
	print_memory(out_analysis, BLOCK_SIZE_X, BLOCK_SIZE_Y, size_constraint);


	hipFree(c_array);
	hipFree(c_new_array);
	hipFree(analysis);
	//hipFree(dev_lock);

	hipFree(out_c_new_array);

	free(temp);
	free(temp2);
	free(o1);
	free(o2);
/*
*/
}
