//============================================================================
// Name        : HelloWorld_CUDA.cu
// Author      : 
// Version     :
// Copyright   : Your copyright notice
// Description : Hello World in CUDA
//============================================================================

/*
* Copyright 2008, Karen Hains, UWA . All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws. Users and possessors of this source code
* are hereby granted a nonexclusive, royalty-free license to use this code
* in individual and commercial software.
*
* WE MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE. IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.
*/

/* HelloWorld Project
* This project demonstrates the basics on how to setup
* an example GPU Computing application.*
* This file contains the CPU (host) code.
*/

// Host defines
#define NUM_THREADS 8
#define STR_SIZE 50

// Includes

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>


// GPU Kernels declarations
__global__ void HelloWorld_kernel(int size, char *gpu_odata);

//////////////////////
// Program main
//////////////////////
int main( int argc, char** argv)
{

	// Host variables
	int i,nBytes;
	unsigned int num_threads;
	char *cpu_odata;
	char *string;

	// GPU variables
	char *gpu_odata;
	int str_size;
	
	// Initialize CPU variables and allocate required memory
	num_threads = (unsigned int) NUM_THREADS;
	nBytes = num_threads*STR_SIZE*sizeof(char);
	
	// Allocate and initialize CPU output vector
	string = (char *) malloc(STR_SIZE);
	if(!string) {
		printf("Cannot allocate string memory on CPU\n");
		exit(-1);
	}
	cpu_odata = (char *) malloc(nBytes);
	if(!cpu_odata) {
		printf("Cannot allocate cpu_odata memory on CPU\n");
		exit(-1);
	}
	
	// Allocate GPU (device) memory and variables
	str_size = (int) STR_SIZE;
	hipMalloc( (void**) &gpu_odata, nBytes);
	
	// Setup kernel execution parameters
	dim3 grid(1,1,1);
	dim3 threads(num_threads,1,1);
	
	// Execute the kernel on the GPU
	HelloWorld_kernel<<< grid, threads >>>(str_size, gpu_odata);
	
	// Copy result from GPU to CPU
	hipMemcpy(cpu_odata,gpu_odata,nBytes,hipMemcpyDeviceToHost);
	
	// Output results is same as the expected solution
	for(i=0;i<num_threads;i++) {
		strncpy(string,&cpu_odata[i*STR_SIZE],STR_SIZE);
		printf("From thread %d: %s\n",i,string);
	}
	
	//////////////////////////////////////////
	// All done - clean up and exit
	//////////////////////////////////////////
	// Free up CPU memory
	free(cpu_odata);
	
	// Free up GPU memory
	hipFree(gpu_odata);

}
