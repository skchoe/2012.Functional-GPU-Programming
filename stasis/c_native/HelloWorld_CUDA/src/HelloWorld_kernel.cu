/*
* Copyright 2008, Karen Hains, UWA . All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws. Users and possessors of this source code
* are hereby granted a nonexclusive, royalty-free license to use this code
* in individual and commercial software.
*
* WE MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE. IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.
*/

#ifndef _HELLOWORLD_KERNEL_H_
#define _HELLOWORLD_KERNEL_H_

#include <stdio.h>


///////////////////////////////////////////////////////////
// Simple Hello World kernel
// @param gpu_odata output data in global memory
///////////////////////////////////////////////////////////
__global__ void HelloWorld_kernel(int size, char *gpu_odata)
{
	int i,k,x,n,last_num;
	int idx,not_done;

	// Access thread id and block id
	const unsigned int thread_idx = threadIdx.x;
	const unsigned int block_idx = blockIdx.x;

	// Write data to global memory
	idx = 0;
	gpu_odata[thread_idx*size + idx++] = 'H';
	gpu_odata[thread_idx*size + idx++] = 'e';
	gpu_odata[thread_idx*size + idx++] = 'l';
	gpu_odata[thread_idx*size + idx++] = 'l';
	gpu_odata[thread_idx*size + idx++] = 'o';
	gpu_odata[thread_idx*size + idx++] = ' ';
	gpu_odata[thread_idx*size + idx++] = 'W';
	gpu_odata[thread_idx*size + idx++] = 'o';
	gpu_odata[thread_idx*size + idx++] = 'r';
	gpu_odata[thread_idx*size + idx++] = 'l';
	gpu_odata[thread_idx*size + idx++] = 'd';
	gpu_odata[thread_idx*size + idx++] = ' ';
	gpu_odata[thread_idx*size + idx++] = 'F';
	gpu_odata[thread_idx*size + idx++] = 'r';
	gpu_odata[thread_idx*size + idx++] = 'o';
	gpu_odata[thread_idx*size + idx++] = 'm';
	gpu_odata[thread_idx*size + idx++] = ' ';
	gpu_odata[thread_idx*size + idx++] = 'T';
	gpu_odata[thread_idx*size + idx++] = 'h';
	gpu_odata[thread_idx*size + idx++] = 'r';
	gpu_odata[thread_idx*size + idx++] = 'e';
	gpu_odata[thread_idx*size + idx++] = 'a';
	gpu_odata[thread_idx*size + idx++] = 'd';
	gpu_odata[thread_idx*size + idx++] = ' ';

	// Convert thread id to chars
	// Determine number of places in thread idx
	not_done = 1;
	k = 10;
	n = 1;
	while(not_done == 1) {
		x = thread_idx/k;
		if (x>0) {
			k = k*10;
			n +=1;
		}
		else
			not_done = 0;
	}

	// Parse out the thread index and convert to chars
	k = k/10;
	last_num = 0;
	for(i=n;i>0;i--) {
		x = thread_idx/k-last_num;
		gpu_odata[thread_idx*size + idx++] = '0' + x;
		last_num = (thread_idx/k)*10;
		k = k/10;
	}

	gpu_odata[thread_idx*size + idx++] = ' ';
	gpu_odata[thread_idx*size + idx++] = 'i';
	gpu_odata[thread_idx*size + idx++] = 'n';
	gpu_odata[thread_idx*size + idx++] = ' ';
	gpu_odata[thread_idx*size + idx++] = 'B';
	gpu_odata[thread_idx*size + idx++] = 'l';
	gpu_odata[thread_idx*size + idx++] = 'o';
	gpu_odata[thread_idx*size + idx++] = 'c';
	gpu_odata[thread_idx*size + idx++] = 'k';
	gpu_odata[thread_idx*size + idx++] = ' ';

	// Convert block id to chars
	// Determine number of places in thread idx
	not_done = 1;
	k = 10;
	n = 1;
	while(not_done == 1) {
		x = block_idx/k;
		if (x>0) {
			k = k*10;
			n +=1;
		}
		else
			not_done = 0;
	}

	// Parse out the block index and convert to chars
	k = k/10;
	last_num = 0;
	for(i=n;i>0;i--) {
		x = block_idx/k-last_num;
		gpu_odata[thread_idx*size + idx++] = '0' + x;
		last_num = (block_idx/k)*10;
		k = k/10;
	}

	// Fill out rest of string
	for(i=idx;i<size;i++)
		gpu_odata[thread_idx*size + idx++] = ' ';
}

#endif // #ifndef _HELLOWORLD_KERNEL_H_
