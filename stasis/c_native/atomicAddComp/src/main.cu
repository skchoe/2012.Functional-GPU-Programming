#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Feb 20, 2012
 *      Author: skchoe
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sm_11_atomic_functions.h>
#include <>

#include <cutil.h>

typedef unsigned char byte;

#define WARP_WIDTH 16
#define W 256
#define H 1




__global__ void kernel_shdatm(int* in, long int* n, int* out)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	__shared__ int shd[WARP_WIDTH];
	//shd[threadIdx.x] = in[j];
	//__syncthreads();

	long int i;
	for(i=0;i<*n;i++){
	  //atomicAdd((int*)&(shd[threadIdx.x]), i );
      shd[threadIdx.x] = 0;
	}

	out[j] =shd[threadIdx.x];
	//__syncthreads();
	return;
}

__global__ void kernel_glbatm(int* in, long int *n, int* out)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	long int i,k;
	for(i=0;i<*n;i++) {
	  //atomicAdd((int*)&(out[j]), i);
	  out[j] = 0;
	}
	//__syncthreads();
	return;
}

int main(int argc, char** argv)
{
	int i;
	int const_dim = W*H;
	size_t memsz = sizeof(int) * const_dim; // byte

	long int n = 100000000000;

	// one memory alloc
	int* in = (int*)malloc(memsz);
//    printf("in-\n");
//    for(i=0;i<W;i++){
//    	in[i]= 0;
//    	printf("%d(%d)\t", i, in[i]);
//    }
//    printf("\n");

    ////////////////////////////////////////////
//    hipEvent_t startg, stopg;
//    hipEventCreate(&startg);
//    hipEventCreate(&stopg);
    long int  *ng;

    hipMalloc(&ng, sizeof(long int));
    hipMemcpy(ng, (const void*)&n, sizeof(long int), hipMemcpyHostToDevice);


    unsigned int timerg = 0;
    cutCreateTimer(&timerg);
    cutResetTimer(timerg);
	cutStartTimer(timerg);

//    hipEventRecord(startg, 0);

    int* g_ing;
        hipMalloc((void**) &g_ing, memsz);
        hipMemcpy((void*)g_ing, in, memsz, hipMemcpyHostToDevice);

        int* g_outg;
        hipMalloc((void**) &g_outg, memsz);
        hipMemcpy((void*)g_outg, in, memsz, hipMemcpyHostToDevice);

        kernel_glbatm<<<W/WARP_WIDTH, WARP_WIDTH>>>(g_ing, ng, g_outg);

        int* outg = (int*)malloc(memsz);
        hipMemcpy(outg, g_outg, memsz, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

//    hipEventRecord(stopg, 0);
    cutStopTimer(timerg);

//    float gpu_timeg = 0.0f;
//    hipEventElapsedTime(&gpu_timeg, startg, stopg);
//    hipEventDestroy(startg);
//    hipEventDestroy(stopg);
//    printf("gpu time glb %.9f.\n", gpu_timeg);
float tvg = cutGetTimerValue(timerg);
printf("gpu time glb(%l): %f (ms)\n", n, tvg);
cutDeleteTimer(timerg);




    ///////////////////////////////////////
//hipEvent_t starts, stops;
//hipEventCreate(&starts);
//hipEventCreate(&stops);

unsigned int timers = 0;
cutCreateTimer(&timers);
cutResetTimer(timers);
cutStartTimer(timers);

//hipEventRecord(starts, 0);

int* g_in;
    hipMalloc((void**) &g_in, memsz);
    hipMemcpy((void*)g_in, in, memsz, hipMemcpyHostToDevice);

    int* g_out;
    hipMalloc((void**) &g_out, memsz);
    hipMemcpy((void*)g_out, in, memsz, hipMemcpyHostToDevice);


    kernel_shdatm<<<W/WARP_WIDTH, WARP_WIDTH>>>(g_in, ng, g_out);


    int* outs = (int*)malloc(memsz);
    hipMemcpy(outs, g_out, memsz, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

//hipEventRecord(stops, 0);
cutStopTimer(timers);

//float gpu_times = 0.0;
//hipEventElapsedTime(&gpu_times, starts, stops);
//hipEventDestroy(starts);
//hipEventDestroy(stops);
//printf("gpu time shd %f.\n", gpu_times);
float tvs = cutGetTimerValue(timers);
printf("gpu time shd(%d): %f (ms)\n", n, tvs);
cutDeleteTimer(timers);


float ratio = tvs / tvg * 100;
printf("shd/global (percent): %f %\n", ratio);
/*
    printf("out-\n");
    for(i=0;i<W;i++)
    	printf("%d(%d)\t", i, outg[i]);
    printf("\n");
*/

	return 0;
}
