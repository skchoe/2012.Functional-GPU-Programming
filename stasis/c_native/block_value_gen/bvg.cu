#include "hip/hip_runtime.h"
/*
 * bvg.cu
 *
 *  Created on: Mar 19, 2012
 *      Author: u0332192
 */


#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void get_block_value(int* inv, int* outv)
{


	return;
}

void main(int argc, char** argv)
{
	unsigned int N = pow(2,4);
	size_t sz = sizeof(int) * N;
	int* inv = (int*)malloc(sz);
	int i;
	for(i=0;i<N;i++) inv[i] = i;

	dim3 grd (N, 1);
	dim3 blk (1, 1);

	int* ginv;
	hipMalloc(&ginv, sz);
	hipMemcpy(ginv, inv, sz, hipMemcpyHostToDevice);

	int* goutv;
	hipMalloc(&goutv, sz);
	hipMemset(goutv, 0, sz);

	get_block_value<<<grd, blk>>>(ginv, goutv);



	return;
}
