#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Feb 23, 2012
 *      Author: u0332192
 */

// input = n x n w/ element are row number
// output = n x n w/ element filled in parallel

#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <sm_11_atomic_functions.h>

__device__ unsigned int* shared_part(unsigned int th, unsigned int spart_width,
		int block_width, int block_idx)
{
	extern __shared__ unsigned int s_lock_nums[];
	unsigned int offset = (th * spart_width) + (block_idx * block_width);
	return (unsigned int*) (s_lock_nums + offset);
}


// need to fit var_lock, var_nums in shared memory.
__global__ void gaddx(int* w, int* h, unsigned int* spart_width, unsigned long* gin, unsigned int* var_lock, unsigned int* var_nums, unsigned long* gout)
{
	// Shared Memory setting
	unsigned int* s_lock = shared_part(0, *spart_width, blockDim.x, blockIdx.x);
	unsigned int* s_nums = shared_part(1, *spart_width, blockDim.x, blockIdx.x);

	unsigned int i;
	for(i=0;i<*spart_width;i++) {
		s_lock[i] = var_lock[blockIdx.x * *spart_width + i];
		s_nums[i] = var_nums[blockIdx.x * *spart_width + i];
	}

	__syncthreads();

	// copy by threads.
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int k = x + y * *w;

	while( atomicCAS(&(s_lock[x]), 0, 1) != 0 );
//	gout[k] = gin[k];
	gout[s_nums[x] + y * *w] = gin[k];
	s_nums[x]++;

	atomicExch(&(s_lock[x]), 0);

	return;
}

void addx(int w, int h, int amt, unsigned long* gin, unsigned long* gout)
{
	int i,j,k;
	for(j=0;j<h;j++) {
		for(i=0;i<w;i++) {
			int k = i + j * w;
			gout[k] = gin[k];
		}
	}
	return;
}

int sharedMemory()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0); // 0=dev single device
	return deviceProp.sharedMemPerBlock;
}


// return pointer to array of x y z.
int* max_grid()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0); // 0=dev single device
	return deviceProp.maxGridSize;
}

int main(int argc, char** argv)
{
	int i, j, k;

	unsigned int W =  16;//16; // W x W is maximum allowable number of threads.
	unsigned int scale = 1000;//50

	int sW = scale*W; // similar to num_var

	int amt = 1;

	size_t sz = sizeof(unsigned long) * sW * sW;
	size_t sz1 = sizeof(unsigned long) * sW;

	unsigned long* pinput = (unsigned long*) malloc (sz);
	for(i=0;i<sW;i++) // height
		for(j=0;j<sW;j++){ //width
			k = j + i*sW;
			pinput[k] = i;
		}

	unsigned long* poutput = (unsigned long*) malloc(sz);

	// begin cpu time check
	clock_t tv_start = clock();

    addx(sW, sW, amt, pinput, poutput);

	clock_t tv_stop = clock();
	double dateclock_diff_ms = (double)(tv_stop - tv_start) * 1000. / CLOCKS_PER_SEC;

	printf("elapsed time for cpu:%e (%f)(ms)\n", dateclock_diff_ms, dateclock_diff_ms);

	//begin gpu timer
    unsigned int timerg = 0;
    cutCreateTimer(&timerg);
    cutResetTimer(timerg);
	cutStartTimer(timerg);

	int SHD_CAP = sharedMemory();

	int num_shdmem_item = 2; // var_lock, var_nums (integers)
	if(SHD_CAP  <= 2 * sW * sizeof(int))
		printf("shared memory is too small(%d byte) to be fit for num_var\n", SHD_CAP);
	else
		printf("shared memory is can contain num_var (%d bytes)\n", SHD_CAP);

	dim3 GRID_DIM = dim3(sW/W, sW/W);
	dim3 NUM_THREAD = dim3(W, W);

	int SHD_NEED = sizeof(int) * (W + W); // one W: var_lock, the other W: var_nums.
	if(SHD_CAP  <= SHD_NEED)
		printf("shared memory (lock+nums) is too small(%d byte) to be fit for num_var\n", SHD_NEED);
	else
		printf("shared memory (lock+nums) is can contain num_var (%d bytes)\n", SHD_NEED);

	// gpu struct
	unsigned long* pginput;
	hipMalloc(&pginput, sz);
	hipMemcpy(pginput, pinput, sz, hipMemcpyHostToDevice);

	unsigned long* pgoutput;
	hipMalloc(&pgoutput, sz);

	// lock for each column
	size_t var_sz = sW * sizeof(unsigned int);
	unsigned int* gvar_lock;
	hipMalloc(&gvar_lock, var_sz);
	hipMemset(gvar_lock, 0, var_sz);

	unsigned int* gvar_nums;
	hipMalloc(&gvar_nums, var_sz);
	hipMemset(gvar_nums, 0, var_sz);

	gaddx<<<GRID_DIM, NUM_THREAD, SHD_NEED>>>(&sW, &sW, &W, pginput, gvar_lock, gvar_nums, pgoutput);

	hipMemcpy(poutput, pgoutput, sz, hipMemcpyDeviceToHost);

    cutStopTimer(timerg);

    double tvg = cutGetTimerValue(timerg);
    printf("gpu time glb(-scale:%d): %e (%f)(ms)\n", scale, tvg, tvg);
    cutDeleteTimer(timerg);


    printf("diff: cpu-gpu:%e, percentage gpu/cpu:%f\n",
    		dateclock_diff_ms - timerg,
    		(double)timerg / (double)dateclock_diff_ms);
//
//	int sum_cpu = 0;
//	for(i=0;i< sW;i++)
//		sum_cpu += poutput[i];
//    for(i=0;i< sW;i++)
//    	printf("[%d]%d, ", i, poutput[i]);
//
//	printf("\n sum of all output (cpu) = %d\n", sum_cpu);
//
//	int sum_gpu = 0;
//	for(j=0;j< sW;j++)
//		sum_gpu += poutput[j];
//
//    for(j=0;j< sW;j++)
//    	printf("[%d]%d, ", j, poutput[j]);
//
//    printf("\n sum of all output (gpu) = %d\n", sum_gpu);
//
//
////	int i,j, k;
////	for(i=0;i<sW;i++) {
////		for(j=0;j<sW;j++){
////			k= j + i * sW;
////			printf("%d,%d => %d\n", i,j,poutput[k]);
////		}
////	}

	return 0;
}
