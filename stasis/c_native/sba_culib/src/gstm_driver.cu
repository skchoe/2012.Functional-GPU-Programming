#include "hip/hip_runtime.h"
#include <stdio.h>

// gpu - cuda includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cutil_common.h"
#include <gstm_common.h>
#include "gstm_kernel.h"

extern "C" int var_size_valid(byte name, int th)
{
	int num_var = constname2numconstvar(name);
	if(th < num_var) return 1;
	else return 0;
}

// const: pointer to a constraint [ name var1 var2 ...]
// th: order in sequence of variable
// var_const_max: bytes to represent the amount of variable name.
// 1) get name, 2) check range, 3) return ptr to the variable at th.(No conversion)
// output vpos: pointer to variable byte string or NULL
extern "C" void get_varstr_inconst(byte* aconst, int th, size_t var_const_max, byte* vpos, int is_uniform_var_width)
{
  byte constname = aconst[0];

  int offset = (is_uniform_var_width)? var_const_max : 1;

  if(var_size_valid(constname, th)) {
    int i;
    for(i=0;i<var_const_max;i++)
      vpos[i] = aconst[th * var_const_max + offset];
  }
  else {
    printf("Err: get_variable_inconst - th is bigg/equal to number of variable for given constraint. constname:%c, requested varidx:%d\n", constname, th);
    vpos = 0;
  }
}
// output: variable value (as integer).
extern "C" int get_variable_inconst(byte* aconst, int th, size_t var_const_max, int is_uniform_var_width)
{
	printf("get_variable in _const:%d\n", aconst[0]);
  print_a_constraint(aconst, var_const_max, is_uniform_var_width);

  // var_str: pointer to byte array where th'th variable sits
  byte* p_var_str = (byte*)malloc(sizeof(byte) * var_const_max);

  get_varstr_inconst(aconst, th, var_const_max, p_var_str, is_uniform_var_width);
  printf("-1, var_str:%s at %dth\n", p_var_str, th);
  int* p_var_num = (int*)malloc (sizeof(int));
  bytes2int(p_var_str, p_var_num, var_const_max);
  printf("-2, *p_var_num = %d\n", *p_var_num);

  return *p_var_num;
}


// output 0, if c is neither analysis name nor constraint name
// output 1, if either analysis name or constraint name
extern "C" int is_const_name(byte c)
{
  int var_size = constname2numconstvar(c);
  if(0 == var_size) return 0;
  else return 1;
}

extern "C" int is_constraintstream_empty(int num_var, size_t var_const_max, SbaStream* ss_const)
{
  int empty = 1; // set empty = TRUE.

  // check only num_const.
  int* num_const = ss_const->num_const;
  int j;
  for(j=0;j<num_var;j++) {
    if((num_const != 0) && (num_const[j] != 0)) empty = 0; // empty==FALSE -> not empty
  }

  return empty;

  /*
  byte** constnames = ss_const->constnames;
  byte** constm = ss_const->constm;

  int k, l, m;
  //check content. 
  for(k=0;k<num_var;k++) {
    int offset = 0;
    int num_const_var = num_const[k];
    byte* cname = constnames[k];
    
    for(l=0;l<num_const_var;l++) {

      // is either analysis or constraint name, then empty = 0 (means not empty).
      if (is_const_name (constm[k][offset])) {
        empty = 0;
        return empty;
      }

      int const_length = (cname[l], var_const_max, 1); // uniform_width
      offset += const_length;
    }
  }

  // at this point, constraint is turned out to be empty.
  return empty;
  */
}

// allocate space as same as input arguments
extern "C" int* init_from_num_const(int num_var, int value, GpuFlag gpu_flag)
{
  int* out_num_const;
  size_t num_const_size = num_var * sizeof(int);

  if(gpu_flag) { // gpu
    CudaSafeCall(hipMalloc((void**)&out_num_const, num_const_size));
    CudaSafeCall(hipMemset((void*)out_num_const, value, num_const_size));
  }
  else {
    out_num_const = (int*)malloc(num_const_size);
    memset((void*) out_num_const, value, num_const_size);
  }
  return out_num_const;
}

// value is initial values in the new storage
extern "C" byte** init_from_constnames(int num_var, int* num_const, byte value, GpuFlag gpu_flag)
{
  int k;
  int mx_num_const;
  size_t sz_num_var = sizeof(int) * num_var;

  printf("-----1.2-----\n");

  if(gpu_flag) { // num_var is host, num_const : byte** points gpu location.
    int* h_num_const = (int*)malloc(sz_num_var);
    CudaSafeCall(hipMemcpy(h_num_const, num_const, sz_num_var, hipMemcpyDeviceToHost));
    mx_num_const = max_num_constraints(num_var, h_num_const);
  }
  else{
	mx_num_const = max_num_constraints(num_var, num_const);
  }

  size_t constnamesp_size = sizeof (byte*) * num_var;
  byte** out_constnames;

  if(gpu_flag) { // gpu
	byte** dcnp = (byte**)malloc(constnamesp_size);;
    int* nck = (int*)malloc(sz_num_var);
    hipMemcpy((void*)nck, num_const, sz_num_var, hipMemcpyDeviceToHost);

    for(k=0;k<num_var;k++)  {
      size_t constnames_size = sizeof(byte) * nck[k];
      CudaSafeCall(hipMalloc((void**) dcnp + k, constnames_size));
      CudaSafeCall(hipMemset((void*) *(dcnp + k), value, constnames_size));
	}
    CudaSafeCall(hipMalloc((void**)&out_constnames, constnamesp_size));
    hipMemcpy(out_constnames, dcnp, constnamesp_size, hipMemcpyDeviceToDevice);
  }
  else {
    out_constnames = (byte**) malloc (constnamesp_size);
    for(k=0;k<num_var;k++) {
      out_constnames[k] = (byte*) malloc (sizeof(byte) * mx_num_const);
      memset((void*)out_constnames[k], value, (size_t)mx_num_const);
    }
  }
  return out_constnames;
}

// value is initial values in the new storage
// num_const is gpu address
extern "C" byte** init_from_constm(int num_var, int* num_const, size_t var_const_max, byte value, GpuFlag gf, int is_uniform_var_width)
{
  // preparing for answer storage by taking biggest constraint with maximum number of constraints
  // for every variable. (sufficiently large area)
  size_t sz_num_var = sizeof(int) * num_var;
  int mx_const_byte = longest_constraint_byte(var_const_max, is_uniform_var_width);
  int mx_num_const;
  if(gf) { // num_var is host, num_const : byte** points gpu location.
    int* h_num_const = (int*)malloc(sz_num_var);
    CudaSafeCall(hipMemcpy(h_num_const, num_const, sz_num_var, hipMemcpyDeviceToHost));
    mx_num_const = max_num_constraints(num_var, h_num_const);
  }
  else{
	mx_num_const = max_num_constraints(num_var, num_const);
  }
  size_t constmp_size = num_var*sizeof(byte*);
  byte** out_constm;
  int k;

  if(gf) { // gpu
	byte** dstmp = (byte**)malloc(constmp_size);;
    int* nck = (int*)malloc(sz_num_var);
    hipMemcpy((void*)nck, num_const, sz_num_var, hipMemcpyDeviceToHost);
	for(k=0;k<num_var;k++)  {

	  size_t constm_size = sizeof(byte) * nck[k] * mx_const_byte;
      CudaSafeCall(hipMalloc((void**) dstmp + k, constm_size));
      CudaSafeCall(hipMemset((void*) *(dstmp + k), value, constm_size));
	}
    CudaSafeCall(hipMalloc((void***)&out_constm, constmp_size));
    hipMemcpy(out_constm, dstmp, constmp_size, hipMemcpyDeviceToDevice);
  }
  else {
    out_constm = (byte**) malloc (sizeof(byte*) * num_var);
    for(k=0;k<num_var;k++) {
      int size_out = mx_const_byte * mx_num_const;
      out_constm[k] = (byte*) malloc (sizeof (byte) * size_out);
      memset((void*)out_constm[k], value, (size_t)size_out);
    }
  }
  return out_constm;
}


extern "C" SbaStream* SbaStream_init_empty (int num_var)
{
  SbaStream *ss = (SbaStream*)malloc(sizeof(SbaStream));
  ss->num_const = (int*)malloc(sizeof(int) * num_var);
  ss->constnames = (byte**)malloc(sizeof(byte*) * num_var);
  ss->constm = (byte**)malloc(sizeof(byte*) * num_var);

  int i;
  for(i=0;i<num_var;i++) {
    ss->num_const[i] = 0;
    ss->constnames[i] = 0;
    ss->constm[i] = 0;
  }

  //printf ("ss:%p, num_const:%p, constnames:%p, constm:%p\n", ss, ss->num_const, ss->constnames, ss->constm);

  return ss;
}

// currently when gpu_flag==1,this proc doesn't do valid operation.
extern "C" void init_from_SbaStream(int num_var, size_t var_const_max, SbaStream* ss_in, SbaStream** ss_out, GpuFlag gf, int is_uniform_var_width)
{
  int value_int = 0;
  byte value_byte = 0;

  if(gf) // gpu initialization
  {
    int* num_const = init_from_num_const(num_var, value_int, gf);

    // constnames
    SbaStream* ssh_in = (SbaStream*)malloc(sizeof(SbaStream));
	CudaSafeCall(hipMemcpy ((void*)ssh_in, ss_in, sizeof(SbaStream), hipMemcpyDeviceToHost));
	  printf("---2---\n");

    byte** constnames = init_from_constnames(num_var, ssh_in->num_const, value_byte, gf);
	  printf("---3---\n");
    byte** constm = init_from_constm(num_var, ssh_in->num_const, var_const_max, value_byte, gf, is_uniform_var_width);
	  printf("---4---\n");

    SbaStream* ss_store;
    CudaSafeCall(hipMalloc((void**) & ss_store, sizeof(SbaStream)));
	  printf("---5---\n");
    CudaSafeCall(hipMemcpy((void*) ss_store->num_const, num_const, sizeof(int*), hipMemcpyDeviceToDevice));
	  printf("---6---\n");
    CudaSafeCall(hipMemcpy((void*) ss_store->constnames, constnames, sizeof(byte**), hipMemcpyDeviceToDevice));
	  printf("---7---\n");
    CudaSafeCall(hipMemcpy((void*) ss_store->constm, constm, sizeof(byte**), hipMemcpyDeviceToDevice));
	  printf("---8---\n");

    CudaSafeCall(hipMemcpy((void*) *ss_out, ss_store, sizeof(SbaStream*), hipMemcpyDeviceToDevice));
  }
  else // gpu_flag==0 host
  {
    int* num_const = init_from_num_const(num_var, value_int, gf);
    byte** constnames = init_from_constnames(num_var, ss_in->num_const, value_byte, gf);
    byte** constm = init_from_constm(num_var, ss_in->num_const, var_const_max, value_byte, gf, is_uniform_var_width);

    SbaStream* ss_store = (SbaStream*)malloc(sizeof(SbaStream));
    ss_store->num_const = num_const;
    ss_store->constnames = constnames;
    ss_store->constm = constm;

    *ss_out = ss_store;
  }
  return;
}

extern "C" int* copy_from_num_const(int num_var, int* num_const)
{
  int i;
  int* out_num_const = (int*)malloc(sizeof(int) * num_var);
  for(i=0;i<num_var;i++)
    out_num_const[i] = num_const[i];

  return out_num_const;
}

extern "C" byte** copy_from_constnames(int num_var, int* num_const, byte** constnames)
{
  int i,j;
  byte** out_constnames = (byte**) malloc (sizeof(byte*) * num_var);
  for(i=0;i<num_var;i++) {
    out_constnames[i] = (byte*) malloc (sizeof(byte) * num_const[i]);
    for(j=0;j<num_const[i];j++)
      out_constnames[i][j] = constnames[i][j];
  }

  printf("copy_from_constnames:%p memcpyn", out_constnames);

  return out_constnames;
}

extern "C" byte** copy_from_constm(int num_var, int* num_const, byte** constname, size_t var_const_max, byte** constm)
{
  int i,j;
  byte** out_bytes = (byte**) malloc (sizeof(byte*) * num_var);
  size_t* vconstm_size = (size_t*) malloc (sizeof (size_t) * num_var);

  for(i=0;i<num_var;i++) {
    for(j=0;j<num_const[i];j++) 
      vconstm_size[i] += constraint_length(constname[i][j], var_const_max, 1);
  }

  for(i=0;i<num_var;i++) {
    int constsize_byte = vconstm_size[i];

    byte* vconst_src = constm[i];
    byte* vconst_dst = (byte*) malloc (sizeof(byte) * constsize_byte);

    memcpy(vconst_dst, vconst_src, constsize_byte);

    out_bytes[i] = vconst_dst;
  }

  return out_bytes;
}


// host code.
extern "C" SbaStream* copy_from_SbaStream(int num_var, size_t var_const_max, SbaStream* ss_in)
{
  //printf("-------reflection---------\n");
  int*   ref_num_const  = copy_from_num_const (num_var, ss_in->num_const);
  byte** ref_constnames = copy_from_constnames(num_var, ss_in->num_const,
                                             ss_in->constnames);
  byte** ref_constm     = copy_from_constm(num_var, ss_in->num_const,
                        ss_in->constnames, var_const_max, ss_in->constm);

  SbaStream *ss_out = (SbaStream*)malloc(sizeof(SbaStream));
    ss_out->num_const = ref_num_const;
    ss_out->constnames = ref_constnames;
    ss_out->constm = ref_constm;

  return ss_out;
}
// ss_f (from) --> ss_t (to)
// currently implemented from HOST to DEVICE
extern "C" void sbastream_alloc_copy(int num_var, size_t var_const_max, SbaStream* ss_f, SbaStream** ss_t, hipMemcpyKind direction)
{
  // HOST -> Device
  if(direction == hipMemcpyHostToDevice) {

    // Number of constraints ///////////////////////////////////////////
    int* num_const_f = ss_f->num_const; //array of number_of_constraints
    size_t num_const_size = num_var * sizeof(int);

    int* num_const_t = 0; // pointer to destination array of num_const
    CudaSafeCall(hipMalloc((void**)&num_const_t,  num_const_size));

    hipMemcpy(num_const_t,  num_const_f,  num_const_size, direction);
    printf("num_const:%p, %d\n", num_const_f, num_const_f[0]);

    size_t pconsts_size = num_var * sizeof(byte*);

    // Constraint Names ///////////////////////////////////////////
    byte** constnames_f = ss_f->constnames;
    byte** h_pcn = (byte**)malloc(num_var * sizeof (byte*));

    int i;
    for (i=0;i<num_var;i++) {
      size_t cnlength = num_const_f[i] * sizeof(byte);
      printf ("constname at %d(%d), %p \n", i, cnlength, constnames_f[i]);
      if(cnlength > 0) printf ("first char:%c\n", constnames_f[i][0]);

      hipMalloc((void**) h_pcn+i, cnlength);
      hipMemcpy(*(h_pcn+i), constnames_f[i], cnlength, hipMemcpyHostToDevice);
    }
    byte** constnames_t;
    hipMalloc((byte**) &constnames_t, pconsts_size);
    CudaSafeCall(hipMemcpy(constnames_t, h_pcn, pconsts_size, hipMemcpyHostToDevice));

    // constraint stream: constm ///////////////////////////////////////////
    byte** constm_f = ss_f->constm;
    byte** h_pctm = (byte**)malloc(num_var * sizeof (byte*));

    for (i=0;i<num_var;i++) {
      size_t constms_byte = sizeof(byte) * sum_const_sizes(constnames_f[i], num_const_f[i], var_const_max);
      hipMalloc((void**) h_pctm + i, constms_byte);
      hipMemcpy(*(h_pctm + i), constm_f[i], constms_byte, hipMemcpyHostToDevice);
      if(constm_f[i] != 0) printf("constm(%d) first char:%c\n", i, constm_f[i][0]);
      else printf("cosntm(%d) is null\n", i);
    }
    byte** constm_t = 0; // device
    hipMalloc((void**) &constm_t, pconsts_size);
    hipMemcpy(constm_t, h_pctm, pconsts_size, hipMemcpyHostToDevice);

    // SbaStream //////////////////////////////////////////////
    SbaStream* ss_htmp = (SbaStream*)malloc(sizeof(SbaStream));
    ss_htmp->num_const = num_const_t;
    ss_htmp->constnames = constnames_t;
    ss_htmp->constm = constm_t;
    printf("Sbastm is made, %p, %p, %p\n", ss_htmp->num_const, ss_htmp->constnames, ss_htmp->constm);
    hipMalloc(ss_t, sizeof(SbaStream));
    hipMemcpy((void*) *ss_t, ss_htmp, sizeof(SbaStream), hipMemcpyHostToDevice);

  }
  // Device -> HOST
  else {
    SbaStream* tmp = (SbaStream*)malloc(sizeof(SbaStream));
    hipMemcpy(tmp, ss_f, sizeof(SbaStream), hipMemcpyDeviceToHost);
    printf("upto here, %p, %p, %p\n", tmp->num_const, tmp->constnames, tmp->constm);

    // num_const
    int num_const_size = sizeof(int) * num_var;
    int* num_const_h = (int*)malloc(num_const_size);
    hipMemcpy(num_const_h, tmp->num_const, num_const_size, hipMemcpyDeviceToHost);
    int i;
	for(i=0;i<num_var;i++) {
	  printf("numconst at %d = %d\n", i, num_const_h[i]);
	}

	// constnames
	int cnp_size = num_var* sizeof(byte*);
	byte** cn_ptr = (byte**)malloc(cnp_size);
	hipMemcpy(cn_ptr, tmp->constnames, cnp_size, hipMemcpyDeviceToHost);
	printf("ptr-cn fetch\n");
	byte** cnp = (byte**) malloc (num_var * sizeof(byte*));
	for(i=0;i<num_var;i++) {
	  size_t constnames_size = sizeof(byte) * num_const_h[i];
	  cnp[i] = (byte*) malloc (constnames_size);
	  hipMemcpy(cnp[i], cn_ptr[i], constnames_size, hipMemcpyDeviceToHost);
	  int j;
	  for(j=0;j<constnames_size;j++)
	    printf("%dth byte:%c\n", j, cnp[i][j]);
	}

	// constm
	byte** cstm_ptr = (byte**)malloc(cnp_size);
	hipMemcpy(cstm_ptr, tmp->constm, cnp_size, hipMemcpyDeviceToHost);  // pointers
	printf("ptr-stc fetch, %d\n", num_var);
	byte** cstmp = (byte**) malloc (num_var * sizeof(byte*));
	for(i=0;i<num_var;i++) {
	  size_t constms_byte = sizeof(byte) * sum_const_sizes(cnp[i], num_const_h[i], var_const_max);
	  cstmp[i] = (byte*) malloc (constms_byte);
	  hipMemcpy(cstmp[i], cstm_ptr[i], constms_byte, hipMemcpyDeviceToHost);
	  int j;
	  for(j=0;j<constms_byte;j++)
	    printf("%dth byte:%d\n", j, cstmp[i][j]);
	}

	printf("constm end\n");

    *ss_t = (SbaStream*)malloc(sizeof(SbaStream));
    (*ss_t)->num_const  = num_const_h;
    (*ss_t)->constnames = cnp;
    (*ss_t)->constm     = cstmp;
  }
  return;
}

//always Host to Device
extern "C" void csr_alloc_copy(int num_var, int num_tot_const, int max_num_const,
		int** d_ptr, int** d_indices, byte** d_data,
		int* s_ptr, int* s_indices, byte* s_data,
		size_t sz_a_const,
		hipMemcpyKind direction) {
	// CSR structure
	size_t sz_ptr = sizeof(int) * (max_num_const + 1);
	size_t sz_indices = sizeof(int) * num_tot_const;
	size_t sz_data = sz_a_const * num_tot_const;

	printf(" csr_alloc_copy, sz_ptr:%d\n", sz_ptr);
	printf(" csr_alloc_copy, sz_indices:%d\n", sz_indices);
	printf(" csr_alloc_copy, sz_data:%d\n", sz_data);

	if(direction == hipMemcpyHostToDevice) {
		CudaSafeCall(hipMalloc(d_ptr, sz_ptr));
		CudaSafeCall(hipMalloc(d_indices, sz_indices));
		CudaSafeCall(hipMalloc(d_data, sz_data));
	}
	else if(direction == hipMemcpyDeviceToHost) {
		*d_ptr = (int*) malloc(sz_ptr);
		*d_indices = (int*) malloc(sz_indices);
		*d_data = (byte*) malloc(sz_data);
	}
	else {
		printf("csr_alloc_copy() - error - csr_alloc_copy only supports H2D, D2H\n");
		exit(1);
	}

	CudaSafeCall(hipMemcpy(*d_ptr, s_ptr, sz_ptr, direction));
	CudaSafeCall(hipMemcpy(*d_indices, s_indices, sz_indices, direction));
	CudaSafeCall(hipMemcpy(*d_data, s_data, sz_data, direction));


	return;
}

// input : uniform var-length, uniform constraint length
// 1 if constraint sizes are uniform
// 0 otherwise
extern "C" int warning_for_non_csr(int is_uniform_var_width, int is_equal_const_size) {
	// if all constraints are in equal length, then each variable in a constratint have equal length too.
	if(is_equal_const_size == 1) //is_uniform_var_width = 1;
		return 1;
	else if(is_uniform_var_width) {
		printf("sba_solver_csr: This case cannot happen:constraints are not same is length, and they have uniform width in variable\n");
		exit(1);
	}
	else {
		printf("sba_solver_csr: This routine only support - equal sized constraints -and- variable lengths in constraint are uniform\n");
		exit(1);
	}
	return 0;
}

// given input from Racket(list of list), produces CSR
// copy of code in project: xform_global_shared
extern "C" void transform_const_csr(int num_var, int num_tot_const, int max_num_const, int* num_const, byte** constm,
						int* ptr, int* indices, byte* data, size_t sz_a_const, int is_uniform_var_width, int is_equal_const_size)
{
	int new_uniform_var_width = warning_for_non_csr(is_uniform_var_width, is_equal_const_size);
	if(new_uniform_var_width) is_uniform_var_width = 1;

	int i,j,k,l;

	k=0, l=0;	// l: num of valid consts

//	for(i=0;i<num_var;i++)
//		for(j=0;j<num_const[i];j++) {
//			k++;
//			if(k>8389000) {
//				printf("srcccccc%d th \t", k);
//				print_a_constraint(&(constm[i][j]), 4, is_uniform_var_width);
//			}
//		}
//
//	k=0;

	for(i=0;i<max_num_const;i++) {
		for(j=0;j<num_var;j++) {
			if (i < num_const[j]) {
				byte* p_const = constm[j]+ i*sz_a_const;
				memcpy(data + k*sz_a_const, p_const, sz_a_const); // general version
				// data[k] = constm[j][i]; // valid only when sz_a_const = 1
				indices[k] = j;
				l++;
				k++;

//				if(k > 8389000) {
//					printf("src-const[%d]:\t", k);
//					print_a_constraint(p_const, 4, is_uniform_var_width);
//				}
			}
		}
		ptr[i+1] = l;
	}
//	for(k=0;k<num_var;k++)
//		printf("num const[%d:%d\n", k, num_const[k]);

//	for(j=0;j<=max_num_const;j++)
//		printf("transform_const's ptr[%d]:%d\n", j, ptr[j]);

	return;
}


// num_elt : length of input array in cpu
// gnum_elt: pointer to length of input arary in gpu
// g_elts: pointer to the array in GPU
// output: pointer to array that contains the sum at the first location in GPU

extern "C" int* sum_gpu_kernel_int32(int num_elt, int* gnum_elt, int* gelts)
{
	size_t sz_ull = sizeof(int);
	size_t sz_elts = sz_ull * num_elt;

	int* gnum_block;
	hipMalloc(&gnum_block, sz_ull);
	hipMemset(gnum_block, 0, sz_ull);

    dim3 block1 = dim3 ((int) fmin((double)512, (double)num_elt/2.0), 1); // 512 threads deals with 1024 data.
    int num_blk;
    num_blk = (num_elt > block1.x)? num_elt / (2 * block1.x): 1; // because each thread process 2 elements

	// return storage gsum
	size_t sz_gsum = sz_ull * num_blk;
	int *gsum;
	hipMalloc(&gsum, sz_gsum);
	hipMemset(gsum, 0, sz_gsum);

	/*
    unsigned int timerg_exe = 0;
    cutCreateTimer(&timerg_exe);
    cutResetTimer(timerg_exe);

	cutStartTimer(timerg_exe);
	*/
	// Timer Event Prepare
	hipEvent_t kernel_start, kernel_stop;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	// Start record
	hipEventRecord(kernel_start, 0);

	///////////////////////////////////////////////////////////////////////////////////////////////////////////
    do {
//    	printf("--------------do-------------\nnumelt:%d\n", num_elt);
//    	int* tmpelt = (int*)malloc(sz_elts);
//    	hipMemcpy(tmpelt, gelts, sz_elts, hipMemcpyDeviceToHost);
//    	for(i=0;i<num_elt;i++)
//    		printf("tmpelt[%d] = %llu\n", i, tmpelt[i]);

        block1 = dim3 ((int) fmin((double)512, (double)num_elt/2.0), 1); // 512 threads deals with 1024 data.
        num_blk = (num_elt > block1.x)? num_elt / (2 * block1.x): 1; // because each thread process 2 elements

        int mxgrd [] = {65535, 65535, 1};
        printf("grid max = %d, %d blocks\n", mxgrd[0], mxgrd[1]);

        int gridx = (num_blk > mxgrd[0])? mxgrd[0] : num_blk;
        int gridy = (num_blk > mxgrd[0])? num_blk / mxgrd[0] + 1: 1;
        printf("num_blk:%d\n", num_blk);

        printf("grid dimension x, y = %d, %d\n", gridx, gridy);


        dim3 grid1 = dim3(gridx, gridy);
        size_t shds1 = 2 * block1.x * sizeof(int); // need factor 2 because a thread add 2 elements.

        printf("blockdim.x = %d threads, shd size = %d bytes\n", block1.x, shds1);

        // clear used location
		size_t sz_gsum = sz_ull * num_blk;

		// new grid, block, shds
		sum_kernel_int32<<<grid1, block1, shds1>>>(gnum_elt, gelts, gnum_block, gsum);

		num_elt = num_blk;
		sz_elts = sz_gsum;

		// interchange:
		int* tmp_num_elt = gnum_elt;
		int* tmp_elts = gelts;

		gnum_elt = gnum_block;
		gelts = gsum;

		gnum_block = tmp_num_elt;
		gsum = tmp_elts;

//		// copy output by printing next inputs
//		int* cnum_elt = (int*)malloc(sz_ull);
//		hipMemcpy(cnum_elt, gnum_elt, sz_ull, hipMemcpyDeviceToHost);
//		printf("next - numelt:%d\n", *cnum_elt);

//		int i;
//		int* celts = (int*)malloc(sz_elts);
//		hipMemcpy(celts, gelts, sz_elts, hipMemcpyDeviceToHost);
//		for(i=0;i<(int)*cnum_elt;i++)
//			printf("%d th next elt:%llu\n", i, celts[i]);

	} while (num_blk != 1);
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////

    /*
    cutStopTimer(timerg_exe);
    double tvg = cutGetTimerValue(timerg_exe);
    printf("gpu time glb(kernel):\n %e \n(%f)(ms)\n", tvg, tvg);

    cutDeleteTimer(timerg_exe);
	*/

	// Stop event
	hipEventRecord(kernel_stop, 0);
	hipEventSynchronize(kernel_stop);
	float kernel_elapsedTime;
	hipEventElapsedTime(&kernel_elapsedTime, kernel_start, kernel_stop); // that's our time!
	// Clean up:
	hipEventDestroy(kernel_start);
	hipEventDestroy(kernel_stop);

	printf("gpu time glb(kernel):\n %e \n(%f)(ms)\n", kernel_elapsedTime, kernel_elapsedTime);

	return gelts;
}


//__global__ void add1(int* x)
//{
//	*x = *x + 100;
//	return;
//}


// convert ss_in into CSR
// copy CSR
// return back ss_out_analysis
extern "C" void sba_solve_csr(SbaStream* ss_in, int num_var, size_t var_const_max, SbaStream* ss_out_analysis,
		int is_uniform_var_width, int is_equal_const_size)
{
	int i;

	int new_uniform_var_width = warning_for_non_csr(is_uniform_var_width, is_equal_const_size);
	if(new_uniform_var_width) is_uniform_var_width = 1;

	// uniform width for constraints
	size_t sz_a_const = var_const_max * 4; // name + 3 references to variables.

	// SbaStream -> num_tot_const
	int num_tot_const = 0;
	for(i=0;i<num_var;i++) {
		num_tot_const += ss_in->num_const[i];
	}

	printf("tot_const:%d before convert to ...\n", num_tot_const);

	// SbaStream -> max_num_const maximum number of constraints
	int max_num_const = ss_in->num_const[0];
	for(i=1;i<num_var;i++) {
		int numcst = ss_in->num_const[i];
		if(numcst > max_num_const) max_num_const = numcst;
	}

	// 3 vars for CSR structure.
	int* ptr = (int*)malloc(sizeof(int) * (1 + max_num_const));
	int* indices = (int*)malloc(sizeof(int) * num_tot_const);
	byte* data = (byte*)malloc(sizeof(byte) * num_tot_const * sz_a_const);

	//print_constraint_stream(num_var, var_const_max, ss_in, is_uniform_var_width, is_equal_const_size, 1000);

	transform_const_csr(num_var, num_tot_const, max_num_const, ss_in->num_const, ss_in->constm,
			ptr, indices, data, sz_a_const, is_uniform_var_width, is_equal_const_size);

	//print_constraints_csr(num_var, num_tot_const, var_const_max, ptr, indices, data, sz_a_const, is_uniform_var_width, is_equal_const_size, 1000);

	//////////////////////////////////////////////////////////////
	//begin gpu timer
	/*
    unsigned int timerg_all = 0;
    cutCreateTimer(&timerg_all);
    cutResetTimer(timerg_all);

    unsigned int timerg_exe = 0;
    cutCreateTimer(&timerg_exe);
    cutResetTimer(timerg_exe);

	cutStartTimer(timerg_all);
	*/
	// Timer Event Prepare
	hipEvent_t all_start, all_stop;
	hipEventCreate(&all_start);
	hipEventCreate(&all_stop);
	// Start record
	hipEventRecord(all_start, 0);



	// copy to gpu
	int* g_ptr;
	int* g_indices;
	byte* g_data;
	csr_alloc_copy(num_var, num_tot_const, max_num_const,
			&g_ptr, &g_indices, &g_data,
			ptr, indices, data,
			sz_a_const, hipMemcpyHostToDevice);

//	// testing to gpu copy by copying back to cpu
//	int* h_ptr;
//	int* h_indices;
//	byte* h_data;
//	csr_alloc_copy(num_var, num_tot_const, max_num_const,
//				&h_ptr, &h_indices, &h_data,
//				g_ptr, g_indices, g_data,
//				sz_a_const, hipMemcpyDeviceToHost);
//
//	printf("num_total constraints:%d\n", num_tot_const);
//	print_constraints_csr(num_var, num_tot_const, var_const_max, h_ptr, h_indices, h_data, sz_a_const, is_uniform_var_width, is_equal_const_size, 1000);
//

	// output matrices of init_constraints_kernel()
	size_t sz_mat = sz_a_const * num_var * max_num_const;

	byte* g_constm;
	CudaSafeCall(hipMalloc(&g_constm, sz_mat));
	CudaSafeCall(hipMemset(g_constm, 0, sz_mat));

	byte* g_asis;
	CudaSafeCall(hipMalloc(&g_asis, sz_mat));
	CudaSafeCall(hipMemset(g_asis, 0, sz_mat));

	// lock, varnum for matrices g_const, g_asis
	size_t sz_locks = sizeof(int) * num_var;
	size_t sz_varnums = sizeof(int) * num_var;

	int* g_lock_const;
	CudaSafeCall(hipMalloc(&g_lock_const, sz_locks));
	CudaSafeCall(hipMemset(g_lock_const, 0, sz_locks));

	int* g_varnum_const;
	CudaSafeCall(hipMalloc(&g_varnum_const, sz_varnums));
	CudaSafeCall(hipMemset(g_varnum_const, 0, sz_varnums));

	int* g_lock_asis;
	CudaSafeCall(hipMalloc(&g_lock_asis, sz_locks));
	CudaSafeCall(hipMemset(g_lock_asis, 0, sz_locks));

	int* g_varnum_asis;
	CudaSafeCall(hipMalloc(&g_varnum_asis, sz_varnums));
	CudaSafeCall(hipMemset(g_varnum_asis, 0, sz_varnums));

	int is_const_empty = 1;
	int *g_is_const_empty;
	CudaSafeCall(hipMalloc(&g_is_const_empty, sizeof(int)));
	CudaSafeCall(hipMemcpy(g_is_const_empty, &is_const_empty, sizeof(int), hipMemcpyHostToDevice));

	// initialize const-mat, asis-mat from ptr, indices, data
	// each matrix is size (max_num_const x num_var),
	// iteration is as long as num_tot_const
	// read indices, data write to the matrix.
	// row in matrix is defined by 1) varnum[] - current empty sp in mat, 2)lock[] - gateway showing accessability.
	unsigned int binary_num_tot_const = least_upper_binary(num_tot_const);

//	printf("binary upper bound of total const: %d\n", binary_num_tot_const);

	/*
	size_t cpyamount_device =
			sizeof(int) * (max_num_const + 1) //ptr
			+ sizeof(int) * num_tot_const		//indices
			+ sz_a_const * num_tot_const		//data
			+ 2*sz_mat + 2*sz_locks + 2*sz_varnums;
	*/
	int max_threads = (int) fmin(binary_num_tot_const, (float)pow(2,8)); //256
	dim3 block_init(max_threads, 1);
	dim3 grid_init(binary_num_tot_const/max_threads,1);

//	if(block.x * grid.x >= 512*65536) {
//		printf("cuda hw cannot support so many constraints\n");
//		exit(1);
//	}
//	else
//		printf("global memory was occupied by copy: %d bytes\n", cpyamount_device);

	/*
	cutStartTimer(timerg_exe);
	*/

	init_constraints_kernel_csr<<<grid_init, block_init>>>(
			num_var, max_num_const, num_tot_const, sz_a_const, // num of variables, longest const, total const, single const size

			g_indices, g_data, // input data in CSR form

			g_lock_const, g_varnum_const, // lock, positioner for new constm
			g_constm, // storage for new constm

			g_lock_asis, g_varnum_asis, // lock, positioner for new asis
			g_asis,

			g_is_const_empty); // set to 0 (not empty) is kernel read constraint during initialization

	// solve_constraint kernel
	printf("INIT-> no constraint exist\n");

	size_t sz_num_var = sizeof(int) * num_var;

	int* h_varnum_const = (int*)malloc(sz_num_var);
	CudaSafeCall(hipMemcpy(h_varnum_const, g_varnum_const, sz_num_var, hipMemcpyDeviceToHost));

	for(i=0;i<num_var;i++)
		printf("org varnum_const[%d] = %d\n", i, h_varnum_const[i]);

	int* h_varnum_asis = (int*)malloc(sz_num_var);
	CudaSafeCall(hipMemcpy(h_varnum_asis, g_varnum_asis, sz_num_var, hipMemcpyDeviceToHost));

	for(i=0;i<num_var;i++)
		printf("org varnum_asis[%d] = %d\n", i, h_varnum_asis[i]);

	// reflection definition
	int* g_varnum_refl;
	CudaSafeCall(hipMalloc(&g_varnum_refl, sz_varnums));
	CudaSafeCall(hipMemcpy(g_varnum_refl, g_varnum_const, sz_varnums, hipMemcpyDeviceToDevice));

	//printf("1\n");

	byte* g_reflection;
	CudaSafeCall(hipMalloc(&g_reflection, sz_mat));
	CudaSafeCall(hipMemcpy(g_reflection, g_constm, sz_mat, hipMemcpyDeviceToDevice));
	//printf("2\n");

	CudaSafeCall(hipMemcpy(&is_const_empty, g_is_const_empty, sizeof(int), hipMemcpyDeviceToHost));

	if(is_const_empty)
		printf ("constraint is not empty");
	else printf ("constraint is empty");

	int* g_lock_new_const;
	int* g_varnum_new_const;
	byte* g_new_constm;

	int binary_max_num_const = least_upper_binary(max_num_const);
	int n_compare = binary_max_num_const * binary_max_num_const;
	int max_threads_block = 512;
	int bl = max_threads_block < n_compare;

	if(bl) {
		printf("constraint - comparisons (%d) are more than max number of threads(%d) -> exit",
				n_compare, max_threads_block);
		exit(1);
	}
	printf ("max num const: %d\n", binary_max_num_const);

	int blk_c = max_num_const;
	int blk_v = max_threads_block / n_compare;
	dim3 block_solve(blk_v, blk_c, blk_c);

	int gl = num_var / blk_v;
	int grd_x = (gl)? gl : 1;
	dim3 grid_solve(grd_x, 1, 1);

	// this is for collecting sum of varnums in each block.
	// we'll check if one of them are not zero -> iteration needed.
	int* gsum_varnum_grid;
	size_t sz_varnum_grid = sizeof(int) * grid_solve.x;
	CudaSafeCall(hipMalloc(&gsum_varnum_grid, sz_varnum_grid));
	CudaSafeCall(hipMemset(gsum_varnum_grid, 0, sz_varnum_grid));

	// Shared memory size definition // for first kernel (solve_constraints)
	size_t shd_solve = 2 * num_var * sizeof(int); // as big as to hold 2 arrays : lock_const, lock_asis

	// Shared memory size definition // for second kernel (or_varnum_grid)
	// for each block, we compute or_varnum_grid_kernel for the block
	// gather to global memory, apply(or_varnum_grid_kernel) again until only one elt left in global.

	//size_t shd_varnum = block_solve.x * sizeof(int);
	int* g_sum_varnum;
	CudaSafeCall(hipMalloc(&g_sum_varnum, sizeof(int)));
	CudaSafeCall(hipMemset(g_sum_varnum, 0, sizeof(int)));

	int cnt=0;
	while(!is_const_empty) {

		CudaSafeCall(hipMemcpy(g_is_const_empty, &is_const_empty, sizeof(int), hipMemcpyHostToDevice));

		// new constraint - pair
		CudaSafeCall(hipMalloc(&g_lock_new_const, sz_varnums));
		CudaSafeCall(hipMemset(g_lock_new_const, 0, sz_varnums));

		CudaSafeCall(hipMalloc(&g_varnum_new_const, sz_varnums));
		CudaSafeCall(hipMemset(g_varnum_new_const, 0, sz_varnums));

		CudaSafeCall(hipMalloc(&g_new_constm, sz_mat));
		CudaSafeCall(hipMemset(g_new_constm, 0, sz_mat));

		// testing sensor insertion
		int sensor = 101;
		int* g_sensor;
		CudaSafeCall(hipMalloc(&g_sensor, sizeof(int)));
		CudaSafeCall(hipMemcpy(g_sensor, &sensor, sizeof(int), hipMemcpyHostToDevice));

//		// test for thread assignment
//		dim3 g(1,1);
//		dim3 b(1,1,1);
//		printf("is constraint empty? = %d\n", is_const_empty);
//		printf("grid x y = %d, %d\n", grid_solve.x, grid_solve.y);
//		printf("block x y z =  %d, %d, %d\n", block_solve.x, block_solve.y, block_solve.z);

		// g_const_sample to see throught the kernel inside.
		int* g_const_sample;
		CudaSafeCall(hipMalloc(&g_const_sample, sz_a_const));

		// kernel
		solve_constraints_kernel<<<grid_solve, block_solve, shd_solve>>>(
				num_var, max_num_const, sz_a_const,
				g_varnum_refl, g_reflection,
				g_varnum_const, g_constm,
				g_lock_asis, g_varnum_asis, g_asis,
				g_lock_new_const, g_varnum_new_const, g_new_constm, g_sensor, g_const_sample);


		sensor = 99;
		CudaSafeCall(hipMemcpy(&sensor, g_sensor, sizeof(int), hipMemcpyDeviceToHost));
		printf("sensor = %d\n", sensor);

		byte* sample = (byte*)malloc(sz_a_const);
		CudaSafeCall(hipMemcpy(sample, g_const_sample, sz_a_const, hipMemcpyDeviceToHost));
		print_a_constraint(sample, var_const_max, is_uniform_var_width);

//		 check emptyness of constraints by adding them all and get is_const_emtpy
		printf("in while empty? = %d\n", is_const_empty);

		int* tmpvarnum = (int*)malloc(sz_varnums);
		CudaSafeCall(hipMemcpy(tmpvarnum, g_varnum_new_const, sz_varnums, hipMemcpyDeviceToHost));
		for(i=0;i<num_var;i++)
			printf("%d - varnum const [%d] = %d\n", cnt, i, tmpvarnum[i]);
		cnt++;

		int* gnum_var;
		CudaSafeCall(hipMalloc(&gnum_var, sizeof(int)));
		CudaSafeCall(hipMemcpy(gnum_var, &num_var, sizeof(int), hipMemcpyHostToDevice));
		g_sum_varnum = sum_gpu_kernel_int32 (num_var, gnum_var, g_varnum_new_const);

		int* sum_varnum = (int*) malloc(sizeof(int));
		CudaSafeCall(hipMemcpy(sum_varnum, g_sum_varnum, sizeof(int), hipMemcpyDeviceToHost));
		printf("sum of all varnum_const = %d\n", *sum_varnum);

		is_const_empty = 1;//(*sum_varnum == 0) ? 1 : 0; // sun=0 -> empty const -> is_const_emtpy  = 1.
	}

	/*
    cutStopTimer(timerg_exe);
    cutStopTimer(timerg_all);
	*/
	printf("end of kernel invoke\n");

	// test g_varnum_const, g_varnum_asis
	int* varnum_const = (int*)malloc(sz_varnums);
	memset(varnum_const, 0, sz_varnums);
	CudaSafeCall(hipMemcpy(varnum_const, g_varnum_const, sz_varnums, hipMemcpyDeviceToHost));

	byte* h_constm = (byte*) malloc(sz_mat);
	CudaSafeCall(hipMemcpy(h_constm, g_constm, sz_mat, hipMemcpyDeviceToHost));

	int* varnum_asis = (int*)malloc(sz_varnums);
	memset(varnum_asis, 0, sz_varnums);
	CudaSafeCall(hipMemcpy(varnum_asis, g_varnum_asis, sz_varnums, hipMemcpyDeviceToHost));

	byte* h_asis = (byte*) malloc(sz_mat);
	CudaSafeCall(hipMemcpy(h_asis, g_asis, sz_mat, hipMemcpyDeviceToHost));

	printf("numvar:%d\n", num_var);
	for(i=0;i<num_var;i+=1) {
		printf("init -- var[%d], numconst:%d, num_ais:%d\n", i, varnum_const[i], varnum_asis[i]);
	}

	int* varnum_new_const = (int*)malloc(sz_varnums);
	memset(varnum_new_const, 0, sz_varnums);
	CudaSafeCall(hipMemcpy(varnum_new_const, g_varnum_new_const, sz_varnums, hipMemcpyDeviceToHost));
	for(i=0;i<num_var;i+=1) {
		printf("after exec -- var[%d], numconst:%d, num_ais:%d\n", i, varnum_new_const[i], varnum_asis[i]);
	}

	/*
    double tvg = cutGetTimerValue(timerg_exe);
    printf("gpu time glb(kernel):\n %e \n(%f)(ms)\n", tvg, tvg);
    cutDeleteTimer(timerg_exe);

    double tvga = cutGetTimerValue(timerg_all);
    printf("gpu time glb(kernel+in-copy):\n %e \n(%f)(ms)\n", tvga, tvga);
    cutDeleteTimer(timerg_all);
	*/
	// Stop event
	hipEventRecord(all_stop, 0);
	hipEventSynchronize(all_stop);
	float all_elapsedTime;
	hipEventElapsedTime(&all_elapsedTime, all_start, all_stop); // that's our time!
	// Clean up:
	hipEventDestroy(all_start);
	hipEventDestroy(all_stop);

	printf("gpu time glb(all):\n %e \n(%f)(ms)\n", all_elapsedTime, all_elapsedTime);

//    printf("printing - constraints ...\n");
//	print_constraints_gpu(num_var, sz_a_const, varnum_const, h_constm, is_uniform_var_width);
//    printf("printing - asis ...\n");
//	print_constraints_gpu(num_var, sz_a_const, varnum_asis, h_asis, is_uniform_var_width);

	return;
}

extern "C" void sba_solve_stm(SbaStream* ss_in, int num_var, size_t var_const_max, SbaStream* ss_out_analysis,
		int is_uniform_var_width, int is_equal_const_size)
{
	// memalloc/memcpy for ss_in, ss_outconstraints, ss_out_analysis
	// ss_in : host
	SbaStream* ssg_in; //device
	sbastream_alloc_copy(num_var, var_const_max, ss_in, &ssg_in, hipMemcpyHostToDevice);
	printf("ssg_in\n");

	//  printf("test-----------------------------------------------------------------------------\n");
	//  SbaStream* ssh_in;
	//  sbastream_alloc_copy(num_var, var_const_max, ssg_in, &ssh_in, hipMemcpyDeviceToHost);
	//  print_constraint_stream(num_var, var_const_max, ssh_in, is_uniform_var_width, is_equal_const_size);
	//
	//  printf("alloc constraint/analysis--------------------------------------------------------\n");

	  // last arg 1 means copying at gpu
	  SbaStream* ss_dfields = (SbaStream*)malloc(sizeof(SbaStream));
	  CudaSafeCall(hipMemcpy(ss_dfields, ssg_in, sizeof(SbaStream), hipMemcpyDeviceToHost));

	  int* ssg_cst_num_var;
	  byte** ssg_cst_constnames;
	  byte** ssg_cst_constm;
	  ssg_cst_num_var = init_from_num_const(num_var, 0, GPU);
	  ssg_cst_constnames = init_from_constnames(num_var, ss_dfields->num_const, '\0', GPU);
	  ssg_cst_constm = init_from_constm(num_var, ss_dfields->num_const, var_const_max, '\0', GPU, is_uniform_var_width);

	  int* ssg_anlys_num_var;
	  byte** ssg_anlys_constnames;
	  byte** ssg_anlys_constm;
	  ssg_anlys_num_var = init_from_num_const(num_var, 0, GPU);
	  ssg_anlys_constnames = init_from_constnames(num_var, ss_dfields->num_const, '\0', GPU);
	  ssg_anlys_constm = init_from_constm(num_var, ss_dfields->num_const, var_const_max, '\0', GPU, is_uniform_var_width);

	/* Sbastream from given SbaStream at GPU is not dealt as it is, instead each field are created independently.
	  // ss_out_constraints : host
	  SbaStream* ssg_out_constraints; // ssg_out_constraints
	  sbastream_alloc_copy(num_var, var_const_max, ss_out_constraints, &ssg_out_constraints, hipMemcpyHostToDevice);
	  printf("ssg_out\n");
	  // ss_out_analysis : host
	  SbaStream* ssg_out_analysis;  // device
	  sbastream_alloc_copy(num_var, var_const_max, ss_out_analysis, &ssg_out_analysis, hipMemcpyHostToDevice);
	  printf("ssg_out_analysis\n");
	*/

	  int max_const_size = max_num_constraints(num_var, ss_in->num_const);
	  dim3 threads(num_var, max_const_size);
	  dim3 grid(1);

	  init_constraints_kernel_stm<<< grid, threads >>>(
			  ss_dfields->num_const, ss_dfields->constnames, ss_dfields->constm,
			  num_var, var_const_max,
			  ssg_cst_num_var, ssg_cst_constnames, ssg_cst_constm,
			  ssg_anlys_num_var, ssg_anlys_constnames, ssg_anlys_constm,
			  is_uniform_var_width, is_equal_const_size);


	  printf("-------original const---------\n");
	/*  print_constraint_stream(num_var, var_const_max, ss_in, is_uniform_var_width, is_equal_const_size);


	  SbaStream *ss_reflection = copy_from_SbaStream(num_var, var_const_max, ss_out_constraints);

	  printf("-------reflection---------\n");
	  print_constraint_stream(num_var, var_const_max, ss_reflection, is_uniform_var_width, is_equal_const_size);

	  printf("-------pure const---------\n");
	  print_constraint_stream(num_var, var_const_max, ss_out_constraints, is_uniform_var_width, is_equal_const_size);

	  printf("-------analysis-init--------\n");
	  print_constraint_stream(num_var, var_const_max, ss_out_analysis, is_uniform_var_width, is_equal_const_size);

	  // check if constraint stream is empty.
	  int empty_const = is_constraintstream_empty(num_var, var_const_max, ss_out_constraints);
	  printf("constraint is empty? %d\n", empty_const);

	  int while_iter_count = 0;

	  while(!empty_const)
	  {
	    int i;
	    byte* access_lock = (byte*) malloc (sizeof (byte) * num_var);
	    for(i=0;i<num_var;i++) {
	      access_lock[i] = '\0'; // lock deactivated -> okey to update.
	                             // lock activated if '\1'
	      //printf("lock[%d] = %c\n", i, access_lock[i]);
	    }

	    SbaStream *ss_out_new_constraints = SbaStream_init_empty(num_var);

	    //init_constraints_kernel<<grid, threads>>();
	    printf("%dth Iteration\n------------------------------------------------\nGiven ss_out_costraints:\n", while_iter_count);
	    print_constraint_stream(num_var, var_const_max, ss_out_constraints, is_uniform_var_width, is_equal_const_size);
	    print_constraint_stream(num_var, var_const_max, ss_out_analysis, is_uniform_var_width, is_equal_const_size);

	    //iterate for all var / constraints(var).-> fill up ss_out_new_constraints.
	    for(grid=0;grid < num_var;grid++) {
	      printf("-------------------------------------------\nVarNO:%d, NUMber of constraint:%d, w/ thread:%d\n",
	             grid, ss_out_constraints->num_const[grid], threads);

	      for(threads=0;threads < ss_out_constraints->num_const[grid];threads++) {
	        solve_constraints_kernel(ss_reflection, ss_out_constraints, num_var, access_lock,
	                                 ss_out_new_constraints, ss_out_analysis, grid, threads,
	                                 var_const_max, &empty_const);
	      }
	    }

	    //printf("------------------------------------------------\nNewly updated ss_out_costraints:");
	    //print_constraint_stream(num_var, var_const_max, ss_out_constraints, is_uniform_var_width, is_equal_const_size);

	    //printf("------------------------------------------------\nNewly updated ss_out_analysis:");
	    //print_constraint_stream(num_var, var_const_max, ss_out_analysis, is_uniform_var_width, is_equal_const_size);

	    ss_out_constraints = ss_out_new_constraints;

	    while_iter_count++;
	    empty_const = is_constraintstream_empty(num_var, var_const_max, ss_out_constraints);
	    printf("constraint is empty? or not? %d -> check below:\nFinal constraints:", empty_const);
	    //print_constraint_stream(num_var, var_const_max, ss_out_constraints);
	  }

	  printf("While loop ended:%d\n", empty_const);


	  printf("\nreflection printing\n");
	  print_constraint_stream(num_var, var_const_max, ss_reflection, is_uniform_var_width, is_equal_const_size);

	  //printf("\nout_constraint printing\n");
	  //print_constraint_stream(num_var, var_const_max, ss_out_constraints, is_uniform_var_width, is_equal_const_size);

	  printf("\nout_analysis printing\n");
	  print_constraint_stream(num_var, var_const_max, ss_out_analysis, is_uniform_var_width, is_equal_const_size);

	  printf ("Total iteration of kernel:%d\n", while_iter_count);

	*/
}


extern "C" int ffi_tester (int x)
{
  int i;
  for(i=0;i<x;i++) {
    printf("ffi on testing : %d\n", i);
  }
  return 0;
}



