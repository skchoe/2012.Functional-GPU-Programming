#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gstm_common.h>

const int SYS_MAX_A_CONST = 16; //system define maximum byte of a constraint. (variable's max cap is 4 byte)


// ptr-to constraint names (a byte)
// number-constraints : integer
extern "C" size_t sum_const_sizes(byte* pcnames, int len, size_t var_const_max)
{
  int i;
  size_t acc_constsize_byte = 0;
  for(i=0;i<len;i++) {
    //printf("sum_size(%d) = %d\n", i, acc_constsize_byte);
    acc_constsize_byte += constraint_length (pcnames[i], var_const_max, 1);
  }

  return acc_constsize_byte;
}

// elt is a pointer to new constraint [name][byte1] ... [byteN], where N=var_const_max
extern "C" void ss_add_element(SbaStream* ss, int num_var, int var_no, size_t var_const_max, byte* elt, int is_uniform_var_width)
{
//  printf("-----------------------------------------ss_add_elt, varno:%d, constname:%c, p_ss:%p\n", var_no, elt[0], ss);
//  print_a_constraint(elt, var_const_max, is_uniform_var_width);

  // range check of var_no in num_var variables
  if(num_var <= var_no)
  { 
    printf("varno:%d is larger than variable range [0, %d]---> exit\n", var_no, num_var-1);
    exit(1);
  }

  int i, j;

  // num_const
  int* a_num_const;
  size_t int_num_var = num_var * sizeof(int);
  a_num_const = (int*)malloc(int_num_var);
  for(i=0;i<num_var;i++) {
    if (i==var_no) a_num_const[i] = 1 + ss->num_const[i];
    else a_num_const[i] = ss->num_const[i];
  }

  size_t bytep_num_var = num_var * sizeof(byte*);

//  printf("ss_add_element: num_const done\n");

  // constnames
  byte** a_constnames = (byte**)malloc(bytep_num_var);
  for(i=0;i<num_var;i++) {

    if(ss->constnames[i] != NULL){
      if(i!=var_no) {
        int num_const = ss->num_const[i];
        a_constnames[i] = (byte*)malloc(num_const * sizeof(byte));
        for(j=0;j<num_const;j++) a_constnames[i][j] = ss->constnames[i][j];
      }
      else{
        int num_const = ss->num_const[i] + 1;
        a_constnames[i] = (byte*)malloc(num_const * sizeof(byte));
        for(j=0;j<num_const-1;j++) a_constnames[i][j] = ss->constnames[i][j];
	a_constnames[i][num_const-1] = elt[0];
      }
    }
    else {
      if(i!=var_no) a_constnames[i] = NULL;
      else {
        a_constnames[var_no] = (byte*)malloc(sizeof(byte));
        a_constnames[var_no][0] = elt[0];
      }
    }
  }
//  printf("ss_add_element: constnames done, for const %c\n", elt[0]);
/*
  for(i=0;i<num_var;i++)  {
    printf("numvar:%d, numconst[%d]:%d\n", num_var, i, a_num_const[i]);
    for(j=0;j<a_num_const[i];j++) {
      printf("ptr:%d,   var:%d, constraint:%d -> constname:%c\n", a_constnames, i, j, a_constnames[i][j]);
      printf("first constname in constm[%d][%d] = %c\n", i, j, ss->constnames[i][j]);
    }
  }
*/
  // constm
  size_t const_size = constraint_length(elt[0], var_const_max, 1);
  
  byte** a_constm = (byte**)malloc(bytep_num_var);
  for(i=0;i<num_var;i++) {
    if(ss->constm[i] != NULL) {
      size_t acc_constsize_byte = sum_const_sizes(ss->constnames[i], ss->num_const[i], var_const_max); 

      if(i!=var_no) {
        // just copy from ss->constm[i] -> a_constm[i]
        a_constm[i] = (byte*)malloc(acc_constsize_byte);
        memcpy(a_constm[i], ss->constm[i], acc_constsize_byte);
      }
      else {
        // copy ss->constm[var_no] U elt -> a_costm[var_no]
        a_constm[var_no] = (byte*)malloc(const_size + acc_constsize_byte);
        if(0 < acc_constsize_byte)
          memcpy(a_constm[var_no], ss->constm[var_no], acc_constsize_byte);

        memcpy(a_constm[var_no]+acc_constsize_byte, elt, const_size);
      }
    }
    else {
      if(i!=var_no)  a_constm[i] = NULL;
      else {
        a_constm[var_no] = (byte*)malloc(const_size);
        memcpy(a_constm[var_no], elt, const_size);
      }
    }
  }
  ss->num_const = a_num_const;
  ss->constnames = a_constnames;
  ss->constm = a_constm;

  //printf("________________\n");
  //print_constraint_stream(num_var, var_const_max, ss);
  printf("ss_add_element are all done\n");

  return;
}



//} // end of extern "C"
//



__device__
int constname2numconstvar_gpu(byte c)
{
  switch (c)
  {
    case 'v': return 1;
    case 'b': return 1;
    case 'c': return 2;
    case 'l': return 2;
    case 'P': return 1;
    case 'C': return 1;
    case 'D': return 1;
    case 'B': return 3;
    case 'A': return 2;
    default : return 0;
  }
}
// output : byte of all element in constraint including name (initial byte)
__device__
size_t constraint_length_gpu(byte c, int var_max_byte, int is_uniform_var_width)
{

  int numconstvar = constname2numconstvar_gpu(c);
  if (numconstvar == 0) {
    //printf("In constraint length, charinput(%c), numconstvar=0 ----->exit\n", c);
    //exit(1);
  }

  if(is_uniform_var_width == 0) {
	  int var_part = var_max_byte * numconstvar;
	  return sizeof(byte) + var_part;
  }
  else
	  return var_max_byte * (1 + numconstvar);
}

// var_no const_no -> Xth byte
__device__
int const_pos(byte** constm, int* num_const, int var_no, int const_no, size_t var_const_max)
{
  byte* vconstm = constm[var_no];
  int n_vconst = num_const[var_no];

  // find pointer of const_no th byte.
  int counter = 0;
  //int csizebyte = 0;
  int offset = 0;

  byte* ptr = vconstm;
  for(counter=0;counter < n_vconst;counter++){
    //printf("const_pos - for: nvconst:%d, cnt:%d, const_no:%d\n", n_vconst, counter, const_no);
    if (counter == const_no) return offset;
    else{
      offset += constraint_length_gpu(*ptr, var_const_max, 1);
      ptr += offset;
    }
  }

  return -1;
}

// var_no, num_const, constnames -> Xth byte
__device__
int const_pos_available(byte** constm, int* num_const, byte** constnames, int var_no, size_t var_const_max)
{
  byte* vconstm = constm[var_no];
  int n_vconst = num_const[var_no];

  int counter = 0;
  int offset = 0;

  byte* ptr = vconstm;
  for(counter=0;counter < n_vconst;counter++) {
    offset += constraint_length_gpu(*ptr, var_const_max, 1);
    ptr += offset;
  }

  return offset;
}

__device__
byte* get_constraint_ptr(byte** constm, int var_no, int byte_offset)
{
  byte* vconst = constm[var_no];

  byte* cpos = vconst + byte_offset;
  return cpos;
}

// constraint strm, var_no, const_no -> a pointer to constraint stm
__device__
byte* get_stream_element(byte** constm, int* num_const, int var_no, int const_no, size_t var_const_max)
{
  int offset = const_pos(constm, num_const, var_no, const_no, var_const_max);
  byte* ptr = get_constraint_ptr(constm, var_no, offset);

  return ptr;
}

__device__ int is_heap_constraint(byte cname)
{
  if (cname == 'v'|| cname == 'b' || cname == 'c' || cname == 'l') return 1;
  else return 0;
}

__device__ int is_constraint(byte cname)
{
  if (cname == 'P'|| cname == 'C'|| cname == 'D' || cname == 'B' || cname == 'A') return 1;
  else return 0;
}


__device__ int bytes2int(byte* p, size_t varsize_max)
{
	int i;
	int bs = (int)__powf(2, 8);
	int ans = 0;
	for(i=0;i<varsize_max;i++) {
		ans += p[i] * __powf(bs, varsize_max - 1 - i);
	}

	return ans;
}

// constraint with uniform width
__device__ int get_var_in_const(byte* t_const, int th, size_t sz_a_var) {

	int var_index = th + 1;
	int v = bytes2int(t_const + var_index * sz_a_var, sz_a_var);
	return v;
}

__device__ byte* get_elem(int num_var, size_t sz_a_const, byte* constm, int var, int th) {

	byte* con = constm + sz_a_const * (var + th * num_var);
	return con;
}

// num_var : total number of variables
// sz_a_const(byte): uniform size of each constraint
// g_lock: shared memory for locking access to varnum for variable within a block
// varnum: prospective index of the position for variable(t_var)
// t_var: Xth variable(identifier)
// t_new_const: constraint to insert
__device__ void put_elem(int num_var, size_t sz_a_const, byte* constm, int* g_lock, int* varnum, int t_var, byte*  t_new_const)
{
	int pos_input;
	do {
//		pos_input = (t_var + varnum[t_var] * num_var) * sz_a_const;
//		memcpy(&constm[pos_input], t_new_const, sz_a_const);
//		varnum[t_var]++;
		atomicExch(&g_lock[t_var], 0);
	} while(atomicCAS(&(g_lock[t_var]), 0, 1));

//	while(atomicCAS(&(g_lock[t_var]), 0, 1) != 0);
//	pos_input = (t_var + varnum[t_var] * num_var) * sz_a_const;
//	memcpy(&constm[pos_input], t_new_const, sz_a_const);
//	varnum[t_var]++;
//
////	g_lock[t_var] = 0;
//	atomicExch(&g_lock[t_var], 0);
	return;
}


// num_elt - number of elt in global memory
// elts array of elts
// gsum -storage in global memory for result of each block
/* Blelloch 1990
1:  for d = 0 to log_2^n ��� 1 do ;; d=0,1,2,3 if n=16
2:      for all k = 0 to (n ��� 1) by 2^(d+1) in parallel do (0, 2, 4, 8, as init pos), (1, 2, 4, 8: offsets)
3:           x[k] = x[k] + x[k + 2^d]
d=0 k:0,2,4,... (0,1),(2,3),(4,5),(6,7),(8,9),(10,11)...
d=1 k:0,4,8,... (0,2),      (4,6),      (8,10)
d=2 k:0,8,16,...(0,4),                  (8,16),
d=3 k:16,32,... (0,8),

For 1d block, it works
For 2d blocks, need to get block id considering gridDim.x
*/
__global__ void sum_kernel_int32(int *num_elt, int *elts, int* num_block, int *gsum)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = blockIdx.y;
	int g_id = x + blockDim.x * gridDim.x * y; // position in global memory
	int t_id = threadIdx.x;

	// position of block
	int bidx = blockIdx.x + y * gridDim.x;

	int local_num_elt = (*num_elt > 2 * blockDim.x)? 2 * blockDim.x : *num_elt;
	*num_block = *num_elt / local_num_elt;

	// sum block is out of the range of numblocks
	if(bidx < *num_block) {

		extern __shared__ int shds_int32[];
		// 1 thread, 2 copy to shared memory
		shds_int32[2 * t_id]     = elts[2 * g_id];
		shds_int32[2 * t_id + 1] = elts[2 * g_id + 1];

		int i, from, to, offset = 1;
		for(i = local_num_elt >> 1; i > 0; i >>= 1) {
			__syncthreads();
			if(t_id < i) {
				from = offset * (2*t_id + 1) - 1;
				to   = offset * (2*t_id + 2) - 1;
				shds_int32[to] += shds_int32[from];
			}
			offset *= 2;
		}
		int result = shds_int32[to];
		gsum[bidx] = result;
	}

	__syncthreads();

	return;
}


// blockDim = <block.x, 1>
// gridDim  = <binary_num_var / block.x, 1>
__global__ void init_constraints_kernel_csr(
		int num_var, int max_num_const, int num_tot_const, size_t sz_a_const,
		int* indices, byte* data,
		int* lock_const, int* varnum_const,
		byte* constm,
		int* lock_asis, int* varnum_asis,
		byte* asis,
		int* is_const_empty)
{
	int csr_idx = threadIdx.x + blockDim.x * blockIdx.x;

	//	real valid range.
	if(csr_idx < num_tot_const) {
		int var_id = indices[csr_idx];

		byte* input_constm = (byte*) &(data[csr_idx * sz_a_const]); // coalescing too
		byte constname = input_constm[0];

		int pos_input;
		if(is_constraint(constname)) { // constraint
			*is_const_empty = 0;        // is_const_empty flag is OFF -> constraint found
			do {
				pos_input = ( var_id + varnum_const[var_id] * num_var ) * sz_a_const;
				memcpy(constm + pos_input, input_constm, sz_a_const);
				varnum_const[var_id]++;

				lock_const[var_id] = 0;
			} while(atomicCAS((int*)&(lock_const[var_id]), 0, 1));
		}
		else if(is_heap_constraint(constname)) { // analysis
			do {
				pos_input = ( var_id + varnum_asis[var_id] * num_var ) * sz_a_const;
				memcpy(asis + pos_input, input_constm, sz_a_const);
				varnum_asis[var_id]++;

				lock_asis[var_id] = 0;
			} while(atomicCAS((int*)&(lock_asis[var_id]), 0, 1));
		}
		else {
			// unknown constraint name!
			pos_input = 0;
			int value = 88;
			varnum_const[var_id] -= csr_idx;
			varnum_asis[var_id] -= var_id;
			memset(&constm[var_id + varnum_const[var_id] * num_var], value, sz_a_const);
			memset(&asis[var_id + varnum_asis[var_id] * num_var], value, sz_a_const);
		}
	}

	__syncthreads();
	return;
}

// input original constm, num_const: (var->num const), ??
// split constm into 2 pieces(output): new_constm, analysis
//__global__
__global__ void init_constraints_kernel_stm(
		int* ss_in_cst_num_const, byte** ss_in_cst_constnames,	byte** ss_in_cst_constm,
		int num_var, size_t var_const_max,
		int* ss_out_cst_num_const, byte** ss_out_cst_constnames, byte** ss_out_cst_constm,
		int* ss_out_anlys_num_const, byte** ss_out_anlys_constnames, byte** ss_out_anlys_constm,
		int is_uniform_var_width, int is_euqal_const_size)
{
  // idx: index of which - variable
  // idy: order of constraints
  int var_no   = threadIdx.x;
  int const_no = threadIdx.y;

  int const_count = ss_in_cst_num_const[var_no];

  // skip too big constraint number
  if(const_count < const_no) {
    __syncthreads();
    return;
  }

  int* num_const    = ss_in_cst_num_const;
  byte** constnames = ss_in_cst_constnames;
  byte** constm     = ss_in_cst_constm;

  // ptr-to current constraint
  byte* elt = get_stream_element(constm, num_const, var_no, const_no, var_const_max);

  if(is_heap_constraint (elt[0])) {
/*    printf ("----to heap: %c\n", elt[0]);
    ss_add_element(ss_out_analysis, num_var, var_no, var_const_max, elt);
    //print_constraint_stream(num_var, var_const_max, ss_out_analysis);
    //printf("ptraddr:%d\n", ss_out_analysis->constnames);
*/
  }
  else {
/*
    printf ("----to const: %c\n", elt[0]);
    ss_add_element(ss_out_constraints, num_var, var_no, var_const_max, elt);
    //print_constraint_stream(num_var, var_const_max, ss_out_constraints);
    //printf("ptraddr:%d\n", ss_out_constraints->constnames);
*/
  }

  __syncthreads();


  return;
}

__global__ void solve_constraints_kernel(
				int num_var, int max_num_const, size_t sz_a_const,
				int* g_varnum_refl,  byte* g_reflection,
				int* g_varnum_const,  byte* g_constm,
				int* g_lock_asis, int* g_varnum_asis,  byte* g_asis,
				int* g_lock_new_const, int* g_varnum_new_const, byte* g_new_constm,
				int* sensor, int* g_const_sample)
{
	int block_gidx = blockIdx.x  * blockDim.x;
	int var_id     = threadIdx.x + block_gidx;
	int const_id   = threadIdx.y + block_gidx;
	int asis_id    = threadIdx.z + block_gidx;

	if(var_id < num_var) {
		extern __shared__ int s_lock_asis_nconst[];

		int* p_slock_const = NULL;
		int* p_slock_asis = NULL;

		if(threadIdx.y == 0 && threadIdx.z == 0) { // initialize locks for asis, new_const at s_lock_asis_nconst.
			// 1 thread, 2 copy to shared memory
			s_lock_asis_nconst[var_id]           = g_lock_asis[var_id];
			s_lock_asis_nconst[num_var + var_id] = g_lock_new_const[var_id];

			p_slock_const = s_lock_asis_nconst;
			p_slock_asis = &(s_lock_asis_nconst[num_var]);
		}
		__syncthreads();


		int i;

		size_t sz_a_var = sz_a_const / 4;

		register byte r_new_const_0[SYS_MAX_A_CONST];
		memset(r_new_const_0, NULL, SYS_MAX_A_CONST);

		register byte r_new_const_1[SYS_MAX_A_CONST];
		memset(r_new_const_1, NULL, SYS_MAX_A_CONST);

		// total 16*4 bytes in register = 2 words, 2*16(core) regs < 8K (16, 32)
		// 2*512 (core at high end Fermi) = 1024 < 8K (16, 32)
		register byte r_const[SYS_MAX_A_CONST];
		byte* a_const = get_elem(num_var, sz_a_const, g_constm, var_id, const_id);
		memcpy(r_const, a_const, sz_a_const);

		register byte r_asis [SYS_MAX_A_CONST];
		byte* a_asis = get_elem(num_var, sz_a_const, g_asis, var_id, asis_id);
		memcpy(r_asis, a_asis, sz_a_const);

//		// testing for threads.x
//		if(threadIdx.x == 1 || threadIdx.y == 0 || threadIdx.z == 0) {
//			*sensor = g_varnum_const[var_id];//*sensor * 6;
//		}
//		// testing the access of constraint from register
//		if(threadIdx.x == 4 || threadIdx.y == 0 || threadIdx.z == 0) {
//			//memcpy(g_const_sample, a_const, sz_a_const);
//			memcpy(g_const_sample, &g_asis[threadIdx.x * sz_a_const], sz_a_const);
//		}

		// get const/asis, store to register: r_const, r_asis
		if(const_id < g_varnum_const[var_id] && asis_id < g_varnum_asis[var_id]) {
			int cpos = (var_id + const_id * num_var) * sz_a_const;
			memcpy(r_const, &g_constm[cpos], sz_a_const);

			int apos = (var_id + asis_id * num_var) * sz_a_const;
			memcpy(r_asis, &g_asis[apos], sz_a_const);

			// comparision between constraint and analysis
			byte c_name = r_const[0];
			byte a_name = r_asis[0];

			// testing elements
			if(threadIdx.x == 4 || threadIdx.y == 0 || threadIdx.z == 0)
				memcpy(g_const_sample, &g_asis[asis_id], sz_a_const);

			if(c_name == 'P' && (a_name == 'v' || a_name == 'c' ||  a_name == 'l' ||  a_name == 'b')) {

				memcpy(r_new_const_0, r_asis, sz_a_const);

				int c_var = get_var_in_const(r_const, 0, sz_a_var);

				//testing var value
				*sensor = c_var;

				// put new constraint into a_sis for new variable
				put_elem(num_var, sz_a_const, g_asis, g_lock_asis, g_varnum_asis, c_var, r_new_const_0);
				g_varnum_asis[c_var]++;

				// iterate reflection on r_var, propagate new constraint to the reflection
				for(i=0;i<g_varnum_refl[c_var];i++) {
					byte* refl_const = get_elem(num_var, sz_a_const, g_reflection, c_var, i);
					put_elem(num_var, sz_a_const, g_new_constm, g_lock_new_const, g_varnum_new_const, c_var, refl_const);
					g_varnum_new_const[c_var]++;
				}
			}
			else if(c_name == 'C' && a_name == 'c') {
				r_new_const_0[0] = 'P';
				memcpy(r_new_const_0 + 1 * sz_a_var, r_const + 1 * sz_a_var, sz_a_var); // car_part(1) -> 1 element + 2 empty space
				int c_var = get_var_in_const(r_asis, 0, sz_a_var); // get 0th variable from
				put_elem(num_var, sz_a_const, g_new_constm, g_lock_new_const, g_varnum_new_const, c_var, r_new_const_0);
			}
			else if(c_name == 'D' && a_name == 'c') {
				r_new_const_0[0] = 'P';
				memcpy(r_new_const_0 + 1 * sz_a_var, r_const + 2 * sz_a_var, sz_a_var); // cdr_part(1) -> 1 element + 2 empty space
				int c_var = get_var_in_const(r_asis, 1, sz_a_var); // get 1st variable from
				put_elem(num_var, sz_a_const, g_new_constm, g_lock_new_const, g_varnum_new_const, c_var, r_new_const_0);
			}
			else if(c_name == 'A' && a_name == 'l') {
				r_new_const_0[0] = 'P';
				memcpy(r_new_const_0 + 1 * sz_a_var, r_const + 2 * sz_a_var, sz_a_var); // (app, result, arg) -> (prop2 result)
				int r_var_0 = get_var_in_const(r_asis, 1, sz_a_var);
				put_elem(num_var, sz_a_const, g_new_constm, g_lock_new_const, g_varnum_new_const, r_var_0, r_new_const_0);

				r_new_const_1[0] = 'P';
				memcpy(r_new_const_1 + sz_a_var, r_const + 1 * sz_a_var, sz_a_var);
				int r_var_1 = get_var_in_const(r_const, 1, sz_a_var);
				put_elem(num_var, sz_a_const, g_new_constm, g_lock_new_const, g_varnum_new_const, r_var_1, r_new_const_1);
			}
			else if(c_name == 'A' && a_name == 't') { // continuation is not supported
				r_new_const_0[0] = 'P';
				memcpy(r_new_const_0 + 1 * sz_a_var, r_const + 1 * sz_a_var, sz_a_var); // (app, result, arg) -> (prop2 result)
				int c_var = get_var_in_const(r_const, 1, sz_a_var);
				put_elem(num_var, sz_a_const, g_new_constm, g_lock_new_const, g_varnum_new_const, c_var, r_new_const_0);
			}
			else if(c_name == 'B' && a_name == 'v') {
				int var_value = get_var_in_const(r_asis, 0, sz_a_var);
				int var_test = get_var_in_const(r_const, 0, sz_a_var);
				int b_null;
				if(var_value == 0) b_null = 1;
				else b_null = 0;

				if(b_null == var_test){
					r_new_const_0[0] = 'P';
					memcpy(r_new_const_0 + 1 * sz_a_var, r_const + 3 * sz_a_var, sz_a_var); // (app, result, arg) -> (prop2 result)
					int c_var = get_var_in_const(r_const, 1, sz_a_var);
					put_elem(num_var, sz_a_const, g_new_constm, g_lock_new_const, g_varnum_new_const, c_var, r_new_const_0);
				}
			}
			else {


			}
		}
	}

	__syncthreads();

	return;
}
