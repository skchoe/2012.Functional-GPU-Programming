#include "hip/hip_runtime.h"
/*
 * main.c
 *
 *  Created on: Mar 15, 2012
 *      Author: u0332192
 */

#include <hip/hip_runtime.h>
#include <cutil.h>

#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <hip/device_functions.h>
#include <hip/hip_vector_types.h>


typedef unsigned char byte;
typedef unsigned long long ullong;

int64_t timespecDiffNano(struct timespec *timeA_p, struct timespec *timeB_p)
{
	return ((timeA_p->tv_sec * 1000000000) + timeA_p->tv_nsec) -
			((timeB_p->tv_sec * 1000000000) + timeB_p->tv_nsec);
}

// return pointer to array of x y z.
int* max_grid()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0); // 0=dev single device
	return deviceProp.maxGridSize;
}


__global__ void test (ullong* num_elt, ullong *elts, ullong* gsum)
{
	int g_id = threadIdx.x + blockDim.x * blockIdx.x;
	int b_id = threadIdx.x;

	int warp_size = 16;
	int i, n = blockDim.x / warp_size;

	*gsum = 155;

	__syncthreads();

	return;
}

// num_elt - number of elt in global memory
// elts array of elts
// gsum -storage in global memory for result of each block
/* Blelloch 1990
1:  for d = 0 to log_2^n – 1 do ;; d=0,1,2,3 if n=16
2:      for all k = 0 to (n – 1) by 2^(d+1) in parallel do (0, 2, 4, 8, as init pos), (1, 2, 4, 8: offsets)
3:           x[k] = x[k] + x[k + 2^d]
d=0 k:0,2,4,... (0,1),(2,3),(4,5),(6,7),(8,9),(10,11)...
d=1 k:0,4,8,... (0,2),      (4,6),      (8,10)
d=2 k:0,8,16,...(0,4),                  (8,16),
d=3 k:16,32,... (0,8),

For 1d block, it works
For 2d blocks, need to get block id considering gridDim.x
*/
__global__ void sum_kernel_ullong(ullong *num_elt, ullong *elts, ullong* num_block, ullong *gsum)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = blockIdx.y;
	int g_id = x + blockDim.x * gridDim.x * y; // position in global memory
	int t_id = threadIdx.x;

	// position of block
	int bidx = blockIdx.x + y * gridDim.x;

	ullong local_num_elt = (*num_elt > 2 * blockDim.x)? 2 * blockDim.x : *num_elt;
	*num_block = *num_elt / local_num_elt;

	// sum block is out of the range of numblocks
	if(bidx < *num_block) {

		extern __shared__ ullong shds_ullong[];
		// 1 thread, 2 copy to shared memory
		shds_ullong[2 * t_id]     = elts[2 * g_id];
		shds_ullong[2 * t_id + 1] = elts[2 * g_id + 1];

		int i, from, to, offset = 1;
		for(i = local_num_elt >> 1; i > 0; i >>= 1) {
			__syncthreads();
			if(t_id < i) {
				from = offset * (2*t_id + 1) - 1;
				to   = offset * (2*t_id + 2) - 1;
				shds_ullong[to] += shds_ullong[from];
			}
			offset *= 2;
		}
		ullong result = shds_ullong[to];
		gsum[bidx] = result;
	}

	__syncthreads();

	return;
}

__global__ void sum_kernel_int32(int *num_elt, int *elts, int* num_block, int *gsum)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = blockIdx.y;
	int g_id = x + blockDim.x * gridDim.x * y; // position in global memory
	int t_id = threadIdx.x;

	// position of block
	int bidx = blockIdx.x + y * gridDim.x;

	int local_num_elt = (*num_elt > 2 * blockDim.x)? 2 * blockDim.x : *num_elt;
	*num_block = *num_elt / local_num_elt;

	// sum block is out of the range of numblocks
	if(bidx < *num_block) {

		extern __shared__ int shds_int32[];
		// 1 thread, 2 copy to shared memory
		shds_int32[2 * t_id]     = elts[2 * g_id];
		shds_int32[2 * t_id + 1] = elts[2 * g_id + 1];

		int i, from, to, offset = 1;
		for(i = local_num_elt >> 1; i > 0; i >>= 1) {
			__syncthreads();
			if(t_id < i) {
				from = offset * (2*t_id + 1) - 1;
				to   = offset * (2*t_id + 2) - 1;
				shds_int32[to] += shds_int32[from];
			}
			offset *= 2;
		}
		int result = shds_int32[to];
		gsum[bidx] = result;
	}

	__syncthreads();

	return;
}

// num_elt : length of input array in cpu
// gnum_elt: pointer to length of input arary in gpu
// g_elts: pointer to the array in GPU
// output: pointer to array that contains the sum at the first location in GPU
ullong*
sum_gpu_kernel_ullong(ullong num_elt, ullong* gnum_elt, ullong* gelts)
{
	size_t sz_ull = sizeof(ullong);
	size_t sz_elts = sz_ull * num_elt;

	int i;
	ullong* gnum_block;
	hipMalloc(&gnum_block, sz_ull);
	hipMemset(gnum_block, 0, sz_ull);

    dim3 block1 = dim3 (min((double)512, (double)num_elt/2.0), 1); // 512 threads deals with 1024 data.
    int num_blk;
    num_blk = (num_elt > block1.x)? num_elt / (2 * block1.x): 1; // because each thread process 2 elements

	// return storage gsum
	size_t sz_gsum = sz_ull * num_blk;
	ullong *gsum;
	hipMalloc(&gsum, sz_gsum);
	hipMemset(gsum, 0, sz_gsum);

    unsigned int timerg_exe = 0;
    cutCreateTimer(&timerg_exe);
    cutResetTimer(timerg_exe);

	cutStartTimer(timerg_exe);

	///////////////////////////////////////////////////////////////////////////////////////////////////////////
    do {
//    	printf("--------------do-------------\nnumelt:%d\n", num_elt);
//    	ullong* tmpelt = (ullong*)malloc(sz_elts);
//    	hipMemcpy(tmpelt, gelts, sz_elts, hipMemcpyDeviceToHost);
//    	for(i=0;i<num_elt;i++)
//    		printf("tmpelt[%d] = %llu\n", i, tmpelt[i]);

        block1 = dim3 (min((double)512, (double)num_elt/2.0), 1); // 512 threads deals with 1024 data.
        num_blk = (num_elt > block1.x)? num_elt / (2 * block1.x): 1; // because each thread process 2 elements

        int* mxgrd = max_grid();
        printf("grid max = %d, %d blocks\n", mxgrd[0], mxgrd[1]);

        int gridx = (num_blk > mxgrd[0])? mxgrd[0] : num_blk;
        int gridy = (num_blk > mxgrd[0])? num_blk / mxgrd[0] + 1: 1;
        printf("num_blk:%d\n", num_blk);

        printf("grid dimension x, y = %d, %d\n", gridx, gridy);


        dim3 grid1 = dim3(gridx, gridy);
        size_t shds1 = 2 * block1.x * sizeof(ullong); // need factor 2 because a thread add 2 elements.

        printf("blockdim.x = %d threads, shd size = %d bytes\n", block1.x, shds1);

        // clear used location
		size_t sz_gsum = sz_ull * num_blk;

		// new grid, block, shds
		sum_kernel_ullong<<<grid1, block1, shds1>>>(gnum_elt, gelts, gnum_block, gsum);

		num_elt = num_blk;
		sz_elts = sz_gsum;

		// interchange:
		ullong* tmp_num_elt = gnum_elt;
		ullong* tmp_elts = gelts;

		gnum_elt = gnum_block;
		gelts = gsum;

		gnum_block = tmp_num_elt;
		gsum = tmp_elts;

//		// copy output by printing next inputs
//		ullong* cnum_elt = (ullong*)malloc(sz_ull);
//		hipMemcpy(cnum_elt, gnum_elt, sz_ull, hipMemcpyDeviceToHost);
//		printf("next - numelt:%d\n", *cnum_elt);

//		int i;
//		ullong* celts = (ullong*)malloc(sz_elts);
//		hipMemcpy(celts, gelts, sz_elts, hipMemcpyDeviceToHost);
//		for(i=0;i<(int)*cnum_elt;i++)
//			printf("%d th next elt:%llu\n", i, celts[i]);

	} while (num_blk != 1);
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////

    cutStopTimer(timerg_exe);
    double tvg = cutGetTimerValue(timerg_exe);
    printf("gpu time glb(kernel):\n %e \n(%f)(ms)\n", tvg, tvg);

    cutDeleteTimer(timerg_exe);

	return gelts;
}

int*
sum_gpu_kernel_int32(int num_elt, int* gnum_elt, int* gelts)
{
	size_t sz_ull = sizeof(int);
	size_t sz_elts = sz_ull * num_elt;

	int i;
	int* gnum_block;
	hipMalloc(&gnum_block, sz_ull);
	hipMemset(gnum_block, 0, sz_ull);

    dim3 block1 = dim3 (min((double)512, (double)num_elt/2.0), 1); // 512 threads deals with 1024 data.
    int num_blk;
    num_blk = (num_elt > block1.x)? num_elt / (2 * block1.x): 1; // because each thread process 2 elements

	// return storage gsum
	size_t sz_gsum = sz_ull * num_blk;
	int *gsum;
	hipMalloc(&gsum, sz_gsum);
	hipMemset(gsum, 0, sz_gsum);

    unsigned int timerg_exe = 0;
    cutCreateTimer(&timerg_exe);
    cutResetTimer(timerg_exe);

	cutStartTimer(timerg_exe);

	///////////////////////////////////////////////////////////////////////////////////////////////////////////
    do {
//    	printf("--------------do-------------\nnumelt:%d\n", num_elt);
//    	int* tmpelt = (int*)malloc(sz_elts);
//    	hipMemcpy(tmpelt, gelts, sz_elts, hipMemcpyDeviceToHost);
//    	for(i=0;i<num_elt;i++)
//    		printf("tmpelt[%d] = %llu\n", i, tmpelt[i]);

        block1 = dim3 (min((double)512, (double)num_elt/2.0), 1); // 512 threads deals with 1024 data.
        num_blk = (num_elt > block1.x)? num_elt / (2 * block1.x): 1; // because each thread process 2 elements

        int* mxgrd = max_grid();
        printf("grid max = %d, %d blocks\n", mxgrd[0], mxgrd[1]);

        int gridx = (num_blk > mxgrd[0])? mxgrd[0] : num_blk;
        int gridy = (num_blk > mxgrd[0])? num_blk / mxgrd[0] + 1: 1;
        printf("num_blk:%d\n", num_blk);

        printf("grid dimension x, y = %d, %d\n", gridx, gridy);


        dim3 grid1 = dim3(gridx, gridy);
        size_t shds1 = 2 * block1.x * sizeof(int); // need factor 2 because a thread add 2 elements.

        printf("blockdim.x = %d threads, shd size = %d bytes\n", block1.x, shds1);

        // clear used location
		size_t sz_gsum = sz_ull * num_blk;

		// new grid, block, shds
		sum_kernel_int32<<<grid1, block1, shds1>>>(gnum_elt, gelts, gnum_block, gsum);

		num_elt = num_blk;
		sz_elts = sz_gsum;

		// interchange:
		int* tmp_num_elt = gnum_elt;
		int* tmp_elts = gelts;

		gnum_elt = gnum_block;
		gelts = gsum;

		gnum_block = tmp_num_elt;
		gsum = tmp_elts;

//		// copy output by printing next inputs
//		int* cnum_elt = (int*)malloc(sz_ull);
//		hipMemcpy(cnum_elt, gnum_elt, sz_ull, hipMemcpyDeviceToHost);
//		printf("next - numelt:%d\n", *cnum_elt);

//		int i;
//		int* celts = (int*)malloc(sz_elts);
//		hipMemcpy(celts, gelts, sz_elts, hipMemcpyDeviceToHost);
//		for(i=0;i<(int)*cnum_elt;i++)
//			printf("%d th next elt:%llu\n", i, celts[i]);

	} while (num_blk != 1);
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////

    cutStopTimer(timerg_exe);
    double tvg = cutGetTimerValue(timerg_exe);
    printf("gpu time glb(kernel):\n %e \n(%f)(ms)\n", tvg, tvg);

    cutDeleteTimer(timerg_exe);

	return gelts;
}


ullong
sum_gpu_interchange_ullong(ullong num_elt, ullong* elts)
{
	size_t sz_ull = sizeof(ullong);
	size_t sz_elts = sz_ull * num_elt;

	printf("sizeof globalmem alloc:%llu\n", sz_elts);

	ullong *gnum_elt;
    hipMalloc(&gnum_elt, sz_ull);
    hipMemcpy((void*)gnum_elt, &num_elt, sz_ull, hipMemcpyHostToDevice);

    ullong *gelts;
    hipMalloc(&gelts, sz_elts);
    hipMemcpy(gelts, elts, sz_elts, hipMemcpyHostToDevice);

    ullong* gelts_out = sum_gpu_kernel_ullong(num_elt, gnum_elt, gelts);

	ullong answer = 0;

	hipMemcpy(&answer, gelts_out, sz_ull, hipMemcpyDeviceToHost);
	printf("answer in sum_gpu():%llu\n", answer);

    hipFree(gelts);

	return answer;
}

int
sum_gpu_interchange_int32(int num_elt, int* elts)
{
	size_t sz_ull = sizeof(int);
	size_t sz_elts = sz_ull * num_elt;

	printf("sizeof globalmem alloc:%llu\n", sz_elts);

	int *gnum_elt;
    hipMalloc(&gnum_elt, sz_ull);
    hipMemcpy((void*)gnum_elt, &num_elt, sz_ull, hipMemcpyHostToDevice);

    int *gelts;
    hipMalloc(&gelts, sz_elts);
    hipMemcpy(gelts, elts, sz_elts, hipMemcpyHostToDevice);

    int* gelts_out = sum_gpu_kernel_int32(num_elt, gnum_elt, gelts);

	int answer = 0;

	hipMemcpy(&answer, gelts_out, sz_ull, hipMemcpyDeviceToHost);
	printf("answer in sum_gpu():%llu\n", answer);

    hipFree(gelts);

	return answer;
}

ullong
sum_gpu_alloc_ullong(ullong num_elt, ullong* elts)
{
	size_t sz_ull = sizeof(ullong);
	size_t sz_elts = sz_ull * num_elt;

	ullong *gnum_elt;
    hipMalloc(&gnum_elt, sz_ull);
    hipMemcpy((void*)gnum_elt, &num_elt, sz_ull, hipMemcpyHostToDevice);

    ullong *gelts;
    hipMalloc(&gelts, sz_elts);
    hipMemcpy(gelts, elts, sz_elts, hipMemcpyHostToDevice);

    unsigned int timerg_exe = 0;
    cutCreateTimer(&timerg_exe);
    cutResetTimer(timerg_exe);

	cutStartTimer(timerg_exe);

	int i;
    int num_blk;
	ullong* gnum_block;
	ullong *gsum;
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
    do {

//    	printf("--------------do-------------\nnumelt:%d\n", num_elt);
//    	ullong* tmpelt = (ullong*)malloc(sz_elts);
//    	hipMemcpy(tmpelt, gelts, sz_elts, hipMemcpyDeviceToHost);
//    	for(i=0;i<num_elt;i++)
//    		printf("tmpelt[%d] = %llu\n", i, tmpelt[i]);

        dim3 block1 = dim3 (min((double)512, (double)num_elt/2.0), 1); // 512 threads deals with 1024 data.
        num_blk = (num_elt > block1.x)? num_elt / (2 * block1.x): 1; // because each thread process 2 elements

        int* mxgrd = max_grid();
//        printf("grid max = %d, %d blocks\n", mxgrd[0], mxgrd[1]);

        int gridx = (num_blk > mxgrd[0])? mxgrd[0] : num_blk;
        int gridy = (num_blk > mxgrd[0])? num_blk / mxgrd[0] + 1: 1;
//        printf("num_blk:%d\n", num_blk);

//        printf("grid dimension x, y = %d, %d\n", gridx, gridy);


        dim3 grid1 = dim3(gridx, gridy);
        size_t shds1 = 2 * block1.x * sizeof(ullong); // need factor 2 because a thread add 2 elements.

//        printf("blockdim.x = %d threads, shd size = %d bytes\n", block1.x, shds1);

		hipMalloc(&gnum_block, sz_ull);
		hipMemset(gnum_block, 0, sz_ull);

		// return storage gsum
		size_t sz_gsum = sz_ull * num_blk;
		hipMalloc(&gsum, sz_gsum);
		hipMemset(gsum, 0, sz_gsum);


		// new grid, block, shds
		sum_kernel_ullong<<<grid1, block1, shds1>>>(gnum_elt, gelts, gnum_block, gsum);

// hipFree() costs 90ms - huge.
//	    hipFree(gnum_elt);
//		hipFree(gelts);

		num_elt = num_blk;
		sz_elts = sz_gsum;

		gnum_elt = gnum_block;
		gelts = gsum;

	} while (num_blk != 1);
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////

    cutStopTimer(timerg_exe);
    double tvg = cutGetTimerValue(timerg_exe);
    printf("gpu time glb(kernel):\n %e \n(%f)(ms)\n", tvg, tvg);

    cutDeleteTimer(timerg_exe);

	ullong answer = 0;

	hipMemcpy(&answer, gsum, sz_ull, hipMemcpyDeviceToHost);
	printf("answer in sum_gpu():%llu\n", answer);

    hipFree(gelts);
    hipFree(gsum);

	return answer;
}

ullong sum_gpu_ullong(ullong num_elt, ullong* elts)
{
	ullong g_sum;
//	g_sum = sum_gpu_alloc_ullong(num_elt, elts);
	g_sum = sum_gpu_interchange_ullong(num_elt, elts);
	return g_sum;
}

int sum_gpu_int32(int num_elt, int* elts)
{
	int g_sum;
	g_sum = sum_gpu_interchange_int32(num_elt, elts);
	return g_sum;
}

void sum_uulong()
{
	int i;
	// data prep

	// cuda call
//	ullong num_elt = (ullong)pow(2, 1); 		// test for single block
//	ullong num_elt = (ullong)pow(2, 9); 		// test for single block
//	ullong num_elt = (ullong)pow(2, 11); 	//2048 = 1024 + 1024 : 2 blocks
//	ullong num_elt = (ullong)pow(2, 29); // 4GB (too big at global memory)
//	ullong num_elt = (ullong)pow(2, 27); // 1.07 GB  (too big at global memory) -> wrong answer
	ullong num_elt = (ullong)pow(2, 26); // 536 MB - gpu faster than cpu : twice (primary data)
//	ullong num_elt = (ullong)pow(2, 25); // cpu gpu(only gpu side - no hipFree()) similar .1XX ms
//	ullong num_elt = (ullong)pow(2, 24); // cpu faster than gpu 2 times

	size_t sz_elts = sizeof(ullong) * num_elt;
	printf("input data size of length %llu, w/ long long type:%llu bytes\n", num_elt, sz_elts);


	// malloc elts in either normal memory or pinned memory. => pinned memory is 2 times faster in memcpy
//	ullong* elts = (ullong *)malloc(sz_elts);
	ullong *elts;
	hipHostMalloc(&elts, sz_elts);
	for(i=0;i<num_elt;i++) {
		elts[i] = (ullong)i;//(int)rand();
//		printf("%llu th input = %llu\n", i, elts[i]);
	}
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//	// begin cpu time check
	timespec ts_start, ts_stop;
	printf("CLOCk:.....%d\n", CLOCK_MONOTONIC + 2);
	if(0==clock_gettime(CLOCK_MONOTONIC, &ts_start))
		printf("taking clock - start successful\n");

	ullong sum=0;
	for(i=0;i<num_elt;i++)
		sum += elts[i];

	if(0==clock_gettime(CLOCK_MONOTONIC, &ts_stop))
		printf("taking clock - stop successful\n");
	uint64_t timeElapsedNano = timespecDiffNano(&ts_stop, &ts_start);

	printf("cpu - ptrptr (s):(%u->%u)\ncpu(ns):(%u->%u)\n cpu_elapsed(ms):%f \n\n",
			ts_start.tv_sec, ts_stop.tv_sec,
			ts_start.tv_nsec, ts_stop.tv_nsec, (double)timeElapsedNano/(double)1000000000);

	printf("numelt=%d, sum in cpu = %llu\n\n", num_elt, sum);

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//begin gpu timer
    unsigned int timerg_all = 0;
    cutCreateTimer(&timerg_all);
    cutResetTimer(timerg_all);

    cutStartTimer(timerg_all);

    ullong g_sum = 0;
    g_sum = sum_gpu_ullong(num_elt, elts);

    cutStopTimer(timerg_all);

    printf("dbl_sum = %llu, diff(c-g):%llu\n", g_sum, sum - g_sum);

    double tvga = cutGetTimerValue(timerg_all);
    printf("gpu time glb(kernel+in-copy):\n %e \n(%f)(ms)\n", tvga, tvga);
    cutDeleteTimer(timerg_all);
	return;
}

void sum_int32()
{
	int i;
	// data prep

	// cuda call
//	int num_elt = (int)pow(2, 1); 		// test for single block
//	int num_elt = (int)pow(2, 9); 		// test for single block
	int num_elt = (int)pow(2, 14); 	    // 2^10 * 2^4 : 2^4 blocks

	size_t sz_elts = sizeof(int) * num_elt;
	printf("input data size of length %llu, w/ long long type:%llu bytes\n", num_elt, sz_elts);


	// malloc elts in either normal memory or pinned memory. => pinned memory is 2 times faster in memcpy
//	int* elts = (int *)malloc(sz_elts);
	int *elts;
	hipHostMalloc(&elts, sz_elts);
	for(i=0;i<num_elt;i++) {
		elts[i] = (int)i;//(int)rand();
//		printf("%llu th input = %llu\n", i, elts[i]);
	}
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//	// begin cpu time check
	timespec ts_start, ts_stop;
	printf("CLOCk:.....%d\n", CLOCK_MONOTONIC + 2);
	if(0==clock_gettime(CLOCK_MONOTONIC, &ts_start))
		printf("taking clock - start successful\n");

	int sum=0;
	for(i=0;i<num_elt;i++)
		sum += elts[i];

	if(0==clock_gettime(CLOCK_MONOTONIC, &ts_stop))
		printf("taking clock - stop successful\n");
	uint64_t timeElapsedNano = timespecDiffNano(&ts_stop, &ts_start);

	printf("cpu - ptrptr (s):(%u->%u)\ncpu(ns):(%u->%u)\n cpu_elapsed(ms):%f \n\n",
			ts_start.tv_sec, ts_stop.tv_sec,
			ts_start.tv_nsec, ts_stop.tv_nsec, (double)timeElapsedNano/(double)1000000000);

	printf("numelt=%d, sum in cpu = %llu\n\n", num_elt, sum);

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//begin gpu timer
    unsigned int timerg_all = 0;
    cutCreateTimer(&timerg_all);
    cutResetTimer(timerg_all);

    cutStartTimer(timerg_all);

    int g_sum = 0;
    g_sum = sum_gpu_int32(num_elt, elts);

    cutStopTimer(timerg_all);

    printf("dbl_sum = %llu, diff(c-g):%llu\n", g_sum, sum - g_sum);

    double tvga = cutGetTimerValue(timerg_all);
    printf("gpu time glb(kernel+in-copy):\n %e \n(%f)(ms)\n", tvga, tvga);
    cutDeleteTimer(timerg_all);
	return;
}

int main(int argc, char** argv)
{
//	sum_uulong();
	sum_int32();
	return 0;
}
