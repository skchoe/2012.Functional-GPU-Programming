#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Feb 24, 2012
 *      Author: skchoe
 */


#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <sm_11_atomic_functions.h>
#include <hip/hip_vector_types.h>



int sharedMemory()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0); // 0=dev single device
	return deviceProp.sharedMemPerBlock;
}
int maxThreadsPerBlock()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0); // 0=dev single device
	return deviceProp.maxThreadsPerBlock;
}


void split2constanalys_seq(int num_var, int* num_const, int max_num_const,
		int* pinput, int* poutput1, int* poutput2)
{
	int i, j, k;
	for(i=0;i<max_num_const;i++) // height
		for(j=0;j<num_var;j++) {
			k = j + i * max_num_const;
			if(i < num_const[j])
				if(pinput[k] == 1)
					poutput1[k] = 1;
				else if(pinput[k] == 2)
					poutput2[k] = 2;
				else // pinput[k] == 0
				{}
		}

	return;
}

__global__ void split2constanalys_gpumat(
		int *num_var, int* num_const, int *max_num_const,
		int* pginput, int* pgoutput1, int* pgoutput2)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = j + i * *max_num_const;
	if(i < num_const[j]) {
		if(pginput[k] == 1)
			pgoutput1[k] = 1;
		else if(pginput[k] == 2)
			pgoutput2[k] = 2;
		else // pinput[k] == 0
		{}
	}

	return;
}

__global__ void split2constanalys_gpusp(
		int* num_var, int* num_const, int* max_num_const,
		int* psginput,
		int* num_outconst1, int* psgoutput1,
		int* num_outconst2, int* psgoutput2)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;

	__shared__ int s_mat[10000];

	__syncthreads();

	int k = j + i * *max_num_const;
	if(i < num_const[j]) {
		if(psginput[k] == 1)
			psgoutput1[k] = 1;
		else if(psginput[k] == 2)
			psgoutput2[k] = 2;
		else // pinput[k] == 0
		{}
	}

	__syncthreads();



	return;
}


void print_constanalys_mat (int num_var, int max_num_const, int* poutput1, int* poutput2)
{

	int i, j, k;
	printf("array of 1's\n");
	for(i=0;i<max_num_const;i++) {
		for(j=0;j<num_var;j++) {
			k = j + i*max_num_const;
			printf(" %d ", poutput1[k]);
		}
		printf("\n");
	}

	printf("array of 2's\n");
	for(i=0;i<max_num_const;i++) {
		for(j=0;j<num_var;j++) {
			k = j + i*max_num_const;
			printf(" %d ", poutput2[k]);
		}
		printf("\n");
	}
}

void init_int_array(int* pint, int x, int num_ints)
{
	int i;
	for(i=0;i<num_ints;i++)
		pint[i] = x;
	return;
}


int main(int argc, char** argv)
{
	int i, j, k;

	int num_var = (int) pow(2, 8 * 2);//16; // W x W is maximum allowable number of threads.
	int num_const_limit = 16;
//	int mtpb = maxThreadsPerBlock();
//	int Hthreads = min(H, mtpb/W);
	size_t sz1 = sizeof(int) * num_var;

	// list of num_const for each var.
	int* num_const = (int*) malloc(sz1);
	for(i=0;i<num_var;i++)
		num_const[i] = (int) (rand() % (num_const_limit + 1));

	int sum_const = 0;
	int max_num_const = 0;
	for(i=0;i<num_var;i++) {
		if(num_const[i] > max_num_const) max_num_const = num_const[i];
		sum_const += num_const[i];
	}

	printf("num_var: %d, max num const=%d, number of const:%d\n",
			num_var, max_num_const, sum_const);

	///////////////////////////////////////////////////
	// representation
	// original 2D matrix
	size_t sz2 = sizeof(int) * num_var * max_num_const;
	int* pinput = (int*) malloc (sz2);
	for(i=0;i<max_num_const;i++) // height
		for(j=0;j<num_var;j++) {
			k = j + i * max_num_const;
			if(i < num_const[j])
				pinput[k] = rand() % 2 + 1;
			else
				pinput[k] = 0;
		}
//	// test printing
//	for(i=0;i<max_num_const;i++) {
//		for(j=0;j<num_var;j++) {
//			k = j + i*max_num_const;
//			printf(" %d ", pinput[k]);
//		}
//		printf("\n");
//	}

	int* poutput1 = (int*) malloc (sz2);

	int num_int = num_var * max_num_const;
	init_int_array(poutput1, 0, num_int);
	int* poutput2 = (int*) malloc (sz2);
	init_int_array(poutput2, 0, num_int);

	clock_t tv_start = clock();

	split2constanalys_seq(num_var, num_const, max_num_const, pinput, poutput1, poutput2);

	clock_t tv_stop = clock();
	double dateclock_diff_ms = (double)(tv_stop - tv_start) * 1000. / CLOCKS_PER_SEC;

	printf("elapsed time for cpu:%e (%f)(ms)\n", dateclock_diff_ms, dateclock_diff_ms);

//	print_constanalys_mat (num_var, max_num_const, poutput1, poutput2);

	///////////////////////////////////////////////////////////////////////////////

	// gpu handling of the 2d matrix
	int* gnum_var;
	hipMalloc(&gnum_var, sizeof(int));
	hipMemcpy(gnum_var, &num_var, sizeof(int), hipMemcpyHostToDevice);

	int* gnum_const;
	hipMalloc(&gnum_const, sz1);
	hipMemcpy(gnum_const, num_const, sz1, hipMemcpyHostToDevice);

	int* gmax_num_const;
	hipMalloc(&gmax_num_const, sizeof(int));
	hipMemcpy(gmax_num_const, &max_num_const, sizeof(int), hipMemcpyHostToDevice);

	// data storage
	int *pginput, *pgoutput1, *pgoutput2;
	hipMalloc(&pginput, sz2);
	hipMemcpy(pginput, pinput, sz2, hipMemcpyHostToDevice);

    hipMalloc(&pgoutput1, sz2);
	hipMemcpy(pgoutput1, poutput1, sz2, hipMemcpyHostToDevice);
    hipMalloc(&pgoutput2, sz2);
	hipMemcpy(pgoutput2, poutput2, sz2, hipMemcpyHostToDevice);

	int block_w = 16,  block_h = max_num_const;
	printf("block:(%d,%d), grid(%d, %d)\n", block_w, block_h, num_var / block_w, max_num_const/block_h);

    dim3 grid(num_var / block_w, max_num_const/block_h, 1);
    dim3 block(block_w, block_h, 1);

    unsigned int timerg1 = 0;
    cutCreateTimer(&timerg1);
    cutResetTimer(timerg1);
	cutStartTimer(timerg1);

    split2constanalys_gpumat<<<grid, block>>>(gnum_var, gnum_const, gmax_num_const, pginput, pgoutput1, pgoutput2);

    cutStopTimer(timerg1);

    double tvg1 = cutGetTimerValue(timerg1);
    printf("gpu time glb(-num_var:%d): %e (%f)(ms)\n", num_var, tvg1, tvg1);
    cutDeleteTimer(timerg1);


    printf("diff: cpu-gpu:%e, percentage gpu/cpu:%f\n",
    		dateclock_diff_ms - timerg1,
    		(double)timerg1 / (double)dateclock_diff_ms);

    init_int_array(poutput1, 0, num_var * max_num_const);
    init_int_array(poutput2, 0, num_var * max_num_const);
    hipMemcpy(poutput1, pgoutput1, sz2, hipMemcpyDeviceToHost);
    hipMemcpy(poutput2, pgoutput2, sz2, hipMemcpyDeviceToHost);

//	print_constanalys_mat (num_var, max_num_const, poutput1, poutput2);

	//////////////////////////////////////////////////////////////////////////////////////
	// sparse form of the input
	int sz3 = sizeof(int) * sum_const;
	int* psinput = (int*) malloc (sz3);
	int l=0;
	for(i=0;i<max_num_const;i++) // height
		for(j=0;j<num_var;j++) {
			k = j + i * max_num_const;
			if(i < num_const[j])
				psinput[l++] = pinput[k];
		}

//	// checking above.
//	for(j=0;j<sum_const;j++)
//		printf("j:%d (%d)\t", j, psinput[j]);
//	printf("\n");

	int* psginput;
	hipMalloc(&psginput, sz3);
	hipMemcpy(psginput, psinput, sz3, hipMemcpyHostToDevice);

	int* num_outconst1 = (int*) malloc(sz1);
	int* num_outconst2 = (int*) malloc(sz1);
	for(i=0;i<num_var;i++) {
		num_outconst1[i] = 0;
		num_outconst2[i] = 0;
	}

	int* gnum_outconst1, *gnum_outconst2;
	hipMalloc(&gnum_outconst1, sz1);
	hipMemcpy(gnum_outconst1, num_outconst1, sz1, hipMemcpyHostToDevice);
	hipMalloc(&gnum_outconst2, sz1);
	hipMemcpy(gnum_outconst2, num_outconst2, sz1, hipMemcpyHostToDevice);

	int *psgoutput1, *psgoutput2;
	hipMalloc(&psgoutput1, sz3);
	hipMalloc(&psgoutput2, sz3);

    unsigned int timerg2 = 0;
    cutCreateTimer(&timerg2);
    cutResetTimer(timerg2);
	cutStartTimer(timerg2);

	split2constanalys_gpusp<<<grid, block>>>(gnum_var, gnum_const, gmax_num_const,
			                                 psginput,
			                                 gnum_outconst1, psgoutput1,
			                                 gnum_outconst2, psgoutput2);

    cutStopTimer(timerg2);

    double tvg2 = cutGetTimerValue(timerg2);
    printf("gpu time glb(-num_var:%d): %e (%f)(ms)\n", num_var, tvg2, tvg2);
    cutDeleteTimer(timerg2);


    printf("diff: cpu-gpu:%e, percentage gpu/cpu:%f\n",
    		dateclock_diff_ms - timerg2,
    		(double)timerg2 / (double)dateclock_diff_ms);

    int *psoutput1, *psoutput2;
	psoutput1 = (int*)malloc(sz3);
	psoutput2 = (int*)malloc(sz3);

	hipMemcpy(psoutput1, psgoutput1, sz3, hipMemcpyDeviceToHost);
	hipMemcpy(psoutput2, psgoutput2, sz3, hipMemcpyDeviceToHost);



	return 0;
}
