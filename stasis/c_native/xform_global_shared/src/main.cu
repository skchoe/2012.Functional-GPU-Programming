#include "hip/hip_runtime.h"
/*
 * main.c
 *
 *  Created on: Feb 28, 2012
 *      Author: u0332192
 */

#include <stdio.h>
#include <time.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hip/device_functions.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>

typedef unsigned char byte;


int64_t timespecDiffNano(struct timespec *timeA_p, struct timespec *timeB_p)
{
  return ((timeA_p->tv_sec * 1000000000) + timeA_p->tv_nsec) -
           ((timeB_p->tv_sec * 1000000000) + timeB_p->tv_nsec);
}

// output c_constraint, c_asis
void init_constraints_ptrptr_cpu(int num_var, int max_num_const, size_t const_size,
		int* num_const, byte** constm_org,
		int* var_nums_cc, byte* c_constm, int* var_nums_ca, byte* c_asis)
{
	//printf("num_var:%d, max_num_const:%d\n", num_var, max_num_const);
	int i, j;
	for(i=0;i<max_num_const;i++) {
		for(j=0;j<num_var;j++) {
			if (i < num_const[j]) {
				byte cst = constm_org[j][i]; /// assumption : constraint is just one byte.
				//printf("cpu - var:%d, const:%d, cst:%d\n", j, i, cst);
				if(cst == 2) {
					memcpy(&(c_constm[j + num_var * var_nums_cc[j]]), &cst, const_size);
					var_nums_cc[j]++;
				}
				else {// cst==1
					memcpy(&(c_asis[j + num_var * var_nums_ca[j]]), &cst, const_size);
					var_nums_ca[j]++;
				}
			}
		}
	}

	//printf("__________init_cpu done______--\n");
	return;
}

void init_constraints_csr_cpu(int num_var, int tot_num_const, size_t const_size,
		int* indices, byte* data,
		int* var_nums_cc, byte* c_constm, int* var_nums_ca, byte* c_asis)
{
	//printf("num_var:%d, max_num_const:%d\n", num_var, max_num_const);
	printf("csr: tot_num_const:%d\n", tot_num_const);
	int i, j;
	for(i=0;i<tot_num_const;i++) {
		int var = indices[i];
		byte cst = data[i]; /// assumption : constraint is just one byte.
		//printf("cpu - var:%d, const:%d, cst:%d\n", j, i, cst);
		if(cst == 2) {
			memcpy(&(c_constm[var + num_var * var_nums_cc[var]]), &cst, const_size);
			var_nums_cc[var]++;
		}
		else if(cst == 1){// cst==1
			memcpy(&(c_asis[var + num_var * var_nums_ca[var]]), &cst, const_size);
			var_nums_ca[var]++;
		}
		else {
		}
	}

	//printf("__________init_cpu done______--\n");
	return;
}

__device__ unsigned int* shared_part(unsigned int th, unsigned int spart_width,
		int block_width, int block_idx)
{
	extern __shared__ unsigned int s_lock_nums[];
	unsigned int offset = (th * spart_width) + (block_idx * block_width);
	return (unsigned int*) (s_lock_nums + offset);
}


// shared memory structure
// just lock, varnum, output submatrix in block.x range. - read from global memory
//int sz_shd = (block.x * 2 * sizeof(int) * 2); //num_var, 2(lock,varnum), int-content type
//		+ block.x * sizeof(int) // var_name set
//		+ 2 * block.x * max_num_const;  // output index, output data
__global__ void init_constraints_shared(
		size_t* const_size, int* num_var, int* max_num_const, int* num_tot_const,
		int* ptr, int* indices, byte* data,
		unsigned int* var_lock_constm, unsigned int* var_nums_constm,
		int* max_num_constm, int* num_constm, byte* constm,
		unsigned int* var_lock_asis, unsigned int* var_nums_asis,
		int* max_num_asis, int* num_asis, byte* asis)
{
	extern __shared__ byte shared_sp[];

	// copy from global to shared
	size_t loc_lock_constm = 0,                               sz_lock_constm = blockDim.x * sizeof(int);
	size_t loc_num_constm = sz_lock_constm,                   sz_num_constm = blockDim.x * sizeof(int);
	size_t loc_lock_asis = loc_num_constm + sz_num_constm,    sz_lock_asis   = blockDim.x * sizeof(int);
	size_t loc_num_asis = loc_lock_asis + sz_lock_asis,       sz_num_asis = blockDim.x + sizeof(int);

	__shared__ int* s_lock_constm;
	s_lock_constm = (int*) (shared_sp + loc_lock_constm);
	__shared__ int* s_num_constm;
	s_num_constm = (int*) (shared_sp + loc_num_constm);

	__shared__ int* s_lock_asis;
	s_lock_asis = (int*) (shared_sp + loc_lock_asis);
	__shared__ int* s_num_asis;
	s_num_asis = (int*) (shared_sp + loc_num_asis);

	memcpy(s_lock_constm, var_lock_constm, sz_lock_constm);
	memcpy(s_num_constm,  var_nums_constm, sz_num_constm);
	memcpy(s_lock_asis,   var_lock_asis,   sz_lock_asis);
	memcpy(s_num_asis,    var_nums_asis,   sz_num_asis);

	__syncthreads();

	// coordinates in global memory
	int csr_gidx = threadIdx.x + blockIdx.x * blockDim.x;
	int csr_sidx = threadIdx.x;

//	check indexing into target array!!!!!!!!!!

	if (csr_gidx < *num_tot_const)
	{
		int var_id = indices[csr_gidx];
		byte* input_constm = (byte*) &(data[csr_gidx * *const_size]);

		int pos_input;
		if(*input_constm == 2) { // constraint
			do {
				pos_input = var_id + var_nums_constm[var_id] * *num_var;
				//memcpy(&constm[pos_input], input_constm, *const_size);
				constm[pos_input] = *input_constm;
				var_nums_constm[var_id]++;
				num_constm[var_id]++;

				var_lock_constm[var_id] = 0;
			} while(atomicCAS(&(var_lock_constm[var_id]), 0, 1));
		}
		else if(*input_constm == 1) { // analysis
			do {
				pos_input = var_id + var_nums_asis[var_id] * *num_var;
				//memcpy(&asis[pos_input], input_constm, *const_size);
				asis[pos_input] = *input_constm;
				var_nums_asis[var_id]++;
				num_asis[var_id]++;

				var_lock_asis[var_id] = 0;
			} while(atomicCAS(&(var_lock_asis[var_id]), 0, 1));
		}
		else {
			pos_input = 0;
			int value = 88;
			memset(&constm[var_id + var_nums_constm[var_id] * *num_var], value, *const_size);
			memset(&asis[var_id + var_nums_asis[var_id] * *num_var], value, *const_size);
		}
	}

	__syncthreads();

	// copy to global memory

	return;
}

__global__ void init_constraints_global
		(size_t* const_size, int* num_var, int* max_num_const, int* num_tot_const,
		int* ptr, int* indices, byte* data,
		unsigned int* var_lock_constm, unsigned int* var_nums_constm,
		int* max_num_constm, int* num_constm, byte* constm,
		unsigned int* var_lock_asis, unsigned int* var_nums_asis,
		int* max_num_asis, int* num_asis, byte* asis)
{
	// coordinates in global memory
	int csr_idx = threadIdx.x + blockIdx.x * blockDim.x;

//	check indexing into target array!!!!!!!!!!

	if (csr_idx < *num_tot_const)
	{
		int var_id = indices[csr_idx];
		byte* input_constm = (byte*) &(data[csr_idx * *const_size]);

		int pos_input;
		if(*input_constm == 2) { // constraint
			do {
				pos_input = var_id + var_nums_constm[var_id] * *num_var;
				//memcpy(&constm[pos_input], input_constm, *const_size);
				constm[pos_input] = *input_constm;
				var_nums_constm[var_id]++;
				num_constm[var_id]++;

				var_lock_constm[var_id] = 0;
			} while(atomicCAS(&(var_lock_constm[var_id]), 0, 1));
		}
		else if(*input_constm == 1) { // analysis
			do {
				pos_input = var_id + var_nums_asis[var_id] * *num_var;
				//memcpy(&asis[pos_input], input_constm, *const_size);
				asis[pos_input] = *input_constm;
				var_nums_asis[var_id]++;
				num_asis[var_id]++;

				var_lock_asis[var_id] = 0;
			} while(atomicCAS(&(var_lock_asis[var_id]), 0, 1));
		}
		else {
			pos_input = 0;
			int value = 88;
			memset(&constm[var_id + var_nums_constm[var_id] * *num_var], value, *const_size);
			memset(&asis[var_id + var_nums_asis[var_id] * *num_var], value, *const_size);
		}
	}

	__syncthreads();

	return;
}


int sharedMemory()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0); // 0=dev single device
	return deviceProp.sharedMemPerBlock;
}

// given input from Racket(list of list), produces CSR
void transform_const_csr(int num_var, int num_tot_const, int max_num_const, int* num_const, byte** constm,
						int* ptr, int* indices, byte* data, size_t sz_const)
{
	int i,j,k,l;

	k=0, l=0;	// l: num of valid consts

	for(i=0;i<max_num_const;i++) {
		for(j=0;j<num_var;j++) {
			if (i < num_const[j]) {
				memcpy(&data[k], &(constm[j][i]), sz_const);
				data[k] = constm[j][i];
				indices[k] = j;
				l++;
				k++;
			}
		}
		ptr[i+1] = l;
	}
//	for(k=0;k<num_var;k++)
//		printf("num const[%d:%d\n", k, num_const[k]);
//
//	for(j=0;j<=max_num_const;j++)
//		printf("transform_const's ptr[%d]:%d\n", j, ptr[j]);

	return;
}

double ilog(double base, double x) {
	return log(x) / log(base);
}


int main(int argc, char** argv)
{
	int i, j, k;

	//create data
	int expo = 20;
	int num_var = (int) pow(2, expo);//16; // ; we'll 2^9/2^4 constraints in max
	int num_const_limit = (int) pow(2, 4); // per each variable


	size_t const_size = 1; // 1byte k spaces, k=1 for now
	size_t sz_num_const = sizeof(int) * num_var; // number of constraints for each variable

	// list of num_const for each var.
	int* num_const = (int*) malloc(sz_num_const);
	for(i=0;i<num_var;i++) {
		num_const[i] = (int) (rand() % (num_const_limit + 1));
//		printf("%d, %d / %d\n", i, num_const[i], num_const_limit);
	}

	int num_tot_const = 0;
	for(i=0;i<num_var;i++) {
		num_tot_const += num_const[i];
	}
//	printf("total num const:%d\n", num_tot_const);

	int max_num_const = 0;
	for(i=0;i<num_var;i++) {
		int n = num_const[i];
		if(n > max_num_const) max_num_const = n;
	}
	////////////////////////////////////////////////////////////////////////////////////////
	// fill constraint either 1 or 2.
	size_t sz_const = num_tot_const * const_size;
	byte* const_org = (byte*) malloc (sz_const);
	k = 0;
	for(i=0;i<num_var;i++) // height
		for(j=0;j<num_const[i];j++) {
			const_org[k] = rand() % 2 + 1;
			k++;
		}

	printf("num_tot_const:%d, const_size:%d, max_num_const:%d\n", num_tot_const, const_size, max_num_const);
//	k=0;
//	for(i=0;i<num_var;i++) // height
//		for(j=0;j<num_const[i];j++) {
//			printf("var:%d, const:%d, const_org[%d]: %d\n", i, j, k, const_org[k]);
//			k++;
//		}

	k=0;
	//// const_org -> constm: (array of byte) -> (array of pointer of byte)
	byte** constm_org = (byte**) malloc(sizeof(byte*) * num_var);
	for(i=0;i<num_var;i++)
		constm_org[i] = (byte*)malloc(num_const[i] * const_size);
	for(i=0;i<num_var;i++) // height
		for(j=0;j<num_const[i];j++) {
			memcpy(&constm_org[i][j], &const_org[k], const_size);
			k++;
		}



//	for(i=0;i<num_var;i++)  {// height
//		printf("num of constraint at %dth var:\n", i);
//		for(j=0;j<num_const[i];j++) {
//			printf("const[%d,%d] = %d\n", i, j, constm_org[i][j]);
//		}
//	}

	size_t sz_mat = const_size * num_var * max_num_const;
	size_t sz_var_num = sizeof(int) * num_var;

	//	// begin cpu time check
	timespec ts_start, ts_stop;
	printf("CLOCk:.....%d\n", CLOCK_MONOTONIC + 2);
	if(0==clock_gettime(CLOCK_MONOTONIC, &ts_start))
	printf("taking clock - start successful\n");

	byte* c_constm = (byte*) malloc(sz_mat);
	memset(c_constm, NULL, sz_mat);

	int* var_nums_cc = (int*)malloc(sz_var_num);
	memset(var_nums_cc, 0, sz_var_num);

	byte* c_asis = (byte*) malloc(sz_mat);
	memset(c_asis, NULL, sz_mat);

	int* var_nums_ca = (int*)malloc(sz_var_num);
	memset(var_nums_ca, 0, sz_var_num);


	init_constraints_ptrptr_cpu(num_var, max_num_const, const_size,
			num_const, constm_org, var_nums_cc, c_constm, var_nums_ca, c_asis);

	if(0==clock_gettime(CLOCK_MONOTONIC, &ts_stop))
		printf("taking clock - stop successful\n");
	uint64_t timeElapsedNano = timespecDiffNano(&ts_stop, &ts_start);


	// end cpu time check
	//clock_t tv_stop = clock();
	//double diff_ms = ((double)tv_stop - (double) tv_start) * 1000. / CLOCKS_PER_SEC;

	//printf("CPU(clock) (%d->%d)\nDIFF:%e (%f)(ms)\n\n", tv_start, tv_stop, diff_ms, diff_ms);
	printf("cpu - ptrptr (s):(%u->%u)\ncpu(ns):(%u->%u)\n cpu_elapsed(ms):%f \n\n",
			ts_start.tv_sec, ts_stop.tv_sec,
			ts_start.tv_nsec, ts_stop.tv_nsec, (double)timeElapsedNano/(double)1000000000);


//	for(i=0;i<num_var;i++) {
//		printf("var:%d ------- num_const:%d\n", i, var_nums_cc[i]);
//		for(j=0;j<var_nums_cc[i];j++) {
//			printf("var:%d, const:%d, byte-const:%d\n", i, j, c_constm[i + j * num_var]);
//		}
//	}
//
//	for(i=0;i<num_var;i++) {
//		printf("var:%d ------- num_asis:%d\n", i, var_nums_ca[i]);
//		for(j=0;j<var_nums_ca[i];j++) {
//			printf("var:%d, const:%d, byte-asis:%d\n", i, j, c_asis[i + j * num_var]);
//		}
//	}

	/// prepare for GPU compuatation.
	//////////////// CSR format conversion
	size_t sz_ptr = sizeof(int) * (max_num_const + 1);
	int* ptr = (int*) malloc(sz_ptr); // ptr is 1+max num_const
	for(i=0;i<=max_num_const;i++) ptr[i] = 0;
	size_t sz_indices = sizeof(int) * num_tot_const;
	int* indices = (int*) malloc(sz_indices);
	memset(indices, NULL, sz_indices);

	byte* data = (byte*) malloc(sz_const);
	memset(data, NULL, sz_const);
	transform_const_csr(num_var, num_tot_const, max_num_const, num_const, constm_org,
							ptr, indices, data, const_size);

	// check 2nd cpu init
	if(0==clock_gettime(CLOCK_MONOTONIC, &ts_start))
		printf("taking clock - start successful\n");

	// iniitialize again
	memset(c_constm, NULL, sz_mat);
	memset(var_nums_cc, 0, sz_var_num);
	memset(c_asis, NULL, sz_mat);
	memset(var_nums_ca, 0, sz_var_num);

	// init_contraints_cpu by CSRint num_var, int tot_num_const, size_t const_size,
	init_constraints_csr_cpu(num_var, num_tot_const, const_size,
			indices, data, var_nums_cc, c_constm, var_nums_ca, c_asis);

	if(0==clock_gettime(CLOCK_MONOTONIC, &ts_stop))
		printf("taking clock - stop successful\n");
	timeElapsedNano = timespecDiffNano(&ts_stop, &ts_start);

	printf("cpu -csr (s):(%u->%u)\ncpu(ns):(%u->%u)\n cpu_elapsed(ms):%f \n\n",
			ts_start.tv_sec, ts_stop.tv_sec,
			ts_start.tv_nsec, ts_stop.tv_nsec, (double)timeElapsedNano/(double)1000000000);

	////////////////////////////////////////////////////////////////////////////////////////
	// GPU configuration
	// num_var:2^10, max_num_const
	int least_num_threadx = max(16, (int)(num_var/pow(2, 4)));
//	int num_thread_x = least_num_threadx;
	int num_thread_x = min(least_num_threadx , (int)pow(2, 8)); //maximum 256
	printf("decied thread.x = %d\n", num_thread_x);
	int num_thread_y = 1;
	int num_block_x = pow (2, ceil (ilog (2, num_tot_const / num_thread_x)));
	int num_block_y = 1;

	dim3 grid(num_block_x, num_block_y);
	dim3 block(num_thread_x, num_thread_y);

	printf("tot_const:%d, num_thd_x:%d __________________________>>>num_block_x : %d\n", num_tot_const, num_thread_x, num_block_x);


	////////////////////////////////////////////////////////////////////////////////////////
	// time to split from const_org into { constm, analysis }
	int SHD_CAP = sharedMemory();

	// just lock, varnum, output submatrix in block.x range. - read from global memory
	int sz_shd = (block.x * 2 * sizeof(int) * 2); //num_var, 2(lock,varnum), int-content type
//			+ (max_num_const + 1) * sizeof(int) // ptr
//			+ num_tot_const * (sizeof(int) + const_size) // indices and consts
			+ block.x * sizeof(int) // var_name set
			+ 2 * block.x * max_num_const;  // output index, output data

	printf("shared mem: required:%d bytes, cap:%d bytes\n", sz_shd, SHD_CAP);


	//begin gpu timer
    unsigned int timerg_all = 0;
    cutCreateTimer(&timerg_all);
    cutResetTimer(timerg_all);

    unsigned int timerg_exe = 0;
    cutCreateTimer(&timerg_exe);
    cutResetTimer(timerg_exe);

	cutStartTimer(timerg_all);

	// Basic inputs
	size_t* g_const_size;
	hipMalloc(&g_const_size, sizeof(size_t));
	hipMemcpy(g_const_size, &const_size, sizeof(size_t), hipMemcpyHostToDevice);

	int* g_num_var;
	hipMalloc(&g_num_var, sizeof(int));
	hipMemcpy(g_num_var, &num_var, sizeof(int), hipMemcpyHostToDevice);

	int* g_max_num_const;
	hipMalloc(&g_max_num_const, sizeof(int));
	hipMemcpy(g_max_num_const, &max_num_const, sizeof(int), hipMemcpyHostToDevice);

	int* g_num_tot_const;
	hipMalloc(&g_num_tot_const, sizeof(int));
	hipMemcpy(g_num_tot_const, &num_tot_const, sizeof(int), hipMemcpyHostToDevice);

	// CSR structure
	int* g_ptr;
	hipMalloc(&g_ptr, sz_ptr);
	hipMemcpy(g_ptr, ptr, sz_ptr, hipMemcpyHostToDevice);

	int* g_indices;
	hipMalloc(&g_indices, sz_indices);
	hipMemcpy(g_indices, indices, sz_indices, hipMemcpyHostToDevice);

	byte* g_data;
	hipMalloc(&g_data, sz_const);
	hipMemcpy(g_data, data, sz_const, hipMemcpyHostToDevice);


	//output size
	size_t sz_constm = const_size * max_num_const * num_var; // used as matrix alloc for output
	////////////////////////////////////////////////////////////////////////////////////////
	// alloc constraints, & analysis
	// gpu containers for returns
	int* g_max_num_constm;
	hipMalloc(&g_max_num_constm, sizeof(int));
	hipMemset(g_max_num_constm, NULL, sizeof(int));

	int* g_num_constm;
	hipMalloc(&g_num_constm, sz_num_const);
	hipMemset(g_num_constm, NULL, sz_num_const);

	byte* g_constm;
	hipMalloc(&g_constm, sz_constm);
	hipMemset(g_constm, NULL, sz_constm);

	int* g_max_num_asis;
	hipMalloc(&g_max_num_asis, sizeof(int));
	hipMemset(g_max_num_asis, NULL, sizeof(int));

	int* g_num_asis;
	hipMalloc(&g_num_asis, sz_num_const);
	hipMemset(g_num_asis, NULL, sz_num_const);

	byte* g_asis;
	hipMalloc(&g_asis, sz_constm);
	hipMemset(g_asis, NULL, sz_constm);


	// lock for each column - constraint  (values 0 or 1)
	size_t sz_locks = num_var * sizeof(unsigned int);
	unsigned int* gvar_lock_constm;
	hipMalloc(&gvar_lock_constm, sz_locks);
	hipMemset(gvar_lock_constm, 0, sz_locks);

	// current column indicator - constraint
	unsigned int* gvar_nums_constm;
	hipMalloc(&gvar_nums_constm, sz_locks);
	hipMemset(gvar_nums_constm, NULL, sz_locks);

	// lock for each column - analysis (values 0 or 1)
	unsigned int* gvar_lock_asis;
	hipMalloc(&gvar_lock_asis, sz_locks);
	hipMemset(gvar_lock_asis, 0, sz_locks);

	// current column indicator - analysis
	unsigned int* gvar_nums_asis;
	hipMalloc(&gvar_nums_asis, sz_locks);
	hipMemset(gvar_nums_asis, NULL, sz_locks);

	cutStartTimer(timerg_exe);

//	if( SHD_CAP < sz_shd) {
//		printf("shared memory (lock+nums) is too small(%d byte) to be fit for num_var\n", sz_shd);
//

		init_constraints_global<<<grid, block>>>(g_const_size, g_num_var, g_max_num_const, g_num_tot_const,
										 g_ptr, g_indices, g_data,
										 gvar_lock_constm, gvar_nums_constm,
										 g_max_num_constm, g_num_constm, g_constm,
										 gvar_lock_asis, gvar_nums_asis,
										 g_max_num_asis, g_num_asis, g_asis);
//	}
//	else {
//
//		printf("--------shared memory (lock+nums) is can contain num_var (%d bytes=%d x %d)\n", sz_shd, max_num_const, block.x);
//		printf("grid dimx:%d, dimy:%d\n", grid.x, grid.y);
//		init_constraints_shared<<<grid, block, sz_shd>>>(g_const_size, g_num_var, g_max_num_const, g_num_tot_const,
//	 	 	 	 	 	 	 	 	 	 	 g_ptr, g_indices, g_data,
//							 	 	 	 	 gvar_lock_constm, gvar_nums_constm,
//	 	 	 	 	 	 	 	 	 	 	 g_max_num_constm, g_num_constm, g_constm,
//							 	 	 	 	 gvar_lock_asis, gvar_nums_asis,
//	 	 	 	 	 	 	 	 	 	 	 g_max_num_asis, g_num_asis, g_asis);
//	}

    cutStopTimer(timerg_exe);
    cutStopTimer(timerg_all);

//	int* num_constm = (int*) malloc (sz_num_const);
//	hipMemcpy(num_constm, g_num_constm, sz_num_const, hipMemcpyDeviceToHost);

	byte* new_constm = (byte*) malloc (sz_constm);
	hipMemcpy(new_constm, g_constm, sz_constm, hipMemcpyDeviceToHost);

//	int* num_asis = (int*) malloc (sz_num_const);
//	hipMemcpy(num_asis, g_num_asis, sz_num_const, hipMemcpyDeviceToHost);

	byte* asis = (byte*) malloc (sz_constm);
	hipMemcpy(asis, g_asis, sz_constm, hipMemcpyDeviceToHost);

//	for(i=0;i<num_var;i++)
//		for(j=0;j<max_num_const;j++)
//			printf("newconst[%d][%d]:%d\n", i, j, new_constm[j*num_var+i]);
//
//	for(i=0;i<num_var;i++)
//		for(j=0;j<max_num_const;j++)
//			printf("asis[%d][%d]:%d\n", i, j, asis[j*num_var+i]);

	// current column indicator - constraint
	unsigned int* var_nums_constm = (unsigned int*)malloc (sz_locks);
	hipMemcpy(var_nums_constm, gvar_nums_constm, sz_locks, hipMemcpyDeviceToHost);

	// current column indicator - analysis
	unsigned int* var_nums_asis = (unsigned int*)malloc (sz_locks);
	hipMemcpy(var_nums_asis, gvar_nums_asis, sz_locks, hipMemcpyDeviceToHost);



    double tvg = cutGetTimerValue(timerg_exe);
    printf("gpu time glb(kernel):\n %e \n(%f)(ms)\n", tvg, tvg);
    cutDeleteTimer(timerg_exe);

    double tvga = cutGetTimerValue(timerg_all);
    printf("gpu time glb(kernel+in-copy):\n %e \n(%f)(ms)\n", tvga, tvga);
    cutDeleteTimer(timerg_all);

//	for(i=0;i<num_var;i++) {
//		printf("var:%d, num_constm:%d\n", i, var_nums_constm[i]);
//	}
//
//	for(i=0;i<num_var;i++) {
//		printf("var:%d, num_asis:%d\n", i, var_nums_asis[i]);
//	}
//







	return 1;
}
