
#include <hip/hip_runtime.h>
//#ifndef _MATRIXMUL_KERNEL_H_
//#define _MATRIXMUL_KERNEL_H_

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define GRID_SIZE_X 8
#define GRID_SIZE_Y 8

extern "C"

/* Signature:
  float* d_array_in, int count, uint single_in 
  ->
  float* d_array_out, uint* d_array_len, int* memstruct, uint* single_out
*/
__global__ void
addfloat(
float* d_array_in, 
size_t count,
size_t width, size_t height, size_t depth,
float d_single_in, 
float* d_array_out,
size_t* d_array_out_len,
float val)
{

  int idx = threadIdx.x + BLOCK_SIZE_X * blockIdx.x;
  int idy = threadIdx.y + BLOCK_SIZE_Y * blockIdx.y;
  int thd_width = BLOCK_SIZE_X * GRID_SIZE_X;

  int linpos = thd_width * idy + idx;

  *(d_array_out+linpos) = d_array_in[linpos] + val;
  __syncthreads();
}

