#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<cutil.h>
#include "def.h"
#include "hashtab.h"
//#include "lock.h"

#define islower(c) (((c) >= 'a') && ((c) <= 'b'))
#define isupper(c) (((c) >= 'A') && ((c) <= 'Z'))
#define isalpha(c) (islower(c) && isupper(c))
#define tolower(c) ((c)-'A'+'a')

// Hash Function

__device__ int get_hash_from_char(char c){
        if (isalpha(c)){
                if (isupper(c)){
                        return tolower(c) - 'a' + 26;
                } else {
                        return c - 'a';
                }
        }
        return -1;
}

__device__ int hashfunc(char *to){
        int v1 = get_hash_from_char(*to);
        if (v1 == -1) return -1;
        if ( *(to+1) == ' ' || *(to+1) == '\0')
        {
                return v1;
        }

        int v2 = get_hash_from_char(*(to+1));
        if (v2 == -1) return -1;

        return (1+v1)*52 + v2;
}

// return pointer of h[key,index]

__device__ char* get_element2(char* h, int num_var, int key, int index){
	int row = (key + index*num_var)*SIZE_CONSTRAINT;
	char *elem_addr = h + row;
	return elem_addr;
}


char* get_element3(char* h, int num_var, int key, int index){
	int row = (key + index*num_var)*SIZE_CONSTRAINT;
	char *elem_addr = h + row;
	return elem_addr;
}

__device__ int put_element2(char* h, int num_var, int num_const, int key, char* constraint, /*Lock* lock,*/  int flag){
	int i;

	for (i=0 ; i < num_const; i++){
		int row = (key + i*num_var)*SIZE_CONSTRAINT;
		char* temp = h + row;
		if (*temp == '\0'){
			//lock[flag].lock();
			memcpy(temp, constraint, SIZE_CONSTRAINT);
			//lock[flag].unlock();
			return 1;
		}
	}
	return -1;
}
 

__device__ void memcopy(char* dest, char* sour, int size){
	int i;
	for(i=0; i < size; i++)
		dest[i] = sour[i];

}

extern "C"
__global__ void init_constraints_kernel(char *constraints, SizeType2* st2_inc, int num_var, int num_const, const int size_constraint, char* new_constraints, SizeType2* st2_outc, char* analysis, SizeType2* st2_an, int* empty_contraint)
{
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	
	char* a_constraint = get_element2(constraints,num_var,idx,idy);

	int pos = (idx + idy * num_var) * size_constraint;
	if( (constraints[pos] == '_') && ( constraints[pos+1] == 'v' || constraints[pos+1] == 'c' || constraints[pos+1] == 'l')){
       		memcpy(analysis+pos, a_constraint, size_constraint);
	} else {
       		memcpy(new_constraints+pos, a_constraint, size_constraint);
		*empty_contraint = 0;
	}
	__syncthreads();
}



extern "C"
__global__ void solve_constraints_kernel(char* reflection, SizeType2* st2_inr, char* constraint, SizeType2* st2_inc, int num_var, int num_const, 
                                         char* new_constraints, SizeType2* st2_outc, char* analysis, SizeType2* st2_anl, int* empty_constraint, /*Lock* lock, */char* out1,char* out2)
{
	__shared__ char sh_constraints[BLOCK_SIZE_X*BLOCK_SIZE_Y*SIZE_CONSTRAINT];
	register char n_const[8];
	register char n_const2[8];
	register char value[8];

	int idx = threadIdx.x;
	int idy = threadIdx.y;
  	int i;


	char* sharemem_addr = sh_constraints; 
	sharemem_addr = sharemem_addr+idx*SIZE_CONSTRAINT*BLOCK_SIZE_X+idy*SIZE_CONSTRAINT;
	memcpy(sharemem_addr, get_element2(constraint, num_var,idx, idy), SIZE_CONSTRAINT);	

	memcpy(out1,sharemem_addr,SIZE_CONSTRAINT);
	for(i=0; i < num_const; i++){
		memcopy(value,get_element2(analysis, num_var, idx ,i),SIZE_CONSTRAINT);
		//memcopy(out2,value,SIZE_CONSTRAINT);	
		if (*value != '\0'){
			memset(n_const, '\0', 8);
			memset(n_const2, '\0', 8);
			if((*sharemem_addr=='_' && *(sharemem_addr+1)=='P') && (value[0]=='_' && value[1] == 'v')){
				int index = hashfunc(sharemem_addr+2);
				memcpy(n_const,value,8);
				put_element2(analysis,num_var,num_const, index,n_const,/*lock,*/0);
				int j;
				for(j=0;j <num_const; j++){
					char *c = get_element2(reflection,num_var,index, j);
					if (*c != '\0'){
						put_element2(new_constraints,num_var,num_const, index,c, /*lock,*/1);
						*empty_constraint = 0;
					}
				}
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1)=='C') && (value[0]=='_' && value[1] == 'c')){
				int index = hashfunc(value+2);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,sharemem_addr+2,6);
				put_element2(new_constraints, num_var,num_const, index,n_const,/*lock*/1);
				*empty_constraint = 0;
			} else if(((*sharemem_addr=='_' && *(sharemem_addr+1)=='D')) && (value[0]=='_' && value[1] == 'c')){
				int index = hashfunc(value+4);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,sharemem_addr+2,6);
				put_element2(new_constraints, num_var, num_const, index, n_const, /*lock,*/0);
				*empty_constraint = 0;
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1)=='A') && (value[0]=='_' && value[1] == 'l')){
				int index1 = hashfunc(value+6);
				int index2 = hashfunc(sharemem_addr +4);
				n_const[0] = '_';
				n_const2[0]= '_';
				n_const[1] = 'P';
				n_const2[1]= 'P';
				memcopy(n_const+2,sharemem_addr+2,2);
				memcopy(n_const2+2,value+2,2);
				put_element2(new_constraints, num_var, num_const, index1, n_const, /*lock,*/1);
				put_element2(new_constraints, num_var, num_const, index2, n_const2, /*lock,*/1);
				memcopy(out1,n_const,8);
				memcopy(out2,n_const2,8);
				*empty_constraint = 0;
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1) =='A') && (value[0]=='_' && value[1] == 't')){
				int index = hashfunc(sharemem_addr+4);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,value+2,6);
				put_element2(new_constraints, num_var, num_const, index, n_const, /*lock,*/1);
				*empty_constraint = 0;
			} else {
				//*empty_constraint = 1;
			}
		}
	}

	__syncthreads();
}

