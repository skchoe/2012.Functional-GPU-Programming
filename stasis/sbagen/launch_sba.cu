#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
//#include <cutil.h>

#include "hashtab.h"
#include "def.h"

// include kernel
#include <cutil.h> 


//include the head
void print_memory(char* dish, int szx, int szy, int szelt)
{
  int i,j,k,pos;
  for(i = 0 ; i < szx ; i++) {
    for(j = 0 ; j < szy ; j++) {
      pos = (j*szx + i) * szelt;
      printf("[%d,%d]\t", i, j);
      for(k=0;k<szelt;k++)
        printf("%c", dish[pos+k]);
      printf("\t");
    }
    printf ("\n");
  }
  printf ("\n");
}

// those wrappers call kernel functions
void solver_constraint_wrapper(hash_tab* c, char* out_analysis){
	int num_var = c->num_variable;
	int num_const = c->num_constraint;
	int size = num_var * num_const * SIZE_CONSTRAINT;
	char *c_array;
	char *c_new_array;
	char *analysis;
	char *reflection;
	char *temp; // debug 
	/*Lock lock[2];
	Lock *dev_lock;

	CUDA_SAFE_CALL(hipMalloc( (void**) &dev_lock, 2*sizeof(Lock)));
	CUDA_SAFE_CALL(hipMemcpy(dev_lock, lock, 2*sizeof(Lock), hipMemcpyHostToDevice));
	*/

	unsigned int timer = 0;

	CUDA_SAFE_CALL(hipMalloc( (void**)&c_array, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&c_new_array, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&analysis, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&reflection, size));


	temp = (char*) malloc(size);
	memset(temp,'\0',size);	

	CUDA_SAFE_CALL(hipMemcpy(c_array, c->ht, size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(c_new_array, '\0', size));
	CUDA_SAFE_CALL(hipMemset(analysis, '\0', size));
		
	dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 grid(1);

	int empty_const=1;
	int *d_empty_const;
	int size_int = sizeof(int);

	CUDA_SAFE_CALL(hipMalloc((void**)&d_empty_const,size_int));
	CUDA_SAFE_CALL(hipMemcpy(d_empty_const, &empty_const,size_int,hipMemcpyHostToDevice));

	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));

	SizeType2* st_inc = (SizeType2*)malloc(sizeof (SizeType2));
	st_inc->w = num_const * SIZE_CONSTRAINT;
	st_inc->h = num_var;
	SizeType2* st_oc = (SizeType2*)malloc(sizeof (SizeType2));
	SizeType2* st_anl = (SizeType2*)malloc(sizeof (SizeType2));

	/* Call Kernel function */
	init_constraints_kernel<<< grid, threads >>>(c_array, st_inc, num_var, num_const, SIZE_CONSTRAINT, c_new_array, st_oc, analysis, st_anl, d_empty_const); 

	CUDA_SAFE_CALL(hipMemcpy(&empty_const, d_empty_const,4,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(reflection, c_new_array, size, hipMemcpyDeviceToDevice));
	//printf("(let (a (lambda c c)) in\n (let b 1) in\n (app a b) \n");
	
	//printf("frist constraint \n" );
	CUDA_SAFE_CALL(hipMemcpy(temp, c_new_array, size, hipMemcpyDeviceToHost));
	//print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
		
	char* out_c_new_array;
	char* out1,*out2;

	char* o1,*o2;

	o1 =(char*) malloc(9);
	o2 =(char*) malloc(9);
	bzero(o1,9);
	bzero(o2,9);	
	hipMalloc(&out1,8);
	hipMalloc(&out2,8);
	
	// SizeType2 arguments alloc
	SizeType2* st2_refl = (SizeType2*)malloc(sizeof (SizeType2));
	SizeType2* st2_c_in = (SizeType2*)malloc(sizeof (SizeType2));
	SizeType2* st2_c_out = (SizeType2*)malloc(sizeof (SizeType2));
	SizeType2* st2_anl = (SizeType2*)malloc(sizeof (SizeType2));

	while (!empty_const) {	
		empty_const = 1;
		CUDA_SAFE_CALL(hipMemcpy(d_empty_const, &empty_const,size_int,hipMemcpyHostToDevice));
		
		CUDA_SAFE_CALL(hipMalloc(&out_c_new_array, size));
		CUDA_SAFE_CALL(hipMemset(out_c_new_array, '\0', size));
	
		// call a kernel function
		solve_constraints_kernel<<< grid, threads>>> (reflection, st2_refl, c_new_array, st2_c_in, num_var, num_const, 
		                                 out_c_new_array, st2_c_out, analysis, st2_anl, d_empty_const,out1,out2); //dev_lock		//printf("debug11\n");
		CUDA_SAFE_CALL(hipMemcpy(&empty_const, d_empty_const,size_int,hipMemcpyDeviceToHost));

		//printf("new constraint1:\n");		
		hipMemcpy(temp, out_c_new_array, size, hipMemcpyDeviceToHost);
		//print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);

		hipFree(c_new_array);

		c_new_array = out_c_new_array;	
		out_c_new_array = NULL;

		///printf("new analysis:\n");
		//hipMemcpy(temp2, analysis, size, hipMemcpyDeviceToHost);
		//print_memory(temp2, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);

		//printf("new constraint2:\n");
		hipMemcpy(temp, c_new_array, size, hipMemcpyDeviceToHost);
		//print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
	}


	CUT_SAFE_CALL(cutStopTimer(timer));
	printf("Processing time:%f (ms)\n", cutGetTimerValue(timer));
	CUT_SAFE_CALL(cutDeleteTimer(timer));

        hipMemcpy(o1, out1,8,hipMemcpyDeviceToHost);
        hipMemcpy(o2, out2,8,hipMemcpyDeviceToHost);


 	// now analysis contains final answers at each colume(var).
	// Need to show them: var -> value mapping by having way of finding var from column index.
	
	CUDA_SAFE_CALL(hipMemcpy(out_analysis, analysis, size, hipMemcpyDeviceToHost));
	printf("\nSOLUTIONS=========\n");
	print_memory(out_analysis, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
	hipFree(c_array);
	hipFree(c_new_array);
	hipFree(analysis);
	//hipFree(dev_lock);
	hipFree(out_c_new_array);
}
