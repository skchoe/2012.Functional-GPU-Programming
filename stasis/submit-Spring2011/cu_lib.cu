#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<cutil.h>
#include "def.h"
#include "hashtab.h"
#include "cu_lib.h"
//#include "lock.h"

#define islower(c) (((c) >= 'a') && ((c) <= 'b'))
#define isupper(c) (((c) >= 'A') && ((c) <= 'Z'))
#define isalpha(c) (islower(c) && isupper(c))
#define tolower(c) ((c)-'A'+'a')

// Hash Function

__device__ int get_hash_from_char(char c){
        if (isalpha(c)){
                if (isupper(c)){
                        return tolower(c) - 'a' + 26;
                } else {
                        return c - 'a';
                }
        }
        return -1;
}

__device__ int hashfunc(char *to){
        int v1 = get_hash_from_char(*to);
        if (v1 == -1) return -1;
        if ( *(to+1) == ' ' || *(to+1) == '\0')
        {
                return v1;
        }

        int v2 = get_hash_from_char(*(to+1));
        if (v2 == -1) return -1;

        return (1+v1)*52 + v2;
}

// return pointer of h[key,index]

__device__ char* get_element2(char* h, int num_var, int key, int index){
	int row = (key + index*num_var)*SIZE_CONSTRAINT;
	char *elem_addr = h + row;
	return elem_addr;
}


char* get_element3(char* h, int num_var, int key, int index){
	int row = (key + index*num_var)*SIZE_CONSTRAINT;
	char *elem_addr = h + row;
	return elem_addr;
}

__device__ int put_element2(char* h, int num_var, int num_const, int key, char* constraint, /*Lock* lock,*/  int flag){
	int i;

	for (i=0 ; i < num_const; i++){
		int row = (key + i*num_var)*SIZE_CONSTRAINT;
		char* temp = h + row;
		if (*temp == '\0'){
			//lock[flag].lock();
			memcpy(temp, constraint, SIZE_CONSTRAINT);
			//lock[flag].unlock();
			return 1;
		}
	}
	return -1;
}
 

__device__ void memcopy(char* dest, char* sour, int size){
	int i;
	for(i=0; i < size; i++)
		dest[i] = sour[i];

}

extern "C"
__global__ void init_constraints_kernel(char *constraints, int num_var, int num_const, const int size_constraint, char* new_constraints, char* analysis, int* empty_contraint){
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	
	char* a_constraint = get_element2(constraints,num_var,idx,idy);

	int pos = (idx + idy * num_var) * size_constraint;
	if( (constraints[pos] == '_') && ( constraints[pos+1] == 'v' || constraints[pos+1] == 'c' || constraints[pos+1] == 'l')){
       		memcpy(analysis+pos, a_constraint, size_constraint);
	} else {
       		memcpy(new_constraints+pos, a_constraint, size_constraint);
		*empty_contraint = 0;
	}
	__syncthreads();
}



extern "C"
__global__ void solve_constraints_kernel(char *reflection, char* constraint, int num_var, int num_const, char* new_constraints, char* analysis, int* empty_constraint, /*Lock* lock, */char* out1,char* out2){
	__shared__ char sh_constraints[BLOCK_SIZE_X*BLOCK_SIZE_Y*SIZE_CONSTRAINT];
	register char n_const[8];
	register char n_const2[8];
	register char value[8];

	int idx = threadIdx.x;
	int idy = threadIdx.y;
  	int i;


	char* sharemem_addr = sh_constraints; 
	sharemem_addr = sharemem_addr+idx*SIZE_CONSTRAINT*BLOCK_SIZE_X+idy*SIZE_CONSTRAINT;
	memcpy(sharemem_addr, get_element2(constraint, num_var,idx, idy), SIZE_CONSTRAINT);	

	memcpy(out1,sharemem_addr,SIZE_CONSTRAINT);
	for(i=0; i < num_const; i++){
		memcopy(value,get_element2(analysis, num_var, idx ,i),SIZE_CONSTRAINT);
		//memcopy(out2,value,SIZE_CONSTRAINT);	
		if (*value != '\0'){
			memset(n_const, '\0', 8);
			memset(n_const2, '\0', 8);
			if((*sharemem_addr=='_' && *(sharemem_addr+1)=='P') && (value[0]=='_' && value[1] == 'v')){
				int index = hashfunc(sharemem_addr+2);
				memcpy(n_const,value,8);
				put_element2(analysis,num_var,num_const, index,n_const,/*lock,*/0);
				int j;
				for(j=0;j <num_const; j++){
					char *c = get_element2(reflection,num_var,index, j);
					if (*c != '\0'){
						put_element2(new_constraints,num_var,num_const, index,c, /*lock,*/1);
						*empty_constraint = 0;
					}
				}
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1)=='C') && (value[0]=='_' && value[1] == 'c')){
				int index = hashfunc(value+2);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,sharemem_addr+2,6);
				put_element2(new_constraints, num_var,num_const, index,n_const,/*lock*/1);
				*empty_constraint = 0;
			} else if(((*sharemem_addr=='_' && *(sharemem_addr+1)=='D')) && (value[0]=='_' && value[1] == 'c')){
				int index = hashfunc(value+4);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,sharemem_addr+2,6);
				put_element2(new_constraints, num_var, num_const, index, n_const, /*lock,*/0);
				*empty_constraint = 0;
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1)=='A') && (value[0]=='_' && value[1] == 'l')){
				int index1 = hashfunc(value+6);
				int index2 = hashfunc(sharemem_addr +4);
				n_const[0] = '_';
				n_const2[0]= '_';
				n_const[1] = 'P';
				n_const2[1]= 'P';
				memcopy(n_const+2,sharemem_addr+2,2);
				memcopy(n_const2+2,value+2,2);
				put_element2(new_constraints, num_var, num_const, index1, n_const, /*lock,*/1);
				put_element2(new_constraints, num_var, num_const, index2, n_const2, /*lock,*/1);
				memcopy(out1,n_const,8);
				memcopy(out2,n_const2,8);
				*empty_constraint = 0;
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1) =='A') && (value[0]=='_' && value[1] == 't')){
				int index = hashfunc(sharemem_addr+4);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,value+2,6);
				put_element2(new_constraints, num_var, num_const, index, n_const, /*lock,*/1);
				*empty_constraint = 0;
			} else {
				//*empty_constraint = 1;
			}
		}
	}

	__syncthreads();
}



void print_memory(char* dish, int szx, int szy, int szelt)
{
  int i,j,k,pos;
  for(i = 0 ; i < szx ; i++) {
    for(j = 0 ; j < szy ; j++) {
      pos = (j*szx + i) * szelt;
      printf("[%d,%d]\t", i, j);
      for(k=0;k<szelt;k++)
        printf("%c", dish[pos+k]);
      printf("\t");
    }
    printf ("\n");
  }
  printf ("\n");
}

// those wrappers call kernel functions
void solver_constraint_wrapper(hash_tab* c, char* out_analysis){
	int num_var = c->num_variable;
	int num_const = c->num_constraint;
	int size = num_var * num_const * SIZE_CONSTRAINT;
	char *c_array;
	char *c_new_array;
	char *analysis;
	char *reflection;
	char *temp; // debug 
	/*Lock lock[2];
	Lock *dev_lock;

	CUDA_SAFE_CALL(hipMalloc( (void**) &dev_lock, 2*sizeof(Lock)));
	CUDA_SAFE_CALL(hipMemcpy(dev_lock, lock, 2*sizeof(Lock), hipMemcpyHostToDevice));
	*/

	unsigned int timer = 0;

	CUDA_SAFE_CALL(hipMalloc( (void**)&c_array, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&c_new_array, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&analysis, size));
	CUDA_SAFE_CALL(hipMalloc( (void**)&reflection, size));


	temp = (char*) malloc(size);
	memset(temp,'\0',size);	

	CUDA_SAFE_CALL(hipMemcpy(c_array, c->ht, size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(c_new_array, '\0', size));
	CUDA_SAFE_CALL(hipMemset(analysis, '\0', size));
		
	dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 grid(1);

	int empty_const=1;
	int *d_empty_const;
	int size_int = sizeof(int);

	CUDA_SAFE_CALL(hipMalloc((void**)&d_empty_const,size_int));
	CUDA_SAFE_CALL(hipMemcpy(d_empty_const, &empty_const,size_int,hipMemcpyHostToDevice));

	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));

	/* Call Kernel function */
	init_constraints_kernel<<< grid, threads >>>(c_array, num_var, num_const, SIZE_CONSTRAINT, c_new_array, analysis, d_empty_const); 

	CUDA_SAFE_CALL(hipMemcpy(&empty_const, d_empty_const,4,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(reflection, c_new_array, size, hipMemcpyDeviceToDevice));
	//printf("(let (a (lambda c c)) in\n (let b 1) in\n (app a b) \n");
	
	//printf("frist constraint \n" );
	CUDA_SAFE_CALL(hipMemcpy(temp, c_new_array, size, hipMemcpyDeviceToHost));
	//print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
		
	char* out_c_new_array;
	char* out1,*out2;

	char* o1,*o2;

	o1 =(char*) malloc(9);
	o2 =(char*) malloc(9);
	bzero(o1,9);
	bzero(o2,9);	
	hipMalloc(&out1,8);
	hipMalloc(&out2,8);
	
	while (!empty_const) {	
		empty_const = 1;
		CUDA_SAFE_CALL(hipMemcpy(d_empty_const, &empty_const,size_int,hipMemcpyHostToDevice));
		
		CUDA_SAFE_CALL(hipMalloc(&out_c_new_array, size));
		CUDA_SAFE_CALL(hipMemset(out_c_new_array, '\0', size));
	
		// call a kernel function
		solve_constraints_kernel<<< grid, threads>>> (reflection, c_new_array, num_var, num_const, out_c_new_array, analysis, d_empty_const,out1,out2); //dev_lock		//printf("debug11\n");
		CUDA_SAFE_CALL(hipMemcpy(&empty_const, d_empty_const,size_int,hipMemcpyDeviceToHost));

		//printf("new constraint1:\n");		
		hipMemcpy(temp, out_c_new_array, size, hipMemcpyDeviceToHost);
		//print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);

		hipFree(c_new_array);

		c_new_array = out_c_new_array;	
		out_c_new_array = NULL;

		///printf("new analysis:\n");
		//hipMemcpy(temp2, analysis, size, hipMemcpyDeviceToHost);
		//print_memory(temp2, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);

		//printf("new constraint2:\n");
		hipMemcpy(temp, c_new_array, size, hipMemcpyDeviceToHost);
		//print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
	}


	CUT_SAFE_CALL(cutStopTimer(timer));
	printf("Processing time:%f (ms)\n", cutGetTimerValue(timer));
	CUT_SAFE_CALL(cutDeleteTimer(timer));

        hipMemcpy(o1, out1,8,hipMemcpyDeviceToHost);
        hipMemcpy(o2, out2,8,hipMemcpyDeviceToHost);


 	// now analysis contains final answers at each colume(var).
	// Need to show them: var -> value mapping by having way of finding var from column index.
	
	CUDA_SAFE_CALL(hipMemcpy(out_analysis, analysis, size, hipMemcpyDeviceToHost));
	printf("\nSOLUTIONS=========\n");
	print_memory(out_analysis, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
	hipFree(c_array);
	hipFree(c_new_array);
	hipFree(analysis);
	//hipFree(dev_lock);
	hipFree(out_c_new_array);
	
}


