#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<cutil.h>
#include "hashtab.h"
#include "cu_lib.h"

// return pointer of h[key,index]

extern "C"
__device__ int hashfunc(char *to){
	return *to -'a';	
}

extern "C"
__device__ char* get_element2(char* h, int num_var, int key, int index){
	int row = (key + index*num_var)*SIZE_CONSTRAINT;
	char *elem_addr = h + row;
	return elem_addr;
}


char* get_element3(char* h, int num_var, int key, int index){
	int row = (key + index*num_var)*SIZE_CONSTRAINT;
	char *elem_addr = h + row;
	return elem_addr;
}

extern "C"
__device__ int put_element2(char* h, int num_var, int num_const, int key, char* constraint){
	int i;
	for (i=0 ; i < num_const; i++){
		int row = (key + i*num_var)*SIZE_CONSTRAINT;
		char* temp = h + row;
		if (*temp == '\0'){
			memcpy(temp, constraint, SIZE_CONSTRAINT);
			return 1;
		}
	}	return -1; 
} 
	

extern "C"
__device__ void memcopy(char* dest, char* sour, int size){
	int i;
	for(i=0; i < size; i++)
		dest[i] = sour[i];

}

extern "C"
__global__ void test0(char *constraints, int num_var, int num_const, int sz, char* new_const, char* analysis, char* extra)
{
}
extern "C"
__global__ void test1(int arg1, int arg2, int arg3, int arg4, int arg5, int arg6, int arg7, int arg8, int arg9, 
char* byteset, char* byte_return, int* bargs)
{
    int i = threadIdx.x;

    // byte_return
    byte_return[i*4] = byteset[i*4];
    byte_return[i*4+1] = byteset[i*4+1];
    byte_return[i*4+2] = byteset[i*4+2];
    byte_return[i*4+3] = byteset[i*4+3];

    // bargs
    switch (i) {
    case 0:
            bargs[0] = arg1 * 10;
            break;
    case 1:
	    bargs[1] = arg2 * 10;
      	    break;
    case 2:
            bargs[2] = arg3 * 10;
            break;
    case 3:
   	bargs[3] = arg4 * 10;
    	break;
    case 4:
        bargs[4] = arg5 * 10;
        break;
    case 5:
        bargs[5] = arg6 * 10;
        break;
    case 6:
        bargs[6] = arg7 * 10;
           break;
    case 7:
        bargs[7] = arg8 * 10;
        break;
    case 8:
        bargs[8] = arg9 * 10;
     	break;
     }
}


extern "C"
__global__ void init_constraints_kernel(char *constraints, int num_var, int num_const, const int size_constraint, char* new_constraints, char* analysis, int* empty_contraint){

	int idx = threadIdx.x;
	int idy = threadIdx.y;
	
	char* a_constraint = get_element2(constraints,num_var,idx,idy);

	int pos = (idx + idy * num_var) * size_constraint;

/////////////////////////////input checker///////////////////////////
//       	memcpy(analysis+pos, "_|_|_|_|", size_constraint);
//       	memcpy(new_constraints+pos, "_|_|_|_|", size_constraint);
//	*empty_contraint = 0;

/////////////////////////////original///////////////////////////

	if( (constraints[pos] == '_') && ( constraints[pos+1] == 'v' || constraints[pos+1] == 'c' || constraints[pos+1] == 'l')){
       		memcpy(analysis+pos, a_constraint, size_constraint);
	} else {
       		memcpy(new_constraints+pos, a_constraint, size_constraint);
		*empty_contraint = 0;
	}

	__syncthreads();
}



extern "C"
__global__ void solve_constraints_kernel(char *reflection, char* constraint, int num_var, int num_const, char* new_constraints, char* analysis, int* empty_constraint,char* out1,char* out2){
	__shared__ char sh_constraints[BLOCK_SIZE_X * BLOCK_SIZE_Y * SIZE_CONSTRAINT];
	register char n_const[8];
	register char n_const2[8];
	register char value[8];

	int idx = threadIdx.x;
	int idy = threadIdx.y;
  	int i;

	//int z=2;
	//int x = idy*SIZE_CONSTRAINT;
	//int y = idx*SIZE_CONSTRAINT;	

	char* sharemem_addr = sh_constraints; 
	sharemem_addr = sharemem_addr+idx*SIZE_CONSTRAINT*BLOCK_SIZE_X+idy*SIZE_CONSTRAINT;
	memcpy(sharemem_addr, get_element2(constraint, num_var,idx, idy), SIZE_CONSTRAINT);	

	memcpy(out1,sharemem_addr,SIZE_CONSTRAINT);
	for(i=0; i < num_const; i++){
		memcopy(value,get_element2(analysis, num_var, idx ,i),SIZE_CONSTRAINT);
		//memcopy(out2,value,SIZE_CONSTRAINT);	
		if (*value != '\0'){
			memset(n_const, '\0', 8);
			memset(n_const2, '\0', 8);
			if((*sharemem_addr=='_' && *(sharemem_addr+1)=='P') && (value[0]=='_' && value[1] == 'v')){
				int index = hashfunc(sharemem_addr+2);
				memcpy(n_const,value,8);
				put_element2(analysis,num_var,num_const, index,n_const);
				//*empty_constraint = 1;
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1)=='C') && (value[0]=='_' && value[1] == 'c')){
				int index = hashfunc(value+2);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,sharemem_addr+2,6);
				put_element2(new_constraints, num_var,num_const, index,n_const);
				*empty_constraint = 0;
			} else if(((*sharemem_addr=='_' && *(sharemem_addr+1)=='D')) && (value[0]=='_' && value[1] == 'c')){
				int index = hashfunc(value+4);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,sharemem_addr+2,6);
				put_element2(new_constraints, num_var, num_const, index, n_const);
				*empty_constraint = 0;
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1)=='A') && (value[0]=='_' && value[1] == 'l')){
				int index1 = hashfunc(value+6);
				int index2 = hashfunc(sharemem_addr +4);
				n_const[0] = '_';
				n_const2[0]= '_';
				n_const[1] = 'P';
				n_const2[1]= 'P';
				memcopy(n_const+2,sharemem_addr+2,2);
				memcopy(n_const2+2,value+2,2);
				put_element2(new_constraints, num_var, num_const, index1, n_const);
				put_element2(new_constraints, num_var, num_const, index2, n_const2);
				memcopy(out1,n_const,8);
				memcopy(out2,n_const2,8);
				*empty_constraint = 0;
			} else if((*sharemem_addr=='_' && *(sharemem_addr+1) =='A') && (value[0]=='_' && value[1] == 't')){
				int index = hashfunc(sharemem_addr+4);
				n_const[0] = '_';
				n_const[1] = 'P';
				memcpy(n_const+2,value+2,6);
				put_element2(new_constraints, num_var, num_const, index, n_const);
				*empty_constraint = 0;
			} else {
				//*empty_constraint = 1;
			}
		}
	}

	__syncthreads();
}



void print_memory(char* dish, int szx, int szy, int szelt)
{
  int i,j,k,pos;
  for(i = 0 ; i < szx ; i++)
  for(j = 0 ; j < szy ; j++) {
    pos = (j*szx + i) * szelt;
    printf("[%d,%d]\t", i, j);
    for(k=0;k<szelt;k++)
      printf("%c", dish[pos+k]);
    printf("\t");
  }
  printf ("\n");
}

// those wrappers call kernel functions
void solver_constraint_wrapper(hash_tab* c, char* out_analysis){
	int num_var = c->num_variable;
	int num_const = c->num_constraint;
	int size = num_var * num_const * SIZE_CONSTRAINT;
	//int i;
	char *c_array;
	char *c_new_array;
	char *analysis;
	char *temp; 
	char *reflection;

	hipMalloc( (void**)&c_array, size);
	hipMalloc( (void**)&c_new_array, size);
	hipMalloc( (void**)&analysis, size);
	hipMalloc( (void**)&reflection, size);


	temp = (char*) malloc(size);
	memset(temp,'\0',size);	

	hipMemcpy(c_array, c->ht, size, hipMemcpyHostToDevice);
	hipMemcpy(reflection, c->ht, size, hipMemcpyHostToDevice);
	hipMemset(c_new_array, '\0', size);
	hipMemset(analysis, '\0', size);
		
	dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 grid(1);

	int empty_const=1;
	int *d_empty_const;
	int size_int = sizeof(int);
	hipMalloc((void**)&d_empty_const,size_int);
	hipMemcpy(d_empty_const, &empty_const,size_int,hipMemcpyHostToDevice);

	init_constraints_kernel<<< grid, threads >>>(c_array, num_var, num_const, SIZE_CONSTRAINT, c_new_array, analysis, d_empty_const); 

	hipMemcpy(&empty_const, d_empty_const,4,hipMemcpyDeviceToHost);
	printf("(let (a (lambda c c)) in\n (let b 1) in\n (app a b) \n");
	
	printf("frist constraint \n" );
	hipMemcpy(temp, c_new_array, size, hipMemcpyDeviceToHost);
	print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
		
	char* out_c_new_array;
	char* out1,*out2;

	char* o1,*o2;

	o1 =(char*) malloc(9);
	o2 =(char*) malloc(9);
	bzero(o1,9);
	bzero(o2,9);	
	hipMalloc(&out1,8);
	hipMalloc(&out2,8);

	while (!empty_const) {	
		empty_const = 1;
		hipMemcpy(d_empty_const, &empty_const,size_int,hipMemcpyHostToDevice);
		
		hipMalloc(&out_c_new_array, size);
		hipMemset(out_c_new_array, '\0', size);
		
		solve_constraints_kernel<<< grid, threads>>> (reflection, c_new_array, num_var, num_const, out_c_new_array, analysis, d_empty_const,out1,out2);
		//printf("debug1\n");
		hipMemcpy(&empty_const, d_empty_const,size_int,hipMemcpyDeviceToHost);

		//printf("new constraint1:\n");		
		//hipMemcpy(temp, out_c_new_array, size, hipMemcpyDeviceToHost);
		//print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);

		hipFree(c_new_array);

		c_new_array = out_c_new_array;	
		out_c_new_array = NULL;

		///printf("new analysis:\n");
		//hipMemcpy(temp2, analysis, size, hipMemcpyDeviceToHost);
		//print_memory(temp2, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);

		printf("new constraint2:\n");
		hipMemcpy(temp, c_new_array, size, hipMemcpyDeviceToHost);
		print_memory(temp, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
	}

        hipMemcpy(o1, out1,8,hipMemcpyDeviceToHost);
        hipMemcpy(o2, out2,8,hipMemcpyDeviceToHost);
	printf("debug:::<%s> %s\n",o1,o2);


 	// now analysis contains final answers at each colume(var).
	// Need to show them: var -> value mapping by having way of finding var from column index.
	
	hipMemcpy(out_analysis, analysis, size, hipMemcpyDeviceToHost);
	printf("\nSOLUTIONS=========\n");
	print_memory(out_analysis, BLOCK_SIZE_X, BLOCK_SIZE_Y, SIZE_CONSTRAINT);
	hipFree(c_array);
	hipFree(c_new_array);
	hipFree(analysis);
	hipFree(out_c_new_array);
	
}


